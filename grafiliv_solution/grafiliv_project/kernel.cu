
#include <hip/hip_runtime.h>
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);
    
	// Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

	hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
}
