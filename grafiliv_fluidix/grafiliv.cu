#include "hip/hip_runtime.h"
#include "fluidix.h"

#define W 100 // world size (cube)
#define N 1000 // number of particles
#define STEPS 100000 // number of simulation steps
#define DT 0.01f // integration time-step
#define RANGE 2.0f // fluid interaction cutoff range
#define HARDNESS 20.0f // repulsive force strength
#define FLUID_DENSITY 1.0f

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 40


#define PHOTOSYNTHESIS_MAX 10.0f

#define G -9.81f

struct Global {
} g;

struct Particle {
	xyz r, v, f; 				// position, velocity, force
	xyz signal;
	xyz sense;
	float density;
	float radius;
	float energy;
	float color; 				// display color
	float alpha;

	//Genome:

	float phoSyAbility; 		// Ability to photosynthesize
	float moveAbility;			// Ability to move
	float senseAbility;			// Ability to sense surroundings

	bool  invertSignal;			// Inverts signal before acting or transmitting
	float recieveSigAbility;  	// Ability to recieve signals	
	float sendSigAbility;		// Ability to send signals
};

// initialize particles
FUNC_EACH(init,

	p.energy = 10.0f;
	p.r = make_xyz_uniform() * W; 					//Random position within box
	p.radius = abs(rnd_normal()); 					//Random size
	p.density = abs(rnd_normal()+FLUID_DENSITY); 	//Random density
	p.alpha = 1.0f;
	
	p.invertSignal 		= rnd_uniform() < 0.5f;
	p.phoSyAbility 		= abs(rnd_normal());
	p.moveAbility 		= abs(rnd_normal());
	p.senseAbility 		= abs(rnd_normal());
	p.recieveSigAbility = abs(rnd_normal());
	p.sendSigAbility 	= abs(rnd_normal());

	//p.color = p.moveAbility;
	
	
	p.sense  = make_xyz(0, 0, 0);
	p.signal = make_xyz_uniform()*5;	//Random initial signal
)

// linear repulsion + attraction at distance
FUNC_PAIR(pair,
	float ratio = dr / range;

	xyz f = u * (REPULSION_FORCE * (1-ratio) - ATTRACTION_FORCE * ratio);
	addVector(p1.f, f);
	addVector(p2.f, -f);

	// Update sense values
	addVector(p1.sense, 2*(-u));	// These should be normalized...
	addVector(p2.sense, 2*u);

	// Update signal values with sense
	addVector(p1.signal, p1.sense * p1.senseAbility);
	addVector(p2.signal, p2.sense * p2.senseAbility);

	// Transmit signal
	addVector(p1.signal, p2.signal * p1.recieveSigAbility * p2.sendSigAbility * (p1.invertSignal ? -1 : 1));
	addVector(p2.signal, p1.signal * p2.recieveSigAbility * p1.sendSigAbility * (p2.invertSignal ? -1 : 1));
	
	p1.color = xyz_len(p1.signal)/100;
	p2.color = xyz_len(p2.signal)/100;
)


// buoyancy 
FUNC_EACH(buoyancy,
	float volume = p.radius * p.radius * PI;
	p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

// photosynthesis 
FUNC_EACH(photosynthesis,
	p.energy += PHOTOSYNTHESIS_MAX * p.phoSyAbility;
)

// move 
FUNC_EACH(move,
	xyz f = p.signal * p.moveAbility * 0.001f;
	addVector(p.f, f);
)

// handle energy usage
FUNC_EACH(handleEnergy,
	p.energy -= (
		p.moveAbility * 5 + 
		p.senseAbility * 3
	);
	//p.color = p.energy;

	// If dead
	if(p.energy <= 0)
	{
		p.alpha = 0.5f;
		p.density = FLUID_DENSITY;
		p.phoSyAbility = p.moveAbility = p.senseAbility = p.recieveSigAbility = p.sendSigAbility = 0.0f;
	}
)

// Euler integration
FUNC_EACH(integrate,
	//p.color = xyz_len(p.f) / 50.0f;

	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
	if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }
)

// deflate old signals so that they won't overflow 
FUNC_EACH(deflateSignals,
	p.signal = xyz_norm(p.signal);
	p.sense = xyz_norm(p.signal);
	
	//p.color = p.signal;
)


// simulation
int main(int argc, char **argv) {
	Fluidix<> *fx = new Fluidix<>(&g);

	int A = fx->createParticleSet(N);
	fx->runEach(init(), A);

	for (int i = 0; i < STEPS; i++) {

		printf("step %d / %d: ", i, STEPS);
		fx->setTimer();

		// execute interactions
		fx->runPair(pair(), A, A, RANGE);
		fx->runEach(buoyancy(), A);
		fx->runEach(move(), A);
		fx->runEach(integrate(), A);
		fx->runEach(handleEnergy(), A);
		fx->runEach(boundary(), A);
		fx->runEach(deflateSignals(), A);
		fx->runEach(photosynthesis(), A);

		printf("%.1f ms\n", fx->getTimer());

		// only output to file every 10th step
		if (i % 10 == 0) fx->outputFrame("sample");
	}

	delete fx;
}

