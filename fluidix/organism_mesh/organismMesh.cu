#include "hip/hip_runtime.h"
#include "fluidix.h"
#include "../lib/genome.h"

#define L 8

struct Global {
	float grid_pack;
	int3 grid_num;
	xyz grid_origin;
} g;

struct Particle {
	xyz r, v, f;
	float color;
	float alpha;
};

FUNC_EACH(put_in_grid,
	int3 ijk;
	ijk.x = p_index % g.grid_num.x;
	ijk.y = (p_index / g.grid_num.x) % g.grid_num.y;
	ijk.z = p_index / (g.grid_num.x * g.grid_num.y);

	p.r = g.grid_origin + g.grid_pack * int3_to_xyz(ijk);
)

void createParticles(Fluidix<> *fx, xyz origin, int side){
	xyz box_min = origin;
	xyz box_max = origin + make_xyz(side, side, side);

	g.grid_pack = 1.0f; // distance between adjacent particles

	xyz box_size = box_max - box_min;

	g.grid_num.x = (int)roundf(box_size.x / g.grid_pack);
	g.grid_num.y = (int)roundf(box_size.y / g.grid_pack);
	g.grid_num.z = (int)roundf(box_size.z / g.grid_pack);

	g.grid_origin = (box_max + box_min - g.grid_pack * (int3_to_xyz(g.grid_num) - 1)) / 2;

	int setA = fx->createParticleSet(g.grid_num.x * g.grid_num.y * g.grid_num.z);
	fx->runEach(put_in_grid(), setA);

	int inputs = 4;
	int outputs = 2;
	Genome g(inputs, outputs);
	g.mutate();
	g.printMathematica();

	Particle *p = fx->getParticleArray(setA);
	int n = fx->getParticleCount(setA);

	for(int step=0; step<20; step++) {
		g.mutate();
		for (int i = 0; i < n; i++) {
			xyz dr = p[i].r - make_xyz(L/2, L/2, L/2);
	
			vector<float> input;
			input.push_back(dr.x);
			input.push_back(dr.y);
			input.push_back(dr.z);
			input.push_back(xyz_len(dr));
	
			vector<float> output = g.getOutput(input);
	
			printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
			printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");
	
			p[i].color = output[0];
			p[i].alpha = output[1]>0 ? 1.0f : 0.0f;
	
			// apply changes before any operation
			fx->applyParticleArray(setA);
		}
		g.printMathematica();
		g.printGenome();
		fx->outputFrame("output");
	}
}

int main() {
	Fluidix<> *fx = new Fluidix<>(&g);
	
	createParticles(fx, make_xyz(0, 0, 0), L);


	delete fx;
}
