#include "hip/hip_runtime.h"
//#include "fluidix.h"
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/genome.h"
#include <queue>
#include <ppl.h>
#include <concurrent_queue.h>

#define DT 0.01f // integration time-step
#define CELL_INITIAL_ENERGY 100.0f
#define PELLET_LIFETIME 5.0f

#define W 300
#define N 100000
#define N_ORIGIN_CELLS 2000
#define N_INITIAL_BUFFER 100
#define N_STEPS 1000000

//Inputs x,y,z,d:
#define N_INPUTS 4

#define RANGE 3.0f
#define MOVE_FACTOR 50

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 250

#define CELL_MIN_ENERGY 0.5f
#define PELLET_MIN_ENERGY 0.01f
#define DIVISION_ENERGY (CELL_MIN_ENERGY * 4.0f)
#define ENERGY_PARTICLE_ENERGY 1.0f
#define MAX_CELL_DIVISIONS 5

#define CELL_METABOLISM 0.5f
#define CELL_DECAY_RATE 0.5f

#define FLUID_DENSITY 1.0f
#define G -9.81f

#define GREEN   0.5f
#define RED     1.0f
#define YELLOW  0.7f
#define BLUE    0.0f
#define CYAN    0.3f
#define ORANGE  0.8f

#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;

enum CellType {
    Photo, Digest, Sting, Vascular, Fat, Sense, Motor, Ballast, Egg, 
    N_CELL_TYPES
};
enum ParticleType {
    Cell, Energy, Pellet, Buffer,
    N_PARTICLE_TYPES
};

int currGenomeIndex;

struct Global {
    int nCells;
} g;

struct Particle {
    ParticleType particleType;
    //float growthProb;
    xyz r, v, f;
    float color;
    float radius;
    float alpha;
    float density;
    float energy;
    float energyIn;
    float energyOut;
    float maxEnergy;
    int nDivisions;
    float signal;
    int organism;
    bool toGrow;
    bool toBuffer;
    bool toReproduce;
    Particle *origin;
    Particle *parent;
    CellType type;
};

#define turnIntoPellet(p) {         \
    p.particleType = Pellet;        \
    p.energy = PELLET_LIFETIME;     \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.organism = -1;                \
}                                   \

#define turnIntoEnergy(p) {         \
    p.particleType = Energy;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W,            \
        rnd_uniform()*W + W,        \
        rnd_uniform()*W             \
    );                              \
    p.color = 0.7f;                 \
    p.energy = ENERGY_PARTICLE_ENERGY; \
    p.alpha = 0.3f;                 \
    p.radius = 0.5f;                \
    p.density = FLUID_DENSITY * 20; \
}

#define turnIntoBuffer(p) {        \
    p.particleType = Buffer;       \
    p.r = make_xyz(                 \
        rnd_uniform()*W,            \
        rnd_uniform()*W - W,        \
        rnd_uniform()*W             \
    );                              \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.color = 0.5f;                 \
    p.radius = 1.0f;                \
    p.organism = -1;                \
}

FUNC_EACH(init,
    turnIntoEnergy(p);
)

FUNC_EACH(integrate,
    p.v += p.f * DT;
    p.r += p.v * DT;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
)

FUNC_EACH(handleEnergy,
    switch (p.particleType) {
    case Cell:
        p.energy -= CELL_METABOLISM * DT;
        if (p.energy < CELL_MIN_ENERGY || p.origin->particleType != Cell)
            turnIntoPellet(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Pellet:
        p.energy -= CELL_DECAY_RATE * DT;
        if (p.energy <= PELLET_MIN_ENERGY)
            p.toBuffer = true;
        break;
    }
    //p.color = mapf(p.energy, 0.0f, 10.0f, 0.0f, 1.0f);
)

FUNC_EACH(buoyancy,
    if (p.particleType == Cell && p.type == Ballast)
        p.density = clamp(p.density + p.signal, 0.5f, 2.0f);
    float volume = p.radius * p.radius * PI;
    p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

FUNC_EACH(moveParticle,
    if (p.particleType == Cell && p.type == Motor) {
        xyz f = xyz_norm(p.origin->r - p.r) * MOVE_FACTOR;
        addVector(p.f, f);
    }
)

FUNC_EACH(reproduction,
    if (p.particleType == Cell && p.type == Egg) {
        if (p.energy >= p.maxEnergy) //rnd_uniform() < p.signal)
            p.toReproduce = true;
    }
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        //printf("Weird particle!!! type (%i:%i)\n", p.particleType, p.type);
        turnIntoBuffer(p);
        p.toBuffer = true;
    } if (p.particleType != Buffer) {
        if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
        if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
        if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
        if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }

        if (p.particleType == Energy){
            if (p.r.y < 0) {
                p.toBuffer = true;
            }
        }
        else {
            if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
            if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
        }
    }
)

FUNC_EACH(growth,
    if (p.particleType == Cell)
        addInteger(g.nCells, 1);

    if (p.particleType == Cell && p.energy > DIVISION_ENERGY && p.nDivisions--) //rnd_uniform() < p.growthProb)
        p.toGrow = true;
)

#define SPRING_K 100.0f // spring constant
FUNC_EACH(springToParent,
    if (p.particleType == Cell && p.parent != nullptr &&
        p.parent->particleType == Cell &&
        !isWeirdParticlePointer(p.parent)
        ) {
        printf("parent.r=(%.2f, %.2f, %.2f)\n", p.parent->r.x, p.parent->r.y, p.parent->r.z);
        xyz u = (p.r - p.parent->r);
        float dr = xyz_len(u);
        xyz f = u * ((dr - (p.radius + p.parent->radius)) * SPRING_K);
        addVector(p.f, f);
        //addVector(p.parent->f, -f);
    }
)

#define consumeParticle(a, b) {     \
    addFloat(a.energy, b.energy * 0.9f);   \
    b.toBuffer = true;             \
}

FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
        //float ratio = dr/range;

        float attraction = 0.0f;
        if (p1.particleType == Cell &&
            p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                //Attraction between cells of same organism
                attraction = ATTRACTION_FORCE * ratio;

                //Signalling between cells of same organism
                float meanSignal = (p1.signal + p2.signal) / 2;
                p1.signal = p2.signal = meanSignal;

                //Energy transmission
                float p1Surplus = maxf(p1.energy - CELL_MIN_ENERGY, 0);
                float p2Surplus = maxf(p2.energy - CELL_MIN_ENERGY, 0);
                transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting) turnIntoPellet(p2);
                if (p2.type == Sting) turnIntoPellet(p2);
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Pellet)
                ) consumeParticle(p1, p2)
        }
        //If p2 is a cell
        else if (p2.particleType == Cell) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Pellet)
                ) consumeParticle(p2, p1)
        }

        if (p1.particleType == Cell && p1.type == Sense) p1.signal += 0.1f;
        if (p2.particleType == Cell && p2.type == Sense) p2.signal += 0.1f;

        xyz f = u * (REPULSION_FORCE * (1 - ratio) - attraction);

        addVector(p1.f, f);
        addVector(p2.f, -f);

        //p1.color = p1.signal;
        //p2.color = p2.signal;
    }
)

void setDefaultCellValues(Particle *cell) {
    cell->alpha = 1.0f;
    cell->radius = 1.0f;
    cell->energy = CELL_INITIAL_ENERGY;
    cell->density = FLUID_DENSITY * 1.1f;
    cell->particleType = Cell;
}

void applyPhenotype(vector<float> output, Particle *cell) {
    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    switch (cell->type) {
    case Photo:
        cell->color = GREEN;
        cell->energyIn = 0.0f;
        cell->energyOut = 0.9f;
        cell->maxEnergy = 3.0f;
        break;
    case Digest:
        cell->color = RED;
        cell->energyIn = 0.0f;
        cell->energyOut = 0.9f;
        cell->maxEnergy = 3.0f;
        break;
    case Fat:
        cell->color = YELLOW;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 10.0f;
        break;
    case Motor:
        cell->color = 0.4f;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Sense:
        cell->color = BLUE;
        cell->energyIn = 0.5f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Ballast:
        cell->color = CYAN;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Egg:
        cell->color = ORANGE;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = DIVISION_ENERGY;
        break;
    case Vascular:
        cell->color = 0.2;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.9f;
        cell->maxEnergy = 1.0f;
        break;
    case Sting:
        cell->color = 0.85f;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
    }
    if (cell->origin == cell) {
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
    }
    cell->nDivisions = output[N_CELL_TYPES] * MAX_CELL_DIVISIONS;
}

// Initialize new organism, not inheriting anything
void initializeNewOrganism(Particle *cell, Genome *genome) {
    // Define number of in- and outputs
    int inputs = N_INPUTS;              // X, Y, Z, Dist
    int nonCelltypeOutputs = 1;         // Growth prob
    int outputs = N_CELL_TYPES + nonCelltypeOutputs;

    *genome = Genome(inputs, outputs);
    genome->mutate();

    cell->organism = currGenomeIndex++;
    cell->r = make_xyz(
        rnd_uniform() * W,
        rnd_uniform() * W, // * 0.2f + 0.8f,
        rnd_uniform() * W
        );
    cell->origin = cell;
    cell->parent = nullptr;
    cell->toReproduce = false;
    setDefaultCellValues(cell);

    vector<float> input(inputs, 0.0f); //Input origin
    vector<float> output = genome->getOutput(input);

    applyPhenotype(output, cell);
}

// Initialize organism, inheriting from parent
void initializeOffspring(Particle *cell, Genome *genome) {
    genome->mutate();
    cell->organism = currGenomeIndex++;
    cell->origin = cell;
    cell->parent = nullptr;
    cell->toReproduce = false;

    // Define number of in- and outputs
    int inputs = N_INPUTS; // X, Y, Z, Dist

    vector<float> input(inputs, 0.0f); //Input origin
    vector<float> output = genome->getOutput(input);

    applyPhenotype(output, cell);
}

void growCell(Particle *parent, Particle *child, Genome *genomeParent, Genome *genomeChild) {
    normal_distribution<float> rndNormal(0.0f, 1.0f);
    
    //Half of parent's energy goes to the child
    parent->energy /= 2;

    //  Copy constructor
    *child = Particle(*parent);
    *genomeChild = Genome(*genomeParent);

    // Displace particles from each other
    xyz displacement = xyz_norm(
        make_xyz(
        rndNormal(rndGen),
        rndNormal(rndGen),
        rndNormal(rndGen)
        )) * parent->radius;

    parent->r -= displacement;
    child->r += displacement;

    child->parent = parent;

    xyz dr = child->r - child->origin->r;

    vector<float> input;
    input.push_back(dr.x);
    input.push_back(dr.y);
    input.push_back(dr.z);
    input.push_back(xyz_len(dr));

    genomeChild->mutate();

    vector<float> output = genomeChild->getOutput(input);

    //printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
    //printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

    applyPhenotype(output, child);
}

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

int main() {
    Fluidix<> *fx = new Fluidix<>(&g);
    int setA = fx->createParticleSet(N);

    currGenomeIndex = 0;
    g.nCells = 0;
    Genome *genomes = new Genome[N];

    fx->runEach(init(), setA);

    Particle *p = fx->getParticleArray(setA);
    for (int i = 0; i < N_ORIGIN_CELLS; i++)
        initializeNewOrganism(&p[i], &genomes[i]);

    concurrent_queue<int> particleBuffer;

    for (int i = N_ORIGIN_CELLS; i < N_ORIGIN_CELLS + N_INITIAL_BUFFER; i++) {
        turnIntoBuffer(p[i]);
        p[i].r.y -= W;
        particleBuffer.push(i);
    }
    fx->applyParticleArray(setA);

    for (int step = 0; step < N_STEPS; step++) {
        g.nCells = 0;
//      fx->runEach(springToParent(), setA);
        fx->runEach(boundary(), setA);
        fx->runPair(particlePair(), setA, setA, RANGE);
        fx->runEach(moveParticle(), setA);
        fx->runEach(buoyancy(), setA);
        fx->runEach(handleEnergy(), setA);
        fx->runEach(growth(), setA);
        fx->runEach(integrate(), setA);
        parallel_for (int(0), N, [&](int i)
        {
            if (p[i].toBuffer) {
                if (particleBuffer.unsafe_size() > N_INITIAL_BUFFER) {
                    turnIntoEnergy(p[i]);
                    fx->applyParticleArray(setA);
                } else {
                    turnIntoBuffer(p[i]);
                    particleBuffer.push(i);
                }
                p[i].toBuffer = false;
            }
            if (p[i].particleType == Cell) {
                if (p[i].toGrow &&
                    p[i].origin != nullptr &&
                    p[i].origin->particleType == Cell &&
                    p[i].origin->organism == p[i].organism &&
                    !particleBuffer.empty()
                    ) {
                        int parent = i;
                        int child;
                        if (particleBuffer.try_pop(child)) {
                            p[i].toGrow = false;
                            growCell(&p[parent], &p[child], &genomes[parent], &genomes[child]);
                            fx->applyParticleArray(setA);
                        }
                }
                // Create offspring:
                if (p[i].type == Egg && p[i].toReproduce) {
                    initializeOffspring(&p[i], &genomes[i]);
                    fx->applyParticleArray(setA);
                }
            }
        });

        if (step % 10 == 0) {
            printf("nCells: %i\t", g.nCells);
            printf("step %d\n", step);
            fx->outputFrame("output");
        }

        if (!g.nCells) break;
    }
    delete[] genomes;
    delete fx;

    //system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
}
