#include "hip/hip_runtime.h"
//#include "fluidix.h"
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/genome.h"
#include <queue>

#define DT 0.01f // integration time-step
#define CELL_INITIAL_ENERGY 100.0f
#define PELLET_LIFETIME 5.0f

#define W 300
#define N 100000
#define N_ORIGIN_CELLS 2000
#define N_INITIAL_BUFFER 1000
#define N_STEPS 1000000

//Inputs x,y,z,d:
#define N_INPUTS 4

#define RANGE 3.0f
#define MOVE_FACTOR 100

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 250

#define CELL_MIN_ENERGY 0.5f
#define PELLET_MIN_ENERGY 0.01f
#define DIVISION_ENERGY (CELL_MIN_ENERGY * 8.0f)
#define ENERGY_PARTICLE_ENERGY 10.0f
#define MAX_CELL_DIVISIONS 10

#define CELL_METABOLISM 0.5f
#define CELL_DECAY_RATE 0.1f

#define FLUID_DENSITY 1.0f
#define G -9.81f

#define GREEN   0.5f
#define RED     1.0f
#define YELLOW  0.7f
#define BLUE    0.0f
#define CYAN    0.3f
#define ORANGE  0.8f

#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}

using namespace std;

enum CellType {
    Photo, Digest, Sting, Vascular, Fat, Sense, Motor, Ballast, Egg, 
    N_CELL_TYPES
};
enum ParticleType {
    Cell, Energy, Pellet, Buffert,
    N_PARTICLE_TYPES
};

int currGenomeIndex;

struct Global {
    int nCells;
} g;

struct Particle {
    ParticleType particleType;
    //float growthProb;
    xyz r, v, f;
    float color;
    float radius;
    float alpha;
    float density;
    float energy;
    float energySharing;
    float maxEnergy;
    int nDivisions;
    float signal;
    int organism;
    bool toGrow;
    bool reproduce;
    Genome genome;
    Particle *origin;
    CellType type;
};

#define turnIntoPellet(p) {         \
    p.particleType = Pellet;        \
    p.energy = PELLET_LIFETIME;     \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.organism = NULL;              \
}                                   \

#define turnIntoEnergy(p) {         \
    p.particleType = Energy;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W,            \
        rnd_uniform()*W + W,        \
        rnd_uniform()*W             \
    );                              \
    p.color = 0.7f;                 \
    p.energy = ENERGY_PARTICLE_ENERGY; \
    p.alpha = 0.3f;                 \
    p.radius = 0.5f;                \
    p.density = FLUID_DENSITY * 20; \
}

#define turnIntoBuffert(p) {        \
    p.particleType = Buffert;       \
    p.r = make_xyz(                 \
        rnd_uniform()*W,            \
        rnd_uniform()*W - W,        \
        rnd_uniform()*W             \
    );                              \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.color = 0.5f;                 \
    p.radius = 1.0f;                \
    p.organism = NULL;              \
}

FUNC_EACH(init,
    turnIntoEnergy(p);
    //p.r = (make_xyz_uniform() * W) + make_xyz(0, W, 0);
    //p.color = 0.7f;
    //p.alpha = 0.3f;
    //p.radius = 0.5f;
    //p.density = FLUID_DENSITY * 10;
    //p.particleType = Energy;
    //p.toGrow = false;
    //p.growthProb = -1.0f;
    //p.energy = ENERGY_PARTICLE_ENERGY;
    //p.signal = 0.0f;
    //p.organism = -1;
    //p.reproduce = false;
    //p.origin = NULL;
)

FUNC_EACH(integrate,
    p.v += p.f * DT;
    p.r += p.v * DT;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
)

FUNC_EACH(handleEnergy,
    switch (p.particleType) {
    case Cell:
        p.energy -= CELL_METABOLISM * DT;
        if (p.energy < CELL_MIN_ENERGY || p.origin->particleType != Cell)
            turnIntoPellet(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Pellet:
        p.energy -= CELL_DECAY_RATE * DT;
        if (p.energy <= PELLET_MIN_ENERGY)
            turnIntoBuffert(p)
        break;
    }
    //p.color = mapf(p.energy, 0.0f, 10.0f, 0.0f, 1.0f);
)

FUNC_EACH(buoyancy,
    if (p.particleType == Cell && p.type == Ballast)
        p.density = clamp(p.density + p.signal, 0.5f, 2.0f);
    float volume = p.radius * p.radius * PI;
    p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

FUNC_EACH(moveParticle,
    if (p.particleType == Cell && p.type == Motor) {
        xyz f = xyz_norm(p.origin->r - p.r) * MOVE_FACTOR;
        addVector(p.f, f);
    }
)

FUNC_EACH(reproduction,
    if (p.particleType == Cell && p.type == Egg) {
        if (rnd_uniform() < p.signal)
            p.reproduce = true;
    }
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
    // Check for wierd 1.#R values... NaN?
    if (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z) {
        printf("Weird particle!!! type (%i:%i)\n", p.particleType, p.type);
        turnIntoBuffert(p);
    } if (p.particleType != Buffert) {
        if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
        if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
        if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
        if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }

        if (p.particleType == Energy){
            if (p.r.y < 0) {
                turnIntoBuffert(p);
            }
        }
        else {
            if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
            if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
        }
    }
)

FUNC_EACH(growth,
    if (p.particleType == Cell)
        addInteger(g.nCells, 1);

    if (p.particleType == Cell && p.energy > DIVISION_ENERGY && p.nDivisions--) //rnd_uniform() < p.growthProb)
        p.toGrow = true;
    )

#define consumeParticle(a, b) {     \
    addFloat(a.energy, b.energy);   \
    turnIntoBuffert(b);             \
}


//float temp = a.energy;  \
//printf("Particle (type %i-%i) consumes another, energy increases from %.2f to %.2f\n", a.particleType, a.type, temp, a.energy); \

FUNC_PAIR(particlePair,
    if (p1.particleType != Buffert && p2.particleType != Buffert) {
        float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
        //float ratio = dr/range;

        float attraction = 0.0f;
        if (p1.particleType == Cell &&
            p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                //Attraction between cells of same organism
                attraction = ATTRACTION_FORCE * ratio;

                //Signalling between cells of same organism
                float meanSignal = (p1.signal + p2.signal) / 2;
                p1.signal = p2.signal = meanSignal;

                //Energy transmission
                float p1Surplus = maxf(p1.energy - CELL_MIN_ENERGY, 0);
                float p2Surplus = maxf(p2.energy - CELL_MIN_ENERGY, 0);
                transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energySharing);
                transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energySharing);
            }
            //Cells from different organisms
            else {
                //Damage the other cell if you are sting
                if (p1.type == Sting) p2.energy -= 0.1;
                if (p2.type == Sting) p1.energy -= 0.1;
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Pellet)
                ) consumeParticle(p1, p2)
        }
        //If p2 is a cell
        else if (p2.particleType == Cell) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Pellet)
                ) consumeParticle(p2, p1)
        }

        if (p1.particleType == Cell && p1.type == Sense) p1.signal += 0.1f;
        if (p2.particleType == Cell && p2.type == Sense) p2.signal += 0.1f;

        xyz f = u * (REPULSION_FORCE * (1 - ratio) - attraction);

        addVector(p1.f, f);
        addVector(p2.f, -f);

        //p1.color = p1.signal;
        //p2.color = p2.signal;
    }
)

void setDefaultCellValues(Particle *cell) {
    cell->alpha = 1.0f;
    cell->radius = 1.0f;
    cell->energy = CELL_INITIAL_ENERGY;
    cell->density = FLUID_DENSITY;
    cell->particleType = Cell;
}

void applyPhenotype(vector<float> output, Particle *cell) {
    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    switch (cell->type) {
    case Photo:
        cell->color = GREEN;
        cell->energySharing = 0.8f;
        cell->maxEnergy = 5.0f;
        break;
    case Digest:
        cell->color = RED;
        cell->energySharing = 0.8f;
        cell->maxEnergy = 5.0f;
        break;
    case Fat:
        cell->color = 0.4f;
        cell->energySharing = 0.1f;
        cell->maxEnergy = 20.0f;
        break;
    case Motor:
        cell->color = YELLOW;
        cell->energySharing = 0.0f;
        cell->maxEnergy = 5.0f;
        break;
    case Sense:
        cell->color = BLUE;
        cell->energySharing = 0.5f;
        cell->maxEnergy = 5.0f;
        break;
    case Ballast:
        cell->color = CYAN;
        cell->energySharing = 0.0f;
        cell->maxEnergy = 5.0f;
        break;
    case Egg:
        cell->color = ORANGE;
        cell->energySharing = 0.0f;
        cell->maxEnergy = 10.0f;
        break;
    default:
        cell->energySharing = 0.5f;
        cell->maxEnergy = 5.0f;
    }
    cell->nDivisions = output[N_CELL_TYPES] * MAX_CELL_DIVISIONS;
}

// Initialize new organism, not inheriting anything
void initializeNewOrganism(Particle *cell) {
    // Define number of in- and outputs
    int inputs = N_INPUTS;              // X, Y, Z, Dist
    int nonCelltypeOutputs = 1;         // Growth prob
    int outputs = N_CELL_TYPES + nonCelltypeOutputs;

    cell->genome = Genome(inputs, outputs);
    cell->genome.mutate();

    cell->organism = currGenomeIndex++;
    cell->r = make_xyz(
        rnd_uniform() * W,
        rnd_uniform() * W, // * 0.2f + 0.8f,
        rnd_uniform() * W
        );
    cell->origin = cell;
    cell->reproduce = false;
    setDefaultCellValues(cell);

    vector<float> input(inputs, 0.0f); //Input origin
    vector<float> output = cell->genome.getOutput(input);

    applyPhenotype(output, cell);
}

// Initialize organism, inheriting from parent
void initializeOffspring(Particle *cell) {
    cell->genome.mutate();
    cell->organism = currGenomeIndex++;
    cell->origin = cell;
    cell->reproduce = false;

    // Define number of in- and outputs
    int inputs = N_INPUTS;                      // X, Y, Z, Dist

    vector<float> input(inputs, 0.0f); //Input origin
    vector<float> output = cell->genome.getOutput(input);

    applyPhenotype(output, cell);
}

void growCell(Particle *parent, Particle *child) {
    normal_distribution<float> rndNormal(0.0f, 1.0f);
    
    //Half of parent's energy goes to the child
    parent->energy /= 2;

    //  Copy constructor
    *child = Particle(*parent);

    // Displace particles from each other
    xyz displacement = xyz_norm(
        make_xyz(
        rndNormal(rndGen),
        rndNormal(rndGen),
        rndNormal(rndGen)
        )) * parent->radius;

    parent->r -= displacement;
    child->r += displacement;

    xyz dr = child->r - child->origin->r;

    vector<float> input;
    input.push_back(dr.x);
    input.push_back(dr.y);
    input.push_back(dr.z);
    input.push_back(xyz_len(dr));

    child->genome.mutate();

    vector<float> output = child->genome.getOutput(input);

    //printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
    //printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

    applyPhenotype(output, child);
}

int main() {
    Fluidix<> *fx = new Fluidix<>(&g);
    int setA = fx->createParticleSet(N);

    currGenomeIndex = 0;
    g.nCells = 0;

    fx->runEach(init(), setA);

    Particle *p = fx->getParticleArray(setA);
    for (int i = 0; i < N_ORIGIN_CELLS; i++)
        initializeNewOrganism(&p[i]);
    
    queue<Particle*> particleBuffert;
    for (int i = N_ORIGIN_CELLS; i < N_ORIGIN_CELLS + N_INITIAL_BUFFER; i++) {
        turnIntoBuffert(p[i]);
        p[i].r.y -= W;
        particleBuffert.push(&p[i]);
    }
    fx->applyParticleArray(setA);

    for (int step = 0; step < N_STEPS; step++) {
        g.nCells = 0;
        fx->runEach(boundary(), setA);
        fx->runPair(particlePair(), setA, setA, RANGE);
        fx->runEach(moveParticle(), setA);
        fx->runEach(buoyancy(), setA);
        fx->runEach(integrate(), setA);
        fx->runEach(handleEnergy(), setA);
        fx->runEach(growth(), setA);
        for (int i = 0; i < N; i++) {
            if (p[i].particleType == Buffert) {
                if (particleBuffert.size() > N_INITIAL_BUFFER) {
                    turnIntoEnergy(p[i]);
                    fx->applyParticleArray(setA);
                } else {
                    particleBuffert.push(&p[i]);
                }
            }
            if (p[i].particleType == Cell) {
                if (p[i].toGrow &&
                    p[i].origin != NULL &&
                    p[i].origin->particleType == Cell &&
                    p[i].origin->organism == p[i].organism &&
                    !particleBuffert.empty()
                    ) {
                        Particle *parent = &p[i];
                        Particle *child  = particleBuffert.front();
                        particleBuffert.pop();
                        p[i].toGrow = false;
                        growCell(parent, child);
                        fx->applyParticleArray(setA);
                }
                // Create offspring:
                if (p[i].type == Egg && p[i].reproduce) {
                    initializeOffspring(&p[i]);
                    fx->applyParticleArray(setA);
                }
            }
        }

        if (step % 10 == 0) {
            printf("nCells: %i\t", g.nCells);
            printf("step %d\n", step);
            fx->outputFrame("output");
        }
    }

    delete fx;
}
