#include "hip/hip_runtime.h"
#include "fluidix.h"
#include "../lib/genome.h"
#include <map>

#define W 100 // simulation size
#define SUNLIGHT_STRENGTH 0.1f
#define MAX_CELL_ENERGY 5.0f
#define DT 0.01f
#define RANGE 3.0f

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 40

#define FLUID_DENSITY 1.0f
#define G -9.81f

#define SPRING_K 100.0f // spring constant
#define SPRING_L 1.0f // natural length

using namespace std;

__host__ __device__ bool operator==(xyz a, xyz b) {
	return (
		a.x == b.x && 
		a.y == b.y &&
		a.z == b.z
	);
}

enum CellType {Blank, Sense, Move, Ballast};

struct Global {
} g;

struct Particle {
	xyz r, v, f;
	float energy;
	float color;
	float alpha;
	float radius;
	float density;
	bool remove;
	bool divide;
	int linkSet;
	int origin;
	CellType type;
	float signal;
};


FUNC_EACH(init,
	p.r = make_xyz_uniform() * W;
	p.energy = 100.0f;
	p.alpha  = 1.0f;
	p.density = rnd_normal() + FLUID_DENSITY;
	p.radius = 1.0f;
	p.remove = false;
	p.divide = false;
	p.linkSet = -1;
	p.origin = p_index;
	p.type = Blank;
)

// pair interaction
FUNC_PAIR(particlePair,
	float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
	//float ratio = dr/range;
	float attraction = 0.0f;
	if(p1.linkSet == p2.linkSet)
		attraction = ATTRACTION_FORCE * ratio;
	xyz f = u * (REPULSION_FORCE * (1-ratio) - attraction);
	addVector(p1.f, f);
	addVector(p2.f, -f);

	if(p1.type == Sense) p1.signal += 0.1f;
	if(p2.type == Sense) p2.signal += 0.1f;

	if(p1.linkSet == p1.linkSet){
		float meanSignal = (p1.signal + p2.signal)/2;
		meanSignal = sin(meanSignal);
		p1.signal = p2.signal = meanSignal;
	}
	//p1.color = p1.signal;
	//p2.color = p2.signal;
)

// buoyancy 
FUNC_EACH(buoyancy,
	if(p.type == Ballast)
		p.density += sin(p.signal);
	float volume = p.radius * p.radius * PI;
	p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

FUNC_EACH(integrate,
	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
)

FUNC_LINK(moveCells,
	if(p1.type == Move) addVector(p1.f, u * p1.signal * 10.0f);
	if(p2.type == Move) addVector(p2.f, u * p2.signal * 10.0f);

	dr -= SPRING_L;
	xyz f = -u * (dr * SPRING_K);
	addVector(p1.f, f);
	addVector(p2.f, -f);
)

FUNC_EACH(photosynthesis,
	p.energy += SUNLIGHT_STRENGTH;
)

FUNC_EACH(friction,
	p.v *= 0.9f;
)

FUNC_EACH(checkDivide,
	if (p.energy > MAX_CELL_ENERGY && rnd_uniform() < 0.1f)
		p.divide = true;
	else if(p.energy <= 0)
		p.remove = true;	
)

FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
	if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }
)

int main(int argc, char **argv) {
	Fluidix<> *fx = new Fluidix<>(&g);

	map<int, Genome> genomes;
	vector<int> linkSets;

	int A = fx->createParticleSet(10);
	fx->runEach(init(), A);
	
	Particle *pArray = fx->getParticleArray(A);
	for (int step = 0; step < 5000; step++) {
		
		Particle *p = fx->getParticleArray(A);
		int n = fx->getParticleCount(A);
		for (int i = 0; i < n; i++) {
			if (p[i].remove == true)
			{
				//printf("Removing particle (%d)\n", i);
/*				memcpy(&p[i], &p[n-1], sizeof(Particle)); 	// replace current with last
				fx->applyParticleArray(A); 					// apply changes before any operation
				fx->resizeParticleSet(A, --n); 				// delete the last particle and decrease n
				Particle *p = fx->getParticleArray(A);
*/			}
			else if (p[i].divide == true)
			{
				//printf("Dividing particle (%d)\n", i);
				
				// If this is the origin cell
				if(p[i].linkSet == -1) {
					p[i].linkSet = fx->createLinkSet();
					linkSets.push_back(p[i].linkSet);

					int inputs = 4;
					int outputs = 4;

					Genome g(inputs, outputs);

					g.mutate();
					g.mutate();
					g.mutate();
					g.mutate();

					g.printMathematica();

					genomes.insert({p[i].linkSet,	g});

					vector<float> input(inputs, 0.0f);
					vector<float> output = g.getOutput(input);
					
					float max = output[0];
					for(int j=1; j<output.size(); j++) {
						if(output[j] > max) {
							max = output[j];
							p[i].type = (CellType) j;
						}
					}
					switch(p[i].type) {
						case Blank:		p[i].color = 0.6f; break;
						case Sense:		p[i].color = 0.3f; break;
						case Move:		p[i].color = 1.0f; break;
						case Ballast:	p[i].color = 0.0f; break;
					}

				}

				p[i].divide = false;
				p[i].energy -= 0.1f;
				p[i].energy /= 2;
				fx->applyParticleArray(A); 
				fx->resizeParticleSet(A, ++n); 				// add a new particle and increase n
				Particle *p = fx->getParticleArray(A);

				memcpy(&p[n-1], &p[i], sizeof(Particle)); 	// copy current to last

				// Displace particles from each other
				xyz dr = xyz_norm(make_xyz_uniform()) * p[i].radius;		
				p[i].r 	-= dr;
				p[n-1].r 	+= dr;

				//Create link between new particle and origin
				//fx->addLink(p[n-1].linkSet, A, p[n-1].origin, A, n-1);
				//printf("parent link set: %i\tchild link set: %i\t",p[i].linkSet, p[n-1].linkSet);
				fx->addLink(p[n-1].linkSet, A, i, A, n-1);

				Genome genome = genomes.at(p[n-1].linkSet);

				dr = p[n-1].r - p[p[n-1].origin].r;

				vector<float> input;
				input.push_back(dr.x);
				input.push_back(dr.y);
				input.push_back(dr.z);
				input.push_back(xyz_len(dr));

				vector<float> output = genome.getOutput(input);

				//printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
				//printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

				float max = output[0];
				for(int j=1; j<output.size(); j++) {
					if(output[j] > max) {
						max = output[j];
						p[n-1].type = (CellType) j;
					}
				}
				switch(p[n-1].type) {
					case Blank:		p[n-1].color = 0.6f; break;
					case Sense:		p[n-1].color = 0.3f; break;
					case Move:		p[n-1].color = 1.0f; break;
					case Ballast:	p[n-1].color = 0.0f; break;
				}
	
				fx->applyParticleArray(A); // apply changes before any operation
			}
		}
		printf("number of particles: %d\n", n);

		fx->runPair(particlePair(), A, A, RANGE);
		fx->runEach(integrate(), A);
		fx->runEach(buoyancy(), A);
		for(int l : linkSets) fx->runLink(moveCells(), l);
		//fx->runEach(photosynthesis(), A);	
		fx->runEach(friction(), A);
		fx->runEach(checkDivide(), A);
		fx->runEach(boundary(), A);

		if (step % 1 == 0) {
			//printf("step %d\n", step);
			fx->outputFrame("output");
		}
	}

	delete fx;
}

