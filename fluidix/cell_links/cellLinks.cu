#include "hip/hip_runtime.h"
//#include "fluidix.h"
#define PARTICLE_BITS 28
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/fluidix_extended.h"
#include "../lib/fluidix_int_all_pairs.h"
#include "../lib/genome.h"
#include <queue>
#include <ppl.h>
#include <concurrent_queue.h>

#define DT 0.01f // integration time-step
#define CELL_INITIAL_ENERGY 10.0f
#define PELLET_LIFETIME 5.0f

#define W make_int3(400, 200, 400)
#define N 100000
#define N_ORIGIN_CELLS 20
#define N_INITIAL_BUFFER 100000
#define N_STEPS 1000000

//Inputs x,y,z,d:
#define N_INPUTS 4

#define RANGE 3.0f
#define MOVE_FACTOR 50

#define REPULSION_FORCE 50

#define CELL_MIN_ENERGY 1.5f
#define PELLET_MIN_ENERGY 0.01f
#define DIVISION_ENERGY (CELL_MIN_ENERGY * 2.5f)
#define ENERGY_PARTICLE_ENERGY 0.5f
#define MAX_CELL_DIVISIONS 100

#define CELL_METABOLISM 0.05f
#define CELL_DECAY_RATE 0.05f

#define FLUID_DENSITY 1.0f
#define G -9.81f

#define GREEN   0.5f
#define RED     1.0f
#define YELLOW  0.7f
#define BLUE    0.0f
#define CYAN    0.3f
#define ORANGE  0.8f

#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;

enum CellType {
    Photo, Digest, Sting, Vascular, Fat, Sense, Motor, Ballast, Egg, 
    N_CELL_TYPES
};
enum ParticleType {
    Cell, Energy, Pellet, Buffer,
    N_PARTICLE_TYPES
};

int currGenomeIndex;

struct Global {
    int nCells;
} g;

struct Particle {
    ParticleType particleType;
    //float growthProb;
    xyz r, v, f;
    float color;
    float radius;
    float alpha;
    float density;
    float energy;
    float energyIn;
    float energyOut;
    float maxEnergy;
    int nDivisions;
    float signal;
    int organism;
    bool toGrow;
    bool toBuffer;
    bool toReproduce;
    Particle *origin;
   // Particle *parent;
    int links[6];
    CellType type;
};

#define turnIntoPellet(p) {         \
    p.particleType = Pellet;        \
    p.energy = PELLET_LIFETIME;     \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.organism = -1;                \
}                                   \

#define turnIntoEnergy(p) {         \
    p.particleType = Energy;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W.x,          \
        rnd_uniform()*W.y + W.y,    \
        rnd_uniform()*W.z           \
    );                              \
    p.color = 0.7f;                 \
    p.energy = ENERGY_PARTICLE_ENERGY; \
    p.alpha = 0.3f;                 \
    p.radius = 0.5f;                \
    p.density = 10.0f; 	  				\
}

#define turnIntoBuffer(p) {         \
    p.particleType = Buffer;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W.x,          \
        rnd_uniform()*W.y - W.y,    \
        rnd_uniform()*W.z           \
    );                              \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.color = 0.5f;                 \
    p.radius = 1.0f;                \
    p.organism = -1;                \
}

FUNC_EACH(init,
    turnIntoEnergy(p);
)

FUNC_EACH(integrate,
    p.v += p.f * DT;
    p.r += p.v * DT;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
)

FUNC_EACH(handleEnergy,
    switch (p.particleType) {
    case Cell:
        p.energy -= CELL_METABOLISM * DT;
        if (p.energy < CELL_MIN_ENERGY || p.origin->particleType != Cell)
            turnIntoPellet(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Pellet:
        p.energy -= CELL_DECAY_RATE * DT;
        if (p.energy <= PELLET_MIN_ENERGY)
            p.toBuffer = true;
        break;
    }
    //p.color = mapf(p.energy, 0.0f, 10.0f, 0.0f, 1.0f);
)

FUNC_EACH(buoyancy,
    if (p.particleType == Cell && p.type == Ballast)
        p.density = clamp(p.density + p.signal, 0.5f, 2.0f);
    float volume = p.radius * p.radius * PI;
    p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

FUNC_EACH(moveParticle,
    if (p.particleType == Cell && p.type == Motor) {
        xyz f = xyz_norm(p.origin->r - p.r) * MOVE_FACTOR;
        addVector(p.f, f);
    }
)

FUNC_EACH(reproduction,
    if (p.particleType == Cell && p.type == Egg)
        if (p.energy >= p.maxEnergy)
            p.toReproduce = true;
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        //printf("Weird particle!!! type (%i:%i)\n", p.particleType, p.type);
        turnIntoBuffer(p);
        p.toBuffer = true;
    } if (p.particleType != Buffer) {
        if (p.r.x < 0)   { p.v.x = 0.9f * (0 - p.r.x)   / DT; p.r.x = 0; }
        if (p.r.x > W.x) { p.v.x = 0.9f * (W.x - p.r.x) / DT; p.r.x = W.x; }
        if (p.r.z < 0)   { p.v.z = 0.9f * (0 - p.r.z)   / DT; p.r.z = 0; }
        if (p.r.z > W.z) { p.v.z = 0.9f * (W.z - p.r.z) / DT; p.r.z = W.z; }

        if (p.particleType == Energy){
            if (p.r.y < 0) {
                p.toBuffer = true;
            }
        }
        else {
            if (p.r.y < 0)   { p.v.y = 0.9f * (0 - p.r.y)   / DT; p.r.y = 0; }
            if (p.r.y > W.y) { p.v.y = 0.9f * (W.y - p.r.y) / DT; p.r.y = W.y; }
        }
    }
)

FUNC_EACH(growth,
    if (p.particleType == Cell)
        addInteger(g.nCells, 1);

    if (p.particleType == Cell && p.energy > DIVISION_ENERGY && p.nDivisions--) //rnd_uniform() < p.growthProb)
        p.toGrow = true;
)

FUNC_ALL_PAIRS(linkCells,
    printf("h");
)

inline __host__ __device__ void syncCuda() {
#ifdef __CUDA_ARCH__
    __syncthreads();
#endif
}


/*
#define SPRING_K 1.0f // spring constant
FUNC_EACH(springToParent,
    if (p.particleType == Cell && p.parent != nullptr &&
        p.parent->particleType == Cell &&
        !isWeirdParticlePointer(p.parent)
        ) {
        xyz parentPos = p.parent->r;
        float parentRad = p.parent->radius;
        syncCuda();
        xyz u = (p.r - parentPos);
        float dr = xyz_len(u);
        if (dr > RANGE) {
            turnIntoBuffer(p);
            p.toBuffer = true;
        } else {
            xyz f = u * ((dr - (p.radius + parentRad)) * SPRING_K);
            //printf("parent.r=(%.2f, %.2f, %.2f)\n", p.parent->r.x, p.parent->r.y, p.parent->r.z);
            addVector(p.f, f);
            //addVector(p.parent->f, -f);
        }
    }
)
*/

#define consumeParticle(a, b) {             \
    addFloat(a.energy, b.energy);    \
    b.energy = 0;                           \
    b.toBuffer = true;                      \
}

#define SPRING_K 100.0f // spring constant

FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
        //float ratio = dr/range;

        xyz f = u * (REPULSION_FORCE * (1 - ratio));
        if (p1.particleType == Cell &&
            p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                bool neighbours = false;
                for (int n = 0; n < 6; n++)
                if (p1.links[n] == p2_index || p2.links[n] == p1_index) {
                    neighbours = true;
                    break;
                }
                if (neighbours) {
                    //Spring force between neighbours
                    f = -u * ((dr + (p1.radius+p2.radius)) * SPRING_K);

                    //Signalling between cells of same organism
                    float meanSignal = (p1.signal + p2.signal) / 2;
                    p1.signal = p2.signal = meanSignal;

                    //Energy transmission
                    float p1Surplus = maxf(p1.energy - CELL_MIN_ENERGY, 0);
                    float p2Surplus = maxf(p2.energy - CELL_MIN_ENERGY, 0);
                    transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                    transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
                }
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting) turnIntoPellet(p2);
                if (p2.type == Sting) turnIntoPellet(p2);
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Pellet)
                ) consumeParticle(p1, p2)
        }
        //If p2 is a cell
        else if (p2.particleType == Cell) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Pellet)
                ) consumeParticle(p2, p1)
        }

        if (p1.particleType == Cell && p1.type == Sense) p1.signal += 0.1f;
        if (p2.particleType == Cell && p2.type == Sense) p2.signal += 0.1f;

        addVector(p1.f, f);
        addVector(p2.f, -f);

        //p1.color = p1.signal;
        //p2.color = p2.signal;
    }
)

void setDefaultCellValues(Particle *cell) {
    cell->alpha = 1.0f;
    cell->radius = 1.0f;
    cell->energy = CELL_INITIAL_ENERGY;
    cell->density = FLUID_DENSITY; // * 1.1f;
    cell->particleType = Cell;
}

void applyPhenotype(vector<float> output, Particle *cell) {
    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    switch (cell->type) {
    case Photo:
        cell->color = GREEN;
        cell->energyIn = 0.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 3.0f;
        break;
    case Digest:
        cell->color = RED;
        cell->energyIn = 0.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 3.0f;
        break;
    case Fat:
        cell->color = YELLOW;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 10.0f;
        break;
    case Motor:
        cell->color = 0.4f;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Sense:
        cell->color = BLUE;
        cell->energyIn = 0.5f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Ballast:
        cell->color = CYAN;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Egg:
        cell->color = ORANGE;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = CELL_INITIAL_ENERGY;
        break;
    case Vascular:
        cell->color = 0.2;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 1.0f;
        break;
    case Sting:
        cell->color = 0.85f;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
    }
    if (cell->origin == cell) {
        cell->energyIn = 1.0f;
        //cell->energyOut = 0.0f;
        //cell->maxEnergy = 3.0f;
    }
    cell->nDivisions = output[N_CELL_TYPES] * MAX_CELL_DIVISIONS;
}

#define idxFromCoord(x,y,z) cellBuff.at((x)*yMax*zMax + (y)*zMax + (z))

// Initialize new organism, not inheriting anything
void initializeNewOrganism(xyz origin, vector<int> cellBuff, Particle *p, Genome *genome) {
    int xMax = genome->getGridDim().x;
    int yMax = genome->getGridDim().y;
    int zMax = genome->getGridDim().z;

    int organism = currGenomeIndex++;

    for (int x = 0; x < xMax; x++)
    for (int y = 0; y < yMax; y++)
    for (int z = 0; z < zMax; z++) {
        Particle *cell = &p[idxFromCoord(x, y, z)];
        cell->organism = organism;
        cell->r = origin + make_xyz(x, y, z);
        cell->toReproduce = false;
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x,y,z)));

        vector<float> output = genome->getOutput(input);

        applyPhenotype(output, cell);

        cell->links[0] = x + 1 < xMax ? idxFromCoord(x + 1, y, z) : -1;
        cell->links[1] = y + 1 < yMax ? idxFromCoord(x, y + 1, z) : -1;
        cell->links[2] = z + 1 < zMax ? idxFromCoord(x, y, z + 1) : -1;
        cell->links[3] = x - 1 >= 0 ?   idxFromCoord(x - 1, y, z) : -1;
        cell->links[4] = y - 1 >= 0 ?   idxFromCoord(x, y - 1, z) : -1;
        cell->links[5] = z - 1 >= 0 ?   idxFromCoord(x, y, z - 1) : -1;
    }
}

// Initialize organism, inheriting from parent
void initializeOffspring(Particle *cell, Genome *genome) {
    cell->organism = currGenomeIndex++;
    cell->origin = cell;
//    cell->parent = nullptr;
    cell->toReproduce = false;

    // Define number of in- and outputs
    int inputs = N_INPUTS; // X, Y, Z, Dist

    vector<float> input(inputs, 0.0f); //Input origin
    vector<float> output = genome->getOutput(input);

    applyPhenotype(output, cell);

    printf("New organism! cell type = %i\n", cell->type);
}
/*
void growCell(Particle *parent, Particle *child, Genome *genomeParent, Genome *genomeChild) {
    normal_distribution<float> rndNormal(0.0f, 1.0f);
    
    //Half of parent's energy goes to the child
    parent->energy /= 2;

    //  Copy constructor
    *child = Particle(*parent);
    *genomeChild = Genome(*genomeParent);

    // Displace particles from each other
    xyz displacement = xyz_norm(
        make_xyz(
        rndNormal(rndGen),
        rndNormal(rndGen),
        rndNormal(rndGen)
        )) * parent->radius;

    parent->r -= displacement;
    child->r += displacement;

//    child->parent = parent;

    xyz dr = child->r - child->origin->r;

    vector<float> input;
    input.push_back(dr.x);
    input.push_back(dr.y);
    input.push_back(dr.z);
    input.push_back(xyz_len(dr));

    genomeChild->mutate();

    vector<float> output = genomeChild->getOutput(input);

    //printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
    //printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

    applyPhenotype(output, child);
}
*/

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

int main() {
    Fluidix<> *fx = new Fluidix<>(&g);
    int setA = fx->createParticleSet(N);

/*    fx->createGlobalArray(&g.link, N);
    fx->getGlobalArray(&g.link);
    for (int i = 0; i < N; i++) {
        g.link[i] = nullptr;
    }
    fx->applyGlobalArray(&g.link);

    for (int i = 0; i < 100; i++){
        printf(g.link[i] == nullptr? "nullptr\n" : " ??? ");
    }
*/
    currGenomeIndex = 0;
    g.nCells = 0;
    Genome *genomes = new Genome[N];
    
    int linkSet = fx->createLinkSet();

    fx->runEach(init(), setA);
    concurrent_queue<int> particleBuffer;
    Particle *p = fx->getParticleArray(setA);

    for (int i = 0; i < N_INITIAL_BUFFER; i++) {
        turnIntoBuffer(p[i]);
        p[i].r.y -= W.y;
        particleBuffer.push(i);
    }

    for (int i = 0; i < N_ORIGIN_CELLS; i++) {
        int3 d = make_int3(10,10,10); //genomes[iOrigin].gridDim;

        // Define number of in- and outputs
        int inputs = N_INPUTS;              // X, Y, Z, Dist
        int nonCelltypeOutputs = 1;         // Growth prob
        int outputs = N_CELL_TYPES + nonCelltypeOutputs;
        Genome g = Genome(inputs, outputs, d);
        g.mutate(); g.mutate(); g.mutate(); g.mutate(); g.mutate();

        int neededParticles = g.getGridDim().x * g.getGridDim().y * g.getGridDim().z;

        if (neededParticles > particleBuffer.unsafe_size()) {
            printf("Not enought particles in buffer\n");
            break;
        }

        vector<int> particles;
        while (neededParticles) {
            int particle;
            if (particleBuffer.try_pop(particle)) {
                particles.push_back(particle);
                neededParticles--;
            }
        }
        xyz origin = make_xyz_uniform() * int3_to_xyz(W);
       
        initializeNewOrganism(origin, particles, p, &g);
    }
    fx->applyParticleArray(setA);

    for (int step = 0; step < N_STEPS; step++) {
        g.nCells = 0;
        fx->runEach(boundary(), setA);
        fx->runPair(particlePair(), setA, setA, RANGE);
  //      fx->runEach(moveParticle(), setA);
        fx->runEach(buoyancy(), setA);
  //      fx->runEach(handleEnergy(), setA);
  //      fx->runEach(growth(), setA);
  //      fx->runEach(reproduction(), setA);
        fx->runEach(integrate(), setA);
        //fx->runAllPairs(linkCells(), setA);
        //parallel_for (int(0), N, [&](int i)
/*        for (int i = 0; i<N; i++)
        {
            if (p[i].toBuffer) {
                if (particleBuffer.unsafe_size() > N_INITIAL_BUFFER) {
                    turnIntoEnergy(p[i]);                  
                } else {
                    turnIntoBuffer(p[i]);
                    particleBuffer.push(i);
                }
                p[i].toBuffer = false;
                fx->applyParticleArray(setA);
            }
            if (p[i].particleType == Cell) {
                // Create offspring:
                if (p[i].type == Egg && p[i].toReproduce) {
                    initializeOffspring(&p[i], &genomes[i]);
                    fx->applyParticleArray(setA);
                }
                // Cell division
                if (p[i].toGrow &&
                    p[i].origin != nullptr &&
                    p[i].origin->particleType == Cell &&
                    p[i].origin->organism == p[i].organism &&
                    !particleBuffer.empty()
                    ) {
                        int parent = i;
                        int child;
                        if (particleBuffer.try_pop(child)) {
                            p[i].toGrow = false;
                            growCell(&p[parent], &p[child], &genomes[parent], &genomes[child]);
                            fx->applyParticleArray(setA);
                        }
                }
            }
        } //);
        */
//        fx->runEach(springToParent(), setA);

        if (step % 1 == 0) {
            printf("nCells: %i\t", g.nCells);
            printf("currgenomeIndex: %i\t", currGenomeIndex);
            printf("step %d\n", step);
            fx->outputFrame("output");
        }

//        if (!g.nCells) {
            /*printf("\nrebooting...\n");
            for (int i = 0; i < N_INITIAL_BUFFER; i++) {
                turnIntoBuffer(p[i]);
                p[i].r.y -= W;
                particleBuffer.push(i);
            }
            initializeNewOrganism(&p[N_INITIAL_BUFFER], &genomes[N_INITIAL_BUFFER]);*/
//            break;
//        }
    }
    delete[] genomes;
    delete fx;

    //system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
}
