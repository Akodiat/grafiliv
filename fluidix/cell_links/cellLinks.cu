#include "hip/hip_runtime.h"
//#include "fluidix.h"
#define PARTICLE_BITS 28
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/fluidix_extended.h"
#include "../lib/fluidix_int_all_pairs.h"
#include "../lib/genome.h"
#include <queue>
#include <ppl.h>
#include <concurrent_queue.h>

#define DT 0.01f // integration time-step
#define PELLET_LIFETIME 5.0f

#define W make_int3(200, 100, 200)
#define N 100000
#define N_ORIGIN_CELLS 1000
#define N_INITIAL_BUFFER 50000
#define N_STEPS 1000000
#define INITIAL_ORGANISM_DIMENSIONS make_int3(1, 1, 1)

//Inputs x,y,z,d:
#define N_INPUTS 4

#define RANGE 10.0f
#define MOVE_FACTOR 10

#define REPULSION_FORCE 300
#define SPRING_K 150.0f // spring constant
#define WALL 100.0f // repulsive wall force

#define CELL_INITIAL_ENERGY 2.0f
#define CELL_MIN_ENERGY 1.0f
#define PELLET_MIN_ENERGY 0.01f
#define DIVISION_ENERGY (CELL_MIN_ENERGY * 2.5f)
#define ENERGY_PARTICLE_ENERGY 0.5f
#define MAX_CELL_DIVISIONS 100
#define CELL_EXISTENCE_THRESHOLD 0.0f

#define CELL_METABOLISM 0.02f
#define CELL_DECAY_RATE 0.01f

#define FLUID_DENSITY 1.0f
#define G -9.81f

#define GREEN   0.5f
#define RED     1.0f
#define YELLOW  0.7f
#define BLUE    0.0f
#define CYAN    0.3f
#define ORANGE  0.8f

#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;

enum CellType {
    Photo, Digest, Sting, Vascular, Fat, Sense, Motor, Ballast, Egg, 
    N_CELL_TYPES
};
enum ParticleType {
    Cell, Energy, Pellet, Buffer,
    N_PARTICLE_TYPES
};
enum Neigbour { Front, Right, Up, Back, Left, Down };

int currGenomeIndex;

struct Global {
    int nCells;
} g;

struct Particle {
    ParticleType particleType;
    //float growthProb;
    xyz r, v, f;
    float color;
    float radius;
    float alpha;
    float density;
    float energy;
    float energyIn;
    float energyOut;
    float maxEnergy;
    int nDivisions;
    float signal;
    int organism;
    Neigbour toGrow;
    bool toBuffer;
    bool toReproduce;
    Particle *origin;
   // Particle *parent;
    int links[6];
    bool missingNeighbour[6];
    CellType type;
};

#define turnIntoPellet(p) {         \
    p.particleType = Pellet;        \
    p.energy = PELLET_LIFETIME;     \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.organism = -1;                \
}                                   \

#define turnIntoEnergy(p) {         \
    p.particleType = Energy;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W.x,          \
        rnd_uniform()*W.y + W.y,    \
        rnd_uniform()*W.z           \
    );                              \
    p.color = 0.7f;                 \
    p.energy = ENERGY_PARTICLE_ENERGY; \
    p.alpha = 0.3f;                 \
    p.radius = 0.5f;                \
    p.density = 10.0f; 	  				\
}

#define turnIntoBuffer(p) {         \
    p.particleType = Buffer;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W.x,          \
        rnd_uniform()*W.y - W.y,    \
        rnd_uniform()*W.z           \
    );                              \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.color = 0.5f;                 \
    p.radius = 1.0f;                \
    p.organism = -1;                \
}

FUNC_EACH(init,
    turnIntoEnergy(p);
)

FUNC_EACH(integrate,
    p.v += p.f * DT; //Mass?
    p.r += p.v * DT;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
)

FUNC_EACH(handleEnergy,
    switch (p.particleType) {
    case Cell:
        p.energy -= CELL_METABOLISM * DT;
        if (p.energy < CELL_MIN_ENERGY ) // || p.origin->particleType != Cell)
            turnIntoPellet(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Pellet:
        p.energy -= CELL_DECAY_RATE * DT;
        if (p.energy <= PELLET_MIN_ENERGY)
            p.toBuffer = true;
        break;
    }
    //p.color = mapf(p.energy, 0.0f, 3.0f, 0.0f, 1.0f);
)

FUNC_EACH(buoyancy,
    if (p.particleType == Cell && p.type == Ballast)
        p.density = clamp(p.density + p.signal, 0.5f, 2.0f);
    float volume = p.radius * p.radius * PI;
    p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

FUNC_EACH(moveParticle,
    if (p.particleType == Cell && p.type == Motor) {
        //xyz f = xyz_norm(p.origin->r - p.r) * MOVE_FACTOR;
        
        xyz f = make_xyz(0, 0, 0);
        if (p.links[Left]  < 0)  f.x -= 1;
        if (p.links[Up]    < 0)  f.y -= 1;
        if (p.links[Back]  < 0)  f.z -= 1;
        if (p.links[Right] < 0)  f.x += 1;
        if (p.links[Down]  < 0)  f.y += 1;
        if (p.links[Front] < 0)  f.z += 1;
        f = xyz_norm(f) * MOVE_FACTOR; //* p.signal;

        addVector(p.f, f);

        //p.radius = clamp(p.radius + sin(p.signal)*0.1f, 0.8f, 1.5f);
    }
)

FUNC_EACH(reproduction,
    if (p.particleType == Cell && p.type == Egg)
        if (p.energy >= p.maxEnergy){
            p.toReproduce = true;
            printf("toReproduce!!!\n");
        }
)

FUNC_EACH(resetMissingNeighbours,
    for (int i = 0; i < 6; i++)
        p.missingNeighbour[i] = true;
)

FUNC_EACH(handleMissingNeighbours,
    for (int i = 0; i < 6; i++)
    if (p.missingNeighbour[i] && p.links[i] != -1){
            p.toBuffer = true;
            break;
        }
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
    if (p.particleType == Cell)
    addInteger(g.nCells, 1);

    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        //printf("Weird particle!!! type (%i:%i)\n", p.particleType, p.type);
        turnIntoBuffer(p);
        p.toBuffer = true;
    } if (p.particleType != Buffer) {
        //if (p.r.x < 0)   /*p.r.x = W.x; */ { p.v.x = 0.9f * (0 - p.r.x)   / DT; p.r.x = 0; }
        //if (p.r.x > W.x) /*p.r.x = 0;   */ { p.v.x = 0.9f * (W.x - p.r.x) / DT; p.r.x = W.x; }
        //if (p.r.z < 0)   /*p.r.z = W.z; */ { p.v.z = 0.9f * (0 - p.r.z)   / DT; p.r.z = 0; }
        //if (p.r.z > W.z) /*p.r.z = 0;   */ { p.v.z = 0.9f * (W.z - p.r.z) / DT; p.r.z = W.z; }
        if (p.r.x < 0)   p.f.x += WALL * (0 - p.r.x);
        if (p.r.x > W.x) p.f.x += WALL * (W.x - p.r.x);
        if (p.r.z < 0)   p.f.z += WALL * (0 - p.r.z);
        if (p.r.z > W.x) p.f.z += WALL * (W.x - p.r.z);

        if (p.particleType == Energy){
            if (p.r.y < 0) {
                p.toBuffer = true;
            }
        }
        else {
            if (p.r.y < 0)   { p.v.y = 0.9f * (0 - p.r.y)   / DT; p.r.y = 0; }
            if (p.r.y > W.y) { p.v.y = 0.9f * (W.y - p.r.y) / DT; p.r.y = W.y; }
        }
    }
    else if (p.r.y < -2 * W.y) p.r.y += W.y;
)

#define consumeParticle(a, b) {             \
    addFloat(a.energy, b.energy);    \
    b.energy = 0;                           \
    b.toBuffer = true;                      \
}

FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        //float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
        //float ratio = dr/range;

        //xyz f = u * (REPULSION_FORCE * (1 - ratio));
        xyz f = u * maxf(
            (REPULSION_FORCE * (1 - dr / (p1.radius + p2.radius))),
            0
        );
        if (p1.particleType == Cell &&
            p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                bool neighbours = false;
                for (int n = 0; n < 6; n++) {
                    if (p1.links[n] == p2_index) {
                        p1.missingNeighbour[n] = false;
                        neighbours = true;
                    }
                    if (p2.links[n] == p1_index) {
                        p1.missingNeighbour[n] = false;
                        neighbours = true;
                    }
                }
                if (neighbours) {
                    //Spring force between neighbours
                    f = -u * ((dr - (p1.radius + p2.radius)/2) * SPRING_K);

                    //Signalling between cells of same organism
                    float meanSignal = (p1.signal + p2.signal) / 2;
                    p1.signal = p2.signal = meanSignal;

                    //Energy transmission
                    float p1Surplus = maxf(p1.energy - CELL_MIN_ENERGY, 0);
                    float p2Surplus = maxf(p2.energy - CELL_MIN_ENERGY, 0);
                    transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                    transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
                }
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting) {
                    turnIntoPellet(p2);
                }
                if (p2.type == Sting) {
                    turnIntoPellet(p2);
                }
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Pellet)
                ) consumeParticle(p1, p2)
        }
        //If p2 is a cell
        else if (p2.particleType == Cell) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Pellet)
                ) consumeParticle(p2, p1)
        }

        if (p1.particleType == Cell && p1.type == Sense) p1.signal += 0.1f;
        if (p2.particleType == Cell && p2.type == Sense) p2.signal += 0.1f;

        addVector(p1.f, f);
        addVector(p2.f, -f);

        //p1.color = p1.signal;
        //p2.color = p2.signal;
    }
)

void setDefaultCellValues(Particle *cell) {
    cell->alpha = 1.0f;
    cell->radius = 1.0f;
    cell->energy = CELL_INITIAL_ENERGY;
    cell->density = FLUID_DENSITY * 1.10f;
    cell->particleType = Cell;
}

bool applyPhenotype(vector<float> output, Particle *cell) {
    // If cell should not exist, return
    if (output[N_CELL_TYPES + 1] < CELL_EXISTENCE_THRESHOLD)
        return false;

    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    switch (cell->type) {
    case Photo:
        cell->color = GREEN;
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 3.0f;
        break;
    case Digest:
        cell->color = RED;
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 3.0f;
        break;
    case Fat:
        cell->color = YELLOW;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 10.0f;
        break;
    case Motor:
        cell->color = 0.4f;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Sense:
        cell->color = BLUE;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Ballast:
        cell->color = CYAN;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Egg:
        cell->color = ORANGE;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 1000.0f;
        break;
    case Vascular:
        cell->color = 0.2;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 1.0f;
        break;
    case Sting:
        cell->color = nanf("");
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
    }
    if (cell->origin == cell) {
        cell->energyIn = 1.0f;
        //cell->energyOut = 0.0f;
        //cell->maxEnergy = 3.0f;
    }
    cell->nDivisions = output[N_CELL_TYPES] * MAX_CELL_DIVISIONS;
    return true;
}

void disconnectCell(Particle *p, int cell, int code) {
    for (int i = 0; i < 6; i++) {
        if (p[cell].links[i] >= 0)
            p[p[cell].links[i]].links[(i + 3) % 6] = code;
    }
    turnIntoBuffer(p[cell]);
}
void emptyCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -1);
}
void deadCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -2);
}

int getIdxFromCoord(int x, int y, int z, int3 br) {
    x += br.x; y += br.y; z += br.z;
    //int lX = 2 * br.z + 1;
    int lY = 2 * br.y + 1;
    int lZ = 2 * br.z + 1;
    return x*lY*lZ + y*lZ + z;
}
#define iFromCoord(x,y,z) cellBuff.at(getIdxFromCoord(x,y,z,br))

// Initialize new organism, not inheriting anything
bool spawnOrganism(xyz origin, concurrent_queue<int> *particleBuffer, Particle *p, Genome *genome, Genome *allGenomes) {
    int nParticlesNeeded = genome->getMaxCellsReq();
    if (nParticlesNeeded > particleBuffer->unsafe_size()) {
        printf("Not enought particles in buffer\n");
        return false;
    }
    vector<int> cellBuff;
    while (nParticlesNeeded) {
        int particle;
        if (particleBuffer->try_pop(particle)) {
            cellBuff.push_back(particle);
            nParticlesNeeded--;
        }
    }

    int3 br = genome->getBoundingRadius();

    int organism = currGenomeIndex++;
    vector<int> removedCells;

    for (int x = -br.x; x <= br.x; x++)
    for (int y = -br.y; y <= br.y; y++)
    for (int z = -br.z; z <= br.z; z++) {
        Particle *cell = &p[iFromCoord(x, y, z)];
        cell->organism = organism;
        cell->r = origin + make_xyz(x, y, z);
        cell->toReproduce = false;
        cell->energy = CELL_INITIAL_ENERGY;
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x,y,z)));

        vector<float> output = genome->getOutput(input);

        if (applyPhenotype(output, cell)) {
            Genome gNew = Genome(*genome);
            gNew.mutate();
            allGenomes[iFromCoord(x, y, z)] = gNew;
            cell->links[Left] = x + 1 < br.x ? iFromCoord(x + 1, y, z) : -1;
            cell->links[Up] = y + 1 < br.y ? iFromCoord(x, y + 1, z) : -1;
            cell->links[Back] = z + 1 < br.z ? iFromCoord(x, y, z + 1) : -1;
            cell->links[Right] = x - 1 >= 0 ? iFromCoord(x - 1, y, z) : -1;
            cell->links[Down] = y - 1 >= 0 ? iFromCoord(x, y - 1, z) : -1;
            cell->links[Front] = z - 1 >= 0 ? iFromCoord(x, y, z - 1) : -1;
        }
        else
            removedCells.push_back(iFromCoord(x, y, z));
    }
    for (int i : removedCells)
        emptyCellPos(p, i);
    return true;
}


// Initialize organism, inheriting from parent
void initializeOffspring(Particle *cell, Genome *genome) {
    cell->organism = currGenomeIndex++;
    cell->origin = cell;
//    cell->parent = nullptr;
    cell->toReproduce = false;

    // Define number of in- and outputs
    int inputs = N_INPUTS; // X, Y, Z, Dist

    vector<float> input(inputs, 0.0f); //Input origin
    vector<float> output = genome->getOutput(input);

    applyPhenotype(output, cell);

    printf("New organism! cell type = %i\n", cell->type);
}
/*
void growCell(Particle *parent, Particle *child, Genome *genomeParent, Genome *genomeChild) {
    normal_distribution<float> rndNormal(0.0f, 1.0f);
    
    //Half of parent's energy goes to the child
    parent->energy /= 2;

    //  Copy constructor
    *child = Particle(*parent);
    *genomeChild = Genome(*genomeParent);

    // Displace particles from each other
    xyz displacement = xyz_norm(
        make_xyz(
        rndNormal(rndGen),
        rndNormal(rndGen),
        rndNormal(rndGen)
        )) * parent->radius;

    parent->r -= displacement;
    child->r += displacement;

//    child->parent = parent;

    xyz dr = child->r - child->origin->r;

    vector<float> input;
    input.push_back(dr.x);
    input.push_back(dr.y);
    input.push_back(dr.z);
    input.push_back(xyz_len(dr));

    genomeChild->mutate();

    vector<float> output = genomeChild->getOutput(input);

    //printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
    //printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

    applyPhenotype(output, child);
}
*/

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

void generateTerrain(Fluidix<> *fx){
    normal_distribution<float> rndNormal(0.0f, W.y * 0.1);

    int meshParticles   = fx->createParticleSet(W.x * W.z);
    int meshLinks       = fx->createLinkSet();
    Particle *mesh      = fx->getParticleArray(meshParticles);

    for (int x = 0; x < W.x; x++)
    for (int z = 0; z < W.z; z++){
        int i = x*W.z + z;
        float y = rndNormal(rndGen);
        mesh[i].r = make_xyz(x, y, z);

        int n = (x+1)*W.z + z;
        int s = (x-1)*W.z + z;
        int e = x*W.z + (z+1);
        int w = x*W.z + (z-1);
        fx->addLink(meshLinks, meshParticles, i, meshParticles, n);
        fx->addLink(meshLinks, meshParticles, i, meshParticles, s);
        fx->addLink(meshLinks, meshParticles, i, meshParticles, e);
        fx->addLink(meshLinks, meshParticles, i, meshParticles, w);
    }
    //fx->initializeMesh(meshLinks);
}

int main() {
    Fluidix<> *fx = new Fluidix<>(&g);
    int setA = fx->createParticleSet(N);

    //generateTerrain(fx);

/*    fx->createGlobalArray(&g.link, N);
    fx->getGlobalArray(&g.link);
    for (int i = 0; i < N; i++) {
        g.link[i] = nullptr;
    }
    fx->applyGlobalArray(&g.link);

    for (int i = 0; i < 100; i++){
        printf(g.link[i] == nullptr? "nullptr\n" : " ??? ");
    }
*/
    currGenomeIndex = 0;
    g.nCells = 0;
    Genome *genomes = new Genome[N];
    
    int linkSet = fx->createLinkSet();

    fx->runEach(init(), setA);
    concurrent_queue<int> particleBuffer;
    Particle *p = fx->getParticleArray(setA);

    for (int i = 0; i < N_INITIAL_BUFFER; i++) {
        turnIntoBuffer(p[i]);
        p[i].r.y -= W.y;
        particleBuffer.push(i);
    }

    for (int i = 0; i < N_ORIGIN_CELLS; i++) {
        int3 gridDim = INITIAL_ORGANISM_DIMENSIONS; //genomes[iOrigin].gridDim;

        // Define number of in- and outputs
        int inputs = N_INPUTS;              // X, Y, Z, Dist
        int nonCelltypeOutputs = 2;         // Growth prob
        int outputs = N_CELL_TYPES + nonCelltypeOutputs;
        Genome g = Genome(inputs, outputs, gridDim);
        g.mutate(); g.mutate(); g.mutate(); g.mutate(); g.mutate();
        xyz origin = make_xyz_uniform() * int3_to_xyz(W);

        spawnOrganism(origin, &particleBuffer, p, &g, genomes);
    }
    fx->applyParticleArray(setA);

    for (int step = 0; step < N_STEPS; step++) {
        g.nCells = 0;
        //fx->runEach(resetMissingNeighbours(), setA);
        fx->runEach(boundary(), setA);
        fx->runPair(particlePair(), setA, setA, RANGE);
        fx->runEach(moveParticle(), setA);
        fx->runEach(buoyancy(), setA);
        fx->runEach(handleEnergy(), setA);
        //fx->runEach(reproduction(), setA);
        fx->runEach(integrate(), setA);
        //fx->runEach(handleMissingNeighbours(), setA);
        //parallel_for (int(0), N, [&](int i)
        for (int i = 0; i<N; i++)
        {
            if (p[i].toBuffer) {
                if (particleBuffer.unsafe_size() > N_INITIAL_BUFFER) {
                    turnIntoEnergy(p[i]);                  
                } else {
                    turnIntoBuffer(p[i]);
                    particleBuffer.push(i);
                }
                p[i].toBuffer = false;
                fx->applyParticleArray(setA);
            }
          if (p[i].particleType == Cell) {
                // Create offspring:
                if (p[i].type == Egg) { //&& p[i].toReproduce) {
                    //printf("Has %.2f, needs %.2f - ", p[i].energy, genomes[i].getMaxCellsReq() * CELL_INITIAL_ENERGY);
                    if (genomes[i].getMaxCellsReq() * CELL_INITIAL_ENERGY <= p[i].energy) {
                        printf("Reproducing!\n");
                        Genome g = genomes[i];
                        g.mutate();
                        spawnOrganism(p[i].r, &particleBuffer, p, &g, genomes);
                        p[i].toReproduce = false;
                        p[i].toBuffer = true;
                        fx->applyParticleArray(setA);
                    }
                }
/*                // Cell division
                if (p[i].toGrow &&
                    p[i].origin != nullptr &&
                    p[i].origin->particleType == Cell &&
                    p[i].origin->organism == p[i].organism &&
                    !particleBuffer.empty()
                    ) {
                        int parent = i;
                        int child;
                        if (particleBuffer.try_pop(child)) {
                            p[i].toGrow = false;
                            growCell(&p[parent], &p[child], &genomes[parent], &genomes[child]);
                            fx->applyParticleArray(setA);
                        }
                } */
            }
        } //);

        //printf("\n");
        if (step % 10 == 0) {
            printf("nCells: %i\t", g.nCells);
            printf("currgenomeIndex: %i\t", currGenomeIndex);
            printf("step %d\n", step);
            fx->outputFrame("output");
        }

//        if (!g.nCells) {
//            break;
//        }
    }
    delete[] genomes;
    delete fx;

    system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
}
