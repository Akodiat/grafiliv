#include "hip/hip_runtime.h"
#include "fluidix.h"
#include "../lib/genome.h"
#include <map>

#define W 500 // world size (cube)
#define N 500 // number of particles
#define STEPS 100000 // number of simulation steps
#define DT 0.01f // integration time-step
#define RANGE 4.0f // fluid interaction cutoff range
#define HARDNESS 20.0f // repulsive force strength
#define FLUID_DENSITY 1.0f

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 40


#define PHOTOSYNTHESIS_MAX 1.0f
#define CELL_BASIC_METABOLISM 0.3f
#define CELL_BASIC_ENERGY 2.0f
#define CELL_MAX_ENERGY 1.0f
#define CELL_INITIAL_ENERGY 4000.0f

#define MUTATION_RATE 0.01f

#define G -9.81f

struct Particle {
	xyz r, v, f; 				// position, velocity, force
	xyz signal;
	xyz sense;
	float density;
	float radius;
	float energy;
	float color; 				// display color
	float alpha;
	bool remove;
	bool divide;
	int linkSet;
	int origin;

	//Genome:
	float phoSyAbility; 		// Ability to photosynthesize
	float moveAbility;			// Ability to move
	float senseAbility;			// Ability to sense surroundings
	bool  invertSignal;			// Inverts signal before acting or transmitting
	float recieveSigAbility;  	// Ability to recieve signals	
	float sendSigAbility;		// Ability to send signals
};

struct Node {
    int x;
    Node *next;
};

struct Global {
} g;

// initialize particles
FUNC_EACH(init,

	p.energy = CELL_INITIAL_ENERGY;
	p.r = make_xyz_uniform() * W; 					//Random position within box
	p.radius = 1.0f; //(p.energy + CELL_BASIC_ENERGY)/2;
	p.density = FLUID_DENSITY; // abs(rnd_normal() / 10 + FLUID_DENSITY); 	//Random density
	p.alpha = 1.0f;
	p.remove = false;	
	p.divide = false;
	
	p.invertSignal 		= rnd_uniform() < 0.5f;
	p.phoSyAbility 		= 0.0f; // abs(rnd_normal());
	p.moveAbility 		= 0.0f; // abs(rnd_normal());
	p.senseAbility 		= 0.0f; // abs(rnd_normal());
	p.recieveSigAbility = 0.0f; // abs(rnd_normal());
	p.sendSigAbility 	= 0.0f; // abs(rnd_normal());

	//p.color = p.moveAbility;
	
	p.sense  = make_xyz(0, 0, 0);
	p.signal = make_xyz_uniform()*5;	//Random initial signal
)

// linear repulsion + attraction at distance
FUNC_PAIR(particlePair,
	float ratio = dr / range;
	float attraction = 0.0f;
	if(p1.linkSet == p2.linkSet)
		attraction = ATTRACTION_FORCE * ratio;
	xyz f = u * (REPULSION_FORCE * (1-ratio) - attraction);
	addVector(p1.f, f);
	addVector(p2.f, -f);

	// Update sense values
	addVector(p1.sense, -u);	// These should be normalized...
	addVector(p2.sense, u);

	// Update signal values with sense
	addVector(p1.signal, p1.sense * p1.senseAbility);
	addVector(p2.signal, p2.sense * p2.senseAbility);

	// Transmit signal
	addVector(p1.signal, p2.signal * p1.recieveSigAbility * p2.sendSigAbility * (p1.invertSignal ? -1 : 1));
	addVector(p2.signal, p1.signal * p2.recieveSigAbility * p1.sendSigAbility * (p2.invertSignal ? -1 : 1));

	// p1 consumes p2
	if(p1.energy > 0 && p2.energy <= 0){
		p2.remove = true;
		addFloat(p1.energy, CELL_BASIC_ENERGY);
	}
	// p2 consumes p1
	else if (p2.energy > 0 && p1.energy <= 0) {
		p1.remove = true;
		addFloat(p2.energy, CELL_BASIC_ENERGY);
	}
/*
	else {
		//Steal energy from other particle proportional to signal alignment with particle direction
		addFloat(p1.energy, xyz_len(xyz_norm(p1.signal) + (-u)) - xyz_len(xyz_norm(p2.signal) + (u)));
		addFloat(p2.energy, xyz_len(xyz_norm(p2.signal) + (u)) - xyz_len(xyz_norm(p1.signal) + (-u)));
		//addFloat(p1.energy, p2.energy/2 - p1.energy/2);
		//addFloat(p2.energy, p1.energy/2 - p2.energy/2);
	}
*/

)

// buoyancy 
FUNC_EACH(buoyancy,
	float volume = p.radius * p.radius * PI;
	p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

// photosynthesis 
FUNC_EACH(photosynthesis,
	p.energy += PHOTOSYNTHESIS_MAX * p.phoSyAbility;
)

// move 
FUNC_EACH(move,
	xyz f = p.signal * p.moveAbility * 0.1f;
	addVector(p.f, f);
)

// handle energy usage
FUNC_EACH(handleEnergy,
	p.energy -= (
		CELL_BASIC_METABOLISM +
		p.moveAbility +
		p.senseAbility
	);

	// If dead
	if(p.energy <= 0)
	{
		p.alpha = 0.5f;
		p.density = FLUID_DENSITY;
		p.phoSyAbility = p.moveAbility = p.senseAbility = p.recieveSigAbility = p.sendSigAbility = 0.0f;
	}
	// If there is enought energy to divide
	if(p.energy > CELL_MAX_ENERGY){
		p.divide = true;
	}

	//p.radius = (p.energy + CELL_BASIC_ENERGY)/2;
)

// Euler integration
FUNC_EACH(integrate,
	//p.color = xyz_len(p.f) / 50.0f;

	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
	if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }
)

// deflate old signals so that they won't overflow 
FUNC_EACH(deflateSignals,
	//p.color = xyz_len(p.signal)/100;
	p.color = p.senseAbility;
	p.signal = xyz_norm(p.signal);
	p.sense = make_xyz(0, 0, 0);
)

int main(int argc, char **argv) {
	Fluidix<> *fx = new Fluidix<>(&g);

	map<int, Genome> genomes;

	int A = fx->createParticleSet(10);
	fx->runEach(init(), A);
	
	Particle *pArray = fx->getParticleArray(A);
	for (int step = 0; step < 5000; step++) {
		
		Particle *p = fx->getParticleArray(A);
		int n = fx->getParticleCount(A);
		for (int i = 0; i < n; i++) {
			if (p[i].remove == true)
			{
				//printf("Removing particle (%d)\n", i);
/*				memcpy(&p[i], &p[n-1], sizeof(Particle)); 	// replace current with last
				fx->applyParticleArray(A); 					// apply changes before any operation
				fx->resizeParticleSet(A, --n); 				// delete the last particle and decrease n
				Particle *p = fx->getParticleArray(A);
*/			}
			else if (p[i].divide == true)
			{
				//printf("Dividing particle (%d)\n", i);
				
				// If this is the origin cell
				if(p[i].linkSet == -1) {
					p[i].linkSet = fx->createLinkSet();

					int inputs = 4;
					int outputs = 3;

					Genome g(inputs, outputs);

					g.mutate();
					g.mutate();
					g.mutate();
					g.mutate();

					g.printMathematica();

					genomes.insert({p[i].linkSet,	g});

					vector<float> input(inputs, 0.0f);
					vector<float> output = g.getOutput(input);
	
					p[i].color = output[0];
					p[i].density = output[1];
					p[i].remove = output[2]<=0;
				}

				p[i].divide = false;
				p[i].energy -= 0.1f;
				p[i].energy /= 2;
				fx->applyParticleArray(A); 
				fx->resizeParticleSet(A, ++n); 				// add a new particle and increase n
				Particle *p = fx->getParticleArray(A);

				memcpy(&p[n-1], &p[i], sizeof(Particle)); 	// copy current to last

				// Displace particles from each other
				xyz dr = xyz_norm(make_xyz_uniform()) * p[i].radius;		
				p[i].r 	-= dr;
				p[n-1].r 	+= dr;

				//Create link between new particle and origin
				//fx->addLink(p[n-1].linkSet, A, p[n-1].origin, A, n-1);
				printf("parent link set: %i\tchild link set: %i\t",p[i].linkSet, p[n-1].linkSet);
				fx->addLink(p[n-1].linkSet, A, i, A, n-1);

				Genome genome = genomes.at(p[n-1].linkSet);

				dr = p[n-1].r - p[p[n-1].origin].r;

				vector<float> input;
				input.push_back(dr.x);
				input.push_back(dr.y);
				input.push_back(dr.z);
				input.push_back(xyz_len(dr));

				vector<float> output = genome.getOutput(input);

				printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
				printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

				p[n-1].color = output[0];
				p[n-1].density = output[1];
				p[n-1].remove = output[2]<=0;
	
				fx->applyParticleArray(A); 					// apply changes before any operation
			}
		}
		//printf("number of particles: %d\n", n);

		fx->runPair(particlePair(), A, A, RANGE);
		fx->runEach(integrate(), A);
		fx->runEach(buoyancy(), A);
		//fx->runEach(photosynthesis(), A);	
		//fx->runEach(friction(), A);
		fx->runEach(boundary(), A);

		if (step % 1 == 0) {
			//printf("step %d\n", step);
			fx->outputFrame("output");
		}
	}

	delete fx;
}

