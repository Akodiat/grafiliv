#include "hip/hip_runtime.h"
//#include "fluidix.h"
#define PARTICLE_BITS 28
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/structures.h"
#include "../lib/genome.h"
#include "../lib/nerveSystem.h"
#include "../lib/linearAlgebra.h"
#include "../lib/io.h"
#include <queue>
#include <ppl.h>

// Transfer amount f from a to b
#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}

#define sphereVolume(r) (4.0f / 3.0f) * r * r * r * PI;

// Check if particle position is defined correctly
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;

int currGenomeIndex;
int step;

// Turn a particle into Detritus (dead cell) type
// Previous energy is preserved
#define turnIntoDetritus(p) {       \
    p.particleType = Detritus;      \
    p.density = g.fluidDensity * 2; \
    p.organism = -1;                \
}                                   \

// Turn a particle into energy type
#define resetEnergy(p) {               \
    p.particleType = Energy;           \
    p.r.x = rnd_uniform() * g.w.x;     \
    p.r.y += g.w.y;                    \
    p.r.z = rnd_uniform() * g.w.z;     \
    p.energy = g.energyParticleEnergy; \
    p.radius = g.energyParticleRadius; \
    p.density = 10.0f;                 \
}

// Turn a particle into buffer type
// Place below arena
#define turnIntoBuffer(p) {            \
    p.particleType = Buffer;           \
    p.energy = 0.0f;                   \
    p.r = make_xyz(                    \
        rnd_uniform() * g.w.x,         \
        rnd_uniform() * g.w.y - g.w.y, \
        rnd_uniform() * g.w.z          \
    );                                 \
    p.links[0] = p.links[1] = p.links[2] =     \
    p.links[3] = p.links[4] = p.links[5] = -1; \
    p.density = g.fluidDensity;        \
    p.signal = 0.0f;                   \
    p.color = 0.5f;                    \
    p.radius = 1.0f;                   \
    p.organism = -1;                   \
}

// Update position r of particles given
// velocity v and force f
FUNC_EACH(integrate,
    p.v += p.f * g.dt;
    p.r += p.v * g.dt;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
    )

// Decrease energy through metabolism in
// cells and decay in detritus.
// cell --> detritus --> buffer
FUNC_EACH(handleEnergy,
    p.color = p.energy * 0.2f;
    switch (p.particleType) {
    case Cell:
        //printf("Cell energy: %.2f\n", p.energy);
        p.energy -= p.metabolism * g.dt;
        if (p.energy < g.minCellEnergy)
            turnIntoDetritus(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Detritus:
        p.energy -= g.cellDecayRate * g.dt;
        if (p.energy <= g.minDetritusEnergy)
            p.toBuffer = true;
        break;
    case Energy:
        if (p.energy <= 0)
            p.toBuffer = true;
    }
)

// Particles float depending on their density
FUNC_EACH(buoyancy,
    float volume = sphereVolume(p.radius);
    float weight = p.density * volume;
    float displacedFluidWeight = g.fluidDensity * volume;
    float apparentWeight = weight - displacedFluidWeight;
    p.f.y += apparentWeight * g.gravity;
)

FUNC_EACH(countParticles,
    switch (p.particleType) {
    case Cell:
        addInteger(g.nCells, 1); break;
    case Detritus:
        addInteger(g.nDetritus, 1); break;
    case Buffer:
        addInteger(g.nBuffer, 1); break;
    case Energy:
        addInteger(g.nEnergy, 1); break;
    }
)

#define WALL 100.0f // repulsive wall force
// Periodic boundary conditions
FUNC_EACH(boundary,
    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        printf("Wierd particle at (%.2f, %.2f, %.2f) type=%i\n", p.r.x, p.r.y, p.r.z, p.particleType);
        if (p.particleType == Energy) {
            p.r.y = rnd_uniform() * g.w.y;
            resetEnergy(p);
        }
        else {
            turnIntoBuffer(p);
            p.toBuffer = true;
        }
    } 
    if (p.particleType != Buffer) {
        /*
        if (p.r.x < 0)     p.r.x = g.w.x;
        if (p.r.x > g.w.x) p.r.x = 0;
        if (p.r.z < 0)     p.r.z = g.w.z;
        if (p.r.z > g.w.x) p.r.z = 0;
        */
        if (p.r.x < 0) p.f.x += WALL * (0 - p.r.x);
        if (p.r.x > g.w.x) p.f.x += WALL * (g.w.x - p.r.x);
        if (p.r.z < 0) p.f.z += WALL * (0 - p.r.z);
        if (p.r.z > g.w.z) p.f.z += WALL * (g.w.z - p.r.z);

        if (p.particleType == Energy) {
            if (p.r.y < 0) {
                resetEnergy(p);
            }
        }
        else {
            if (p.r.y < 0) {
				p.v.y = 0.9f * (0 - p.r.y) / g.dt;
				p.r.y = 0;
			}
            if (p.r.y > g.w.y) {
				p.v.y = 0.9f * (g.w.y - p.r.y) / g.dt;
				p.r.y = g.w.y;
			}
        }
    }
    else if (p.r.y < -2 * g.w.y) p.r.y += g.w.y;
)

// Let particle a eat of particle b until full
#define getEnergyNeed(a,b) maxf(minf((a.maxEnergy - a.energy), b.energy),0)

// For each particle within a predifined distance
FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        xyz f = u * maxf(
            (g.repulsiveForce * (1 - dr / (p1.radius + p2.radius))),
            0
        );
        if (p1.particleType == Cell && p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                bool neighbours = false;
                for (int n = 0; n < 6; n++) {
                    if (p1.links[n] == p2_index || p2.links[n] == p1_index) {
                        neighbours = true;
                        break;
                    }
                }
                if (neighbours) {
                    //Spring force between neighbours
                    f = -u * ((dr - (p1.radius + p2.radius)/2) * g.springForce);

                    float p1Surplus = maxf(p1.energy - g.minCellEnergy, 0);
                    float p2Surplus = maxf(p2.energy - g.minCellEnergy, 0);
                    transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                    transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
                }
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoDetritus(p2);
                    //transmitFloat(p2.energy, p1.energy, 0.5f);
                    float need = getEnergyNeed(p1, p2);
                    transmitFloat(p2.energy, p1.energy, need);
                }
                if (p2.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoDetritus(p2);
                    //transmitFloat(p1.energy, p2.energy, 0.5f);
                    float need = getEnergyNeed(p2, p1);
                    transmitFloat(p1.energy, p2.energy, need);
                }
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Detritus)
                )
            {
                float need = getEnergyNeed(p1, p2);
                transmitFloat(p2.energy, p1.energy, need);
            }
        }
        //If p2 is a cell
        else if (p2.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Detritus)
                )
            {
                float need = getEnergyNeed(p2, p1);
                transmitFloat(p1.energy, p2.energy, need);
            }
        }

        if (p1.particleType == Cell && p1.type == Sense) addFloat(p1.signal, 1.0f/dr);
        if (p2.particleType == Cell && p2.type == Sense) addFloat(p2.signal, 1.0f/dr);

        addVector(p1.f, f);
        addVector(p2.f, -f);
    }
)

// Collision with terrain
FUNC_SURFACE(collideGround,
	if (p.particleType != Energy){
		if (dr > 1) dr = 1;
		p.f += g.groundRepulsiveForce * u * dr;
	}
)

// Initialize particle as cell
void setDefaultCellValues(Particle *cell) {
    //cell->radius = 1.0f;
    cell->energy = g.initialCellEnergy;
    //cell->density = g.fluidDensity * 1.10f;
    cell->particleType = Cell;
}

// Given a phenotype network output, apply it to the cell
bool applyPhenotype(vector<float> output, Particle *cell) {
    // If cell should not exist, return
    if (output[N_CELL_TYPES] < g.cellExistenceThreshold)
        return false;
    float radius = output[N_CELL_TYPES + 1];
    cell->radius = clamp((radius/2) + 0.5f, 0.5f, 1.0f);
    //if (cell->radius < 0) cell->radius = 0.5f;
    //printf("Radius mapped from %.2f to %.2f\n", radius, cell->radius);
    //float volume = sphereVolume(cell->radius);
    //float mass = 1.0f;

    cell->density = g.fluidDensity * 1.3f; //mass/volume;

    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    // If no outputs had a positive value, return
    if (max <= 0) {
        return false;
    }
    switch (cell->type) {
    case Photo:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.5f;
        cell->maxEnergy = 10.0f;
        break;
    case Digest:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.5f;
        cell->maxEnergy = 10.0f;
        break;
    case Fat:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 50.0f;
        break;
    case Sense:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
        break;
    case Egg:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 1000.0f;
        break;
    case Vascular:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 3.0f;
        break;
    case Sting:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.5f;
        cell->maxEnergy = 10.0f;
        break;
    case Buoyancy:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
        cell->density = g.fluidDensity * 0.01f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
    }

    return true;
}

// Remove cell links from and to cell
void disconnectCell(Particle *p, int cell, int code) {
    for (int i = 0; i < 6; i++) {
        if (p[cell].links[i] >= 0)
            p[p[cell].links[i]].links[(i + 3) % 6] = code;
    }
    p[cell].toBuffer = true;
}
void emptyCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -1);
}
void deadCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -2);
}

// Helper function to get the 1-dimensional index
// given x,y,z and box size br
int getIdxFromCoord(int x, int y, int z, int3 br)
{
    x += br.x; y += br.y; z += br.z;
    int lY = 2 * br.y + 1;
    int lZ = 2 * br.z + 1;
    return x*lY*lZ + y*lZ + z;
}
#define iFromCoord(x,y,z) cellBuff.at(getIdxFromCoord(x,y,z,br))

// Create cells of an organism given a genome and a nervous system
pair<int, vector<int>> createCellsFromGenotype(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, Genome *genome, NerveSystem *nerveSys, OrganismMap *organisms)
{
    int nParticlesNeeded = genome->getMaxCellsReq();
    if (nParticlesNeeded > particleBuffer->size()) {
        cerr << "Not enought particles in buffer\n" << endl;
    }
    vector<int> cellBuff;
/*  
    while (nParticlesNeeded) {
        int particle;
        if (particleBuffer->try_pop(particle)) {
            cellBuff.push_back(particle);
            nParticlesNeeded--;
        }
        else
            printf("Failed to retrive from buffer, trying again\n");
    }
*/
    while (nParticlesNeeded--) {
        int particle = particleBuffer->front();
        particleBuffer->pop();
        cellBuff.push_back(particle); 
    }

    int3 br = genome->getBoundingRadius();

    int organismID = currGenomeIndex++;
    vector<int> removedCells;
    vector<int> addedCells;

    int nSensors = 0;

    for (int x = -br.x; x <= br.x; x++)
    for (int y = -br.y; y <= br.y; y++)
    for (int z = -br.z; z <= br.z; z++) {
        Particle *cell = &p[iFromCoord(x, y, z)];
        cell->organism = organismID;
        cell->r = origin + make_xyz(x, y, z);
        cell->energy = g.initialCellEnergy;
        cell->metabolism = g.cellMetabolism +
            g.nerveCost * nerveSys->getSize();
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x, y, z)));

        vector<float> output = genome->getOutput(input);

        if (applyPhenotype(output, cell)) {
            cell->links[Left] = x + 1 < br.x ? iFromCoord(x + 1, y, z) : -1;
            cell->links[Up] = y + 1 < br.y ? iFromCoord(x, y + 1, z) : -1;
            cell->links[Back] = z + 1 < br.z ? iFromCoord(x, y, z + 1) : -1;
            cell->links[Right] = x - 1 >= 0 ? iFromCoord(x - 1, y, z) : -1;
            cell->links[Down] = y - 1 >= 0 ? iFromCoord(x, y - 1, z) : -1;
            cell->links[Front] = z - 1 >= 0 ? iFromCoord(x, y, z - 1) : -1;

            if (cell->type == Sense)
                nSensors++;

            float volume = cell->radius * cell->radius * cell->radius * PI * 4 / 3;
            cell->metabolism += volume * 0.05f;
            addedCells.push_back(iFromCoord(x, y, z));
        }
        else
            removedCells.push_back(iFromCoord(x, y, z));
    }
    for (int i : removedCells)
        emptyCellPos(p, i);

    nerveSys->updateInputs(nSensors);

    return pair<int, vector<int>>(organismID, addedCells);
}

//Initialize new organism (without parent)
int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, Genome genome, NerveSystem nerveSys, OrganismMap *organisms)
{
    pair<int, vector<int>> o = createCellsFromGenotype(
        origin, particleBuffer, p, &genome, &nerveSys, organisms
    );
    int organismID    = o.first;
    vector<int> cells = o.second;

    Organism organism = { genome, nerveSys, cells, -1, g.orgInitHealth };

    //Add organism to organism map
    organisms->emplace(organismID, organism);

    //Output organism to disk
    outputOrganism(&organism, organismID, step);

    return organismID;
}

// Initialize new organism from parent
int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, int parent, OrganismMap *organisms)
{
	Genome genome;
    NerveSystem nerveSys;

	if(parent == -1) {
		int3 gridDim = make_int3(1,1,1);

		// Define number of in- and outputs
		int inputs = g.nGenomeInputs;       // X, Y, Z, Dist
		int nonCelltypeOutputs = 2;         // Cell existence, cell radius
		int outputs = N_CELL_TYPES + nonCelltypeOutputs;
		genome = Genome(inputs, outputs, gridDim);

        int nerveOutputs = 3;
        nerveSys = NerveSystem(nerveOutputs);
	}
	else {
		genome = Genome(organisms->at(parent).genome);
        nerveSys = NerveSystem(organisms->at(parent).nerveSystem);
	}

    genome.mutate();
    nerveSys.mutate();

    pair<int, vector<int>> o = createCellsFromGenotype(
        origin, particleBuffer, p, &genome, &nerveSys, organisms
        );
    int organismID    = o.first;
    vector<int> cells = o.second;

    Organism organism = { genome, nerveSys, cells, parent, g.orgInitHealth };

    //Add organism to organism map
    organisms->emplace(organismID, organism);

    //Output organism to disk
    outputOrganism(&organism, organismID, step);

    return organismID;
}

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

// Initialize a random organism
int initializeOrganism(ParticleBuffer *particleBuffer, Particle *p, OrganismMap *organisms)
{
    xyz origin = make_xyz_uniform() * int3_to_xyz(g.w);
    origin.y /= 2;
    origin.y += g.w.y / 2;

    return spawnOrganism(origin, particleBuffer, p, -1, organisms);
}

Matrix3 getTransform(xyz front, xyz right, xyz up, xyz back, xyz left, xyz down) {
    return Matrix3(
        xyz_norm(right - left),
        xyz_norm(up - down),
        xyz_norm(front - back)
    );
}

void addTerrainLink(int n1, int n2, Fluidix<> *fx, int meshLinks, int meshParticles, vector<tuple<int, int>> *links){
    fx->addLink(meshLinks, meshParticles, n1, meshParticles, n2);
    links->push_back(make_tuple(n1, n2));
}

int generateTerrain(Fluidix<> *fx){
    exponential_distribution<float> rndUniform(1);

    int terrDimX = 10;
    int terrDimZ = 10;

    int nParticles = (terrDimX*terrDimZ * 2);

    int meshParticles = fx->createParticleSet(nParticles);
    int meshLinks = fx->createLinkSet();
    Particle *mesh = fx->getParticleArray(meshParticles);

    vector<tuple<int, int>> links;

    float dx = g.w.x / (terrDimX - 1);
    float dz = g.w.z / (terrDimZ - 1);

    float margin = 1.2f;
    float shiftX = ((margin - 1)*g.w.x) / 2;
    float shiftZ = ((margin - 1)*g.w.z) / 2;

    for (int x = 0; x < terrDimX; x++)
    for (int z = 0; z < terrDimX; z++){
        int i = x*terrDimZ + z;
        mesh[i].r = make_xyz(
            (x*dx)*margin - shiftX,
            rndUniform(rndGen) * 10 + 10,
            (z*dz)*margin - shiftZ
            );
        mesh[i + nParticles / 2].r = make_xyz(
            x*dx,
            0,
            z*dz
            );

        //Link terrain particles together:
        int s = (x - 1)*terrDimZ + z;
        int w = x*terrDimZ + (z - 1);
        int sw = (x - 1)*terrDimZ + (z - 1);

        if ((x - 1) >= 0) addTerrainLink(i, s, fx, meshLinks, meshParticles, &links);//fx->addLink(meshLinks, meshParticles, i, meshParticles, s);
        if ((z - 1) >= 0) addTerrainLink(i, w, fx, meshLinks, meshParticles, &links);//fx->addLink(meshLinks, meshParticles, i, meshParticles, w);
        if ((x - 1) >= 0 && (z - 1) >= 0) addTerrainLink(i, sw, fx, meshLinks, meshParticles, &links);//fx->addLink(meshLinks, meshParticles, i, meshParticles, sw);

        if ((x - 1) >= 0) addTerrainLink(i + (nParticles / 2), s + (nParticles / 2), fx, meshLinks, meshParticles, &links); //fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, s + (nParticles / 2));
        if ((z - 1) >= 0) addTerrainLink(i + (nParticles / 2), w + (nParticles / 2), fx, meshLinks, meshParticles, &links); //fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, w + (nParticles / 2));
        if ((x - 1) >= 0 && (z - 1) >= 0) addTerrainLink(i + (nParticles / 2), sw + (nParticles / 2), fx, meshLinks, meshParticles, &links); //fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, sw + (nParticles / 2));

        if (x % (terrDimX - 1) == 0 || z % (terrDimZ - 1) == 0){
            //fx->addLink(meshLinks, meshParticles, i, meshParticles, i + (nParticles / 2));
            addTerrainLink(i, i + (nParticles / 2), fx, meshLinks, meshParticles, &links);

            //if (x > 0 && z > 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, s + (nParticles / 2));
        }
        fx->applyParticleArray(meshParticles);

        saveTerrain(mesh, links, nParticles);

        fx->outputFrame("dump");
    }
    return meshLinks;
}

int main() {
    // Create Fluidix library object
    Fluidix<> *fx = new Fluidix<>(&g);

    // Load configuration file
    g = loadConfig("conf.txt");

    // Create a particle set with number of particles
    // equal to g.nParticles
    int pSet = fx->createParticleSet(g.nParticles);
    int terrain = generateTerrain(fx);

    currGenomeIndex = 0;
    OrganismMap organisms;

    // Initialize buffer
    ParticleBuffer particleBuffer;

    Particle *p = fx->getParticleArray(pSet);


    cout << endl << endl << "Would you like to load a saved state dump?" << endl;
    cout << "(or rather start anew?)\ty/n: ";
    char choice;
    cin >> choice;
    if (choice == 'y') {
        loadCompleteState(&organisms, p, &particleBuffer, fx, pSet, &step, &currGenomeIndex);
        fx->applyParticleArray(pSet);
    }
    else if (choice == 'n') {
        step = 0;
        int i = 0;
        int neededEnergy = g.energyParticleCount;

        // Initialize energy particles
        while (neededEnergy--) {
            p[i].r.y = rnd_uniform() * g.w.y;
            resetEnergy(p[i]);
            i++;
        }

        // Turn the rest of the particles into buffer
        while (i < g.nParticles) {
            turnIntoBuffer(p[i]);
            particleBuffer.push(i);
            i++;
        }

        //for (int i = 0; i < 1000; i++)
        loadOrg("initOrg.json", &particleBuffer, p, &organisms);
    }
    else return -1;

    fx->applyParticleArray(pSet);

    FILE *countCells = fopen("countCells.csv", "w");
    fprintf(countCells, "nDetritus,nBuffer,nEnergy,nCells\n");

    //FILE *monitorParticle = fopen("monitorParticle.csv", "w");
    //fprintf(monitorParticle, "particleType,r.x,r.y,r.z,v.x,v.y,v.z,f.x,f.y,f.z,color,radius,alpha,density,energy,energyIn,energyOut,maxEnergy,signal,metabolism,organism,toBuffer,link0,link1,link2,link3,link4,link5,type\n");

    while(step++ < g.nSteps) {
        fx->runEach(boundary(), pSet);
        fx->runSurface(collideGround(), terrain, pSet);
        fx->runPair(particlePair(), pSet, pSet, g.interactionRange);

        p = fx->getParticleArray(pSet);
        vector<int> organismsToRemove;
        for (auto& iOrg : organisms) {
            Organism *o = &iOrg.second;
            o->health -= g.dt;

            vector<float> inputs;
            vector<int> eggs;
            int nLiving = 0;
            int nDead = 0;
            for (int i : o->cells){
                if (p[i].particleType == Cell){
                    if (p[i].type == Sense)
                        inputs.push_back(p[i].signal);
                    if (p[i].type == Egg)
                        eggs.push_back(i);
                    nLiving++;
                }
                else
                nDead++;
            }
            if (o->health <= 0 || nDead > nLiving) {
                for (int i : o->cells)
                if (p[i].particleType == Cell)
                    turnIntoDetritus(p[i]);
                organismsToRemove.push_back(iOrg.first);

                if (nDead > nLiving) logOrgDeath(iOrg.first, step, "disintegration");
                else logOrgDeath(iOrg.first, step, "age");

                continue;
            }
            vector<float> output = o->nerveSystem.getOutput(inputs);

            xyz f = make_xyz(output[0], output[1], output[2]);
            for (int i : o->cells) {
                if (p[i].particleType == Cell){
                    int *ns = p[i].links;
                    
                    xyz front = ns[Front] >= 0 ? p[ns[Front]].r : make_xyz( 0, 0, 1);
                    xyz right = ns[Right] >= 0 ? p[ns[Right]].r : make_xyz( 1, 0, 0);
                    xyz up    = ns[Up]    >= 0 ? p[ns[Up]].r    : make_xyz( 0, 1, 0);
                    xyz back  = ns[Back]  >= 0 ? p[ns[Back]].r  : make_xyz( 0, 0,-1);
                    xyz left  = ns[Left]  >= 0 ? p[ns[Left]].r  : make_xyz(-1, 0, 0);
                    xyz down  = ns[Down]  >= 0 ? p[ns[Down]].r  : make_xyz( 0,-1, 0);

                    //What if two neigbours are on opposite sides of a boundary??

                    Matrix3 m = getTransform(
                        front - p[i].r,
                        right - p[i].r,
                        up - p[i].r,
                        back - p[i].r,
                        left - p[i].r,
                        down - p[i].r
                    );
                    p[i].f += m.dot(f) * g.moveFactor;
                    
                    //p[i].f += f;
                    p[i].energy -= xyz_len(f) * g.moveCost;
                    p[i].signal *= 0.5f;
                }
            }
            // Hatch eggs if they have enought energy:
            for (int i : eggs) {
                int maxReqEnergy =
                    g.initialCellEnergy *
                    o->genome.getMaxCellsReq() +
                    o->genome.getSize() * g.genomeCost;

                if (p[i].energy >= maxReqEnergy + g.initialCellEnergy) {
                    spawnOrganism(
                        p[i].r, &particleBuffer,
                        p, p[i].organism, &organisms
                    );
                    p[i].energy -= maxReqEnergy;
                    fx->applyParticleArray(pSet);
                }
            }
        }
        for (int i : organismsToRemove)
            organisms.erase(i);
        fx->applyParticleArray(pSet);

        fx->runEach(buoyancy(), pSet);
        fx->runEach(handleEnergy(), pSet);
        fx->runEach(integrate(), pSet);
        
        //if (step % 100 == 0) { ParticleBuffer empty; swap(particleBuffer, empty); }
        for (int i = 0; i<g.nParticles; i++)
        {
            if (p[i].toBuffer && p[i].particleType != Energy) {
                turnIntoBuffer(p[i]);
                particleBuffer.push(i);
                p[i].toBuffer = false;
                fx->applyParticleArray(pSet);
            }
            //else if (step % 100 == 0 && p[i].particleType == Buffer){
            //    particleBuffer.push(i);
            //}

        }

        g.nDetritus = g.nBuffer = g.nEnergy = g.nCells = 0;
        fx->runEach(countParticles(), pSet);

        //If buffer is getting to small, increase it
        //by adding more particles to the simulation
        if (particleBuffer.size() < g.bufferSize) {
            int currentParticleCount = g.nParticles;
            g.nParticles += g.bufferSize;
            printf("Increasing particle array size from %i to %i\n", currentParticleCount, g.nParticles);
            fx->resizeParticleSet(pSet, g.nParticles);
            p = fx->getParticleArray(pSet);
            for (int i = currentParticleCount; i < g.nParticles; i++) {
                turnIntoBuffer(p[i]);
                particleBuffer.push(i);
            }
            fx->applyParticleArray(pSet);
        }
        else if (particleBuffer.size() > g.nParticles / 2) {
            //printf("Decreasing buffer size from %i", g.nParticles);
            int nBuffersAtEnd = 0;
            //printf("End particle type: %i (buffer is %i)\n", p[g.nParticles - nBuffersAtEnd - 1].particleType, Buffer);
            while (p[g.nParticles - nBuffersAtEnd - 1].particleType == Buffer &&
                particleBuffer.size() - nBuffersAtEnd > g.nParticles / 2
            ){
                nBuffersAtEnd++;
            }
            if (nBuffersAtEnd > 0){
                printf("Decreasing particle array size from %i to %i\n", g.nParticles, g.nParticles - nBuffersAtEnd);
                g.nParticles -= nBuffersAtEnd;
                fx->resizeParticleSet(pSet, g.nParticles);
                p = fx->getParticleArray(pSet);
                fx->applyParticleArray(pSet);

                ParticleBuffer empty;
                swap(particleBuffer, empty);

                for (int i = 0; i<g.nParticles; i++)
                    if (p[i].particleType == Buffer)
                        particleBuffer.push(i);
            }
        }

        if (step % g.saveFreq == 0){
            fprintf(countCells, "%i,%i,%i,%i\n", g.nDetritus, g.nBuffer, g.nEnergy, g.nCells);
        }

        if (
            (step % g.saveFreq == 0) &&
            (((int)(step / g.saveIntervalLength)) % (g.saveIntervalDistance / g.saveIntervalLength)) == 0)
        {
            printf("nOrgs: %i\t", organisms.size());
            printf("currgenomeIndex: %i\t", currGenomeIndex);
            printf("buffer: %i (in queue), %i (actual)\t", particleBuffer.size(), g.nBuffer);
            printf("step %d\n", step);
            outputParticles(p, g.nParticles, step);
        }
        else if (step % 100 == 0) printf(".");

        //int mI = g.energyParticleCount+1; //Not energy
        /*
        fprintf(
            monitorParticle, 
            "%i,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%i,%d,%i,%i,%i,%i,%i,%i,%i\n", 
            p[mI].particleType, p[mI].r.x, p[mI].r.y, p[mI].r.z, p[mI].v.x, p[mI].v.y, p[mI].v.z, p[mI].f.x, p[mI].f.y, p[mI].f.z,
            p[mI].color, p[mI].radius, p[mI].alpha, p[mI].density, p[mI].energy, p[mI].energyIn, p[mI].energyOut, p[mI].maxEnergy,
            p[mI].signal, p[mI].metabolism, p[mI].organism, p[mI].toBuffer, p[mI].links[0], p[mI].links[1], p[mI].links[2], p[mI].links[3],
            p[mI].links[4], p[mI].links[5], p[mI].type
        );
        */

        if (step % 10000 == 0) {
            fx->outputFrame("dump");
            dumpCompleteState(p, g.nParticles, step);
        }

        if (organisms.size() == 0) {
            printf("All organisms died. End of simulation\n");
            break;
        }
    }
    fclose(countCells);
    //fclose(monitorParticle);
    delete fx;
    //system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
    return 1;
}
