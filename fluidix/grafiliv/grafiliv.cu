#include "hip/hip_runtime.h"
#include "fluidix.h"

#define W 100 // world size (cube)
#define N 1000 // number of particles
#define STEPS 100000 // number of simulation steps
#define DT 0.01f // integration time-step
#define RANGE 4.0f // fluid interaction cutoff range
#define HARDNESS 20.0f // repulsive force strength
#define FLUID_DENSITY 1.0f

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 40


#define PHOTOSYNTHESIS_MAX 0.1f
#define CELL_BASIC_METABOLISM 0.3f
#define CELL_BASIC_ENERGY 0.1f
#define CELL_MAX_ENERGY 1.0f
#define CELL_INITIAL_ENERGY 1.0f

#define MUTATION_RATE 0.01f

#define G -9.81f

#define RESURRECT(parent, child) {																							\
	child.energy 					= parent.energy/2;																			\
	parent.energy 				= parent.energy/2;																			\
	child.alpha 					= 1.0f;																						\
	child.remove 					= false;																						\
	child.radius 					= (child.energy + CELL_BASIC_ENERGY)/2;													\
	child.invertSignal 			= rnd_uniform() > MUTATION_RATE ? parent.invertSignal : !parent.invertSignal;	\
	child.density 				= parent.density;																			\
	child.phoSyAbility 			= parent.phoSyAbility 		+ rnd_normal() * MUTATION_RATE;							\
	child.moveAbility 			= parent.moveAbility 		+ rnd_normal() * MUTATION_RATE;							\
	child.senseAbility 			= parent.senseAbility 		+ rnd_normal() * MUTATION_RATE;							\
	child.recieveSigAbility		= parent.recieveSigAbility	+ rnd_normal() * MUTATION_RATE;							\
	child.sendSigAbility 		= parent.sendSigAbility 	+ rnd_normal() * MUTATION_RATE;							\
}

struct Particle {
	xyz r, v, f; 				// position, velocity, force
	xyz signal;
	xyz sense;
	float density;
	float radius;
	float energy;
	float color; 				// display color
	float alpha;
	bool remove;

	//Genome:

	float phoSyAbility; 		// Ability to photosynthesize
	float moveAbility;			// Ability to move
	float senseAbility;			// Ability to sense surroundings

	bool  invertSignal;			// Inverts signal before acting or transmitting
	float recieveSigAbility;  	// Ability to recieve signals	
	float sendSigAbility;		// Ability to send signals
};

struct Node {
    int x;
    Node *next;
};

struct Global {
	//Node *head;
} g;

/*
#define add_toDivide(val){Node *node = new Node(); node->x = val; node->next = g.head; g.head = node;}

void next_toDivide(){
    Node *n = g.head;
    g.head = g.head->next;
    delete n;
}
bool has_toDivide() {
	return g.head != NULL;
}
*/

// initialize particles
FUNC_EACH(init,

	p.energy = CELL_INITIAL_ENERGY;
	p.r = make_xyz_uniform() * W; 					//Random position within box
	//p.radius = rnd_uniform(); 						//Random size
	p.radius = (p.energy + CELL_BASIC_ENERGY)/2;
	p.density = FLUID_DENSITY; // abs(rnd_normal() / 10 + FLUID_DENSITY); 	//Random density
	p.alpha = 1.0f;
	p.remove = false;
	
	p.invertSignal 		= rnd_uniform() < 0.5f;
	p.phoSyAbility 		= 0.0f; // abs(rnd_normal());
	p.moveAbility 		= 0.0f; // abs(rnd_normal());
	p.senseAbility 		= 0.0f; // abs(rnd_normal());
	p.recieveSigAbility = 0.0f; // abs(rnd_normal());
	p.sendSigAbility 	= 0.0f; // abs(rnd_normal());

	//p.color = p.moveAbility;
	
	p.sense  = make_xyz(0, 0, 0);
	p.signal = make_xyz_uniform()*5;	//Random initial signal
)

// linear repulsion + attraction at distance
FUNC_PAIR(pair,
	float ratio = dr / range;
	float interactivity =  (p1.recieveSigAbility * p2.sendSigAbility * p2.recieveSigAbility * p1.sendSigAbility);

	xyz f = u * (REPULSION_FORCE * (1-ratio) * interactivity - ATTRACTION_FORCE * ratio );
	addVector(p1.f, f);
	addVector(p2.f, -f);

	// Update sense values
	addVector(p1.sense, -u);	// These should be normalized...
	addVector(p2.sense, u);

	// Update signal values with sense
	addVector(p1.signal, p1.sense * p1.senseAbility);
	addVector(p2.signal, p2.sense * p2.senseAbility);

	// Transmit signal
	addVector(p1.signal, p2.signal * p1.recieveSigAbility * p2.sendSigAbility * (p1.invertSignal ? -1 : 1));
	addVector(p2.signal, p1.signal * p2.recieveSigAbility * p1.sendSigAbility * (p2.invertSignal ? -1 : 1));

	if(p1.energy > 0 && p2.energy <= 0){
		RESURRECT(p1, p2);
		//p2.remove = true;
		//addFloat(p1.energy, CELL_BASIC_ENERGY);
	}
	else if (p2.energy > 0 && p1.energy <= 0) {
		RESURRECT(p2, p1);
		//p1.remove = true;
		//addFloat(p2.energy, CELL_BASIC_ENERGY);
	}
	else {
		//Steal energy from other particle proportional to signal alignment with particle direction
		addFloat(p1.energy, xyz_len(xyz_norm(p1.signal) + (-u)) - xyz_len(xyz_norm(p2.signal) + (u)));
		addFloat(p2.energy, xyz_len(xyz_norm(p2.signal) + (u)) - xyz_len(xyz_norm(p1.signal) + (-u)));
		//addFloat(p1.energy, p2.energy/2 - p1.energy/2);
		//addFloat(p2.energy, p1.energy/2 - p2.energy/2);
	} 
)


// buoyancy 
FUNC_EACH(buoyancy,
	float volume = p.radius * p.radius * PI;
	p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

// photosynthesis 
FUNC_EACH(photosynthesis,
	p.energy += PHOTOSYNTHESIS_MAX * p.phoSyAbility;
)

// move 
FUNC_EACH(move,
	xyz f = p.signal * p.moveAbility * 0.1f;
	addVector(p.f, f);
)

// handle energy usage
FUNC_EACH(handleEnergy,
	p.energy -= (
		p.moveAbility +
		CELL_BASIC_METABOLISM +
		p.senseAbility
	);

	// If dead
	if(p.energy <= 0)
	{
		p.alpha = 0.5f;
		p.density = FLUID_DENSITY;
		p.phoSyAbility = p.moveAbility = p.senseAbility = p.recieveSigAbility = p.sendSigAbility = 0.0f;
	}
	if(p.energy > CELL_MAX_ENERGY){
		p.energy = CELL_MAX_ENERGY;
		//p.energy /= 2;
		//add_toDivide(p_index);
	}

	p.radius = (p.energy + CELL_BASIC_ENERGY)/2;
)

// Euler integration
FUNC_EACH(integrate,
	//p.color = xyz_len(p.f) / 50.0f;

	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
	if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }
)

// deflate old signals so that they won't overflow 
FUNC_EACH(deflateSignals,
	//p.color = xyz_len(p.signal)/100;
	p.color = p.senseAbility;
	p.signal = xyz_norm(p.signal);
	p.sense = make_xyz(0, 0, 0);
)

// simulation
int main(int argc, char **argv) {
	Fluidix<> *fx = new Fluidix<>(&g);

	int A = fx->createParticleSet(N);
	fx->runEach(init(), A);

	for (int i = 0; i < STEPS; i++) {
		printf("step %d / %d: ", i, STEPS);
		fx->setTimer();

		// execute interactions
		fx->runPair(pair(), A, A, RANGE);
		fx->runEach(buoyancy(), A);
		fx->runEach(move(), A);
		fx->runEach(integrate(), A);
		fx->runEach(handleEnergy(), A);
		fx->runEach(boundary(), A);
		fx->runEach(deflateSignals(), A);
		fx->runEach(photosynthesis(), A);

		fx->removeParticles(A);

		printf("%.1f ms\n", fx->getTimer());

		// only output to file every 10th step
		if (i % 1 == 0) fx->outputFrame("output");
	}

	delete fx;
}

