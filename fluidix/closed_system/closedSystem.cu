#include "hip/hip_runtime.h"
#include "fluidix.h"
//#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
//#include "C:\Users\admin\Documents\Joakim\repo\grafiliv\fluidix\lib\genome.h"
#include "../lib/genome.h"

#define DT 0.01f // integration time-step
#define CELL_LIFETIME 50.0f
#define PELLET_LIFETIME 50.0f

#define BOX_MAX 20
#define BOX_MIN 1

#define W 200
#define N 30000
#define N_ORIGIN_CELLS 500
#define N_STEPS 1000000

//Inputs x,y,z,d:
#define N_INPUTS 4

#define RANGE 3.0f
#define MOVE_FACTOR 200

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 250

#define FLUID_DENSITY 1.0f
#define G -9.81f

#define GREEN   0.5f
#define RED     1.0f
#define YELLOW  0.7f
#define BLUE    0.0f
#define CYAN    0.3f
#define ORANGE  0.8f

using namespace std;

enum CellType {Photo, Pred, Sense, Move, Ballast, Sex,
	N_CELL_TYPES
};
enum ParticleType {Cell, Energy, Pellet,
	N_PARTICLE_TYPES
};

struct Organism {
	Genome genome;
	int linkSet;
	Organism() {}
	Organism(Genome g): genome(g), linkSet(-1) {}
};

int currGenomeIndex;

struct Global {
} g;

struct Particle {
	ParticleType particleType;
	float growthProb;
	xyz r, v, f;
	float color;
	float radius;
	float alpha;
	float density;
	float lifeTime;
	float signal;
	int organism;
	int toGrow;
	bool reproduce;
	Genome genome;
	Particle *origin;
	CellType type;
};

FUNC_EACH(init,
	p.r = (make_xyz_uniform() * W) + make_xyz(0, W, 0);
	p.color = 0.7f;
	p.alpha = 0.3f;
	p.radius = 0.5f;
	p.density = FLUID_DENSITY * 2;
	p.particleType = Energy;
	p.toGrow = -1;


	p.growthProb = -1.0f;
	p.lifeTime = 0.0f;
	p.signal = 0.0f;
	p.organism = -1;
	p.reproduce = false;
	p.origin = NULL;
)

FUNC_EACH(integrate,
	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
	p.v *= 0.97f;
)


#define turnIntoPellet(p) {			\
	p.particleType = Pellet;			\
	p.lifeTime = PELLET_LIFETIME;	\
	p.density = FLUID_DENSITY * 2;	\
	p.alpha = 0.5f;						\
	p.organism = NULL;					\
}											\

#define turnIntoEnergy(p) {			\
	p.particleType = Energy;			\
	p.r = make_xyz(						\
		rnd_uniform()*W,				\
		W,									\
		rnd_uniform()*W					\
	);										\
	p.color = 0.7f;						\
	p.alpha = 0.3f;						\
	p.radius = 0.5f;					\
	p.density = FLUID_DENSITY * 2;	\
}

FUNC_EACH(age,
	if (p.particleType != Energy) {
		p.lifeTime -= DT;
		if (p.lifeTime <= 0) {
			if (p.particleType == Cell){
				turnIntoPellet(p);
			} else if (p.particleType == Pellet){
				turnIntoEnergy(p);
			}
		}
		if (p.particleType == Cell && p.origin->particleType != Cell)
			turnIntoPellet(p);
	}
)

FUNC_EACH(buoyancy,
	if(p.particleType == Cell && p.type == Ballast)
		p.density = clamp(p.density + p.signal, 0.5f, 2.0f);
	float volume = p.radius * p.radius * PI;
	p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

FUNC_EACH(moveParticle,
	if(p.particleType == Cell && p.type == Move) {
		xyz f = xyz_norm(p.origin->r - p.r) * MOVE_FACTOR;
		addVector(p.f, f);
	}
)

FUNC_EACH(reproduce,
	if(p.particleType == Cell && p.type == Sex) {
		if(rnd_uniform() < p.signal)
			p.reproduce = true;
	}
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }

	if(p.particleType == Energy){
		if (p.r.y < 0) {
			p.r.x = rnd_uniform() * W;
			p.r.z = rnd_uniform() * W;
			p.r.y = W;
		}
	} else {
		if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
		if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	}
)

#define harvestParticle(a, b, b_index) { 		\
	if(rnd_uniform() < a.growthProb)				\
		a.toGrow = b_index; 							\
	else {													\
		a.origin->lifeTime	+= CELL_LIFETIME/2;	\
		a.lifeTime 			+= 	CELL_LIFETIME/2;	\
		turnIntoEnergy(b);								\
	}														\
}

FUNC_PAIR(particlePair,
	float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
	//float ratio = dr/range;

	float attraction = 0.0f;
	if (p1.particleType == Cell &&
	    p2.particleType == Cell)
	{
		//Cells from the same organism
		if (p1.organism == p2.organism) {
			//Attraction between cells of same organism
			attraction = ATTRACTION_FORCE * ratio;

			//Signalling between cells of same organism
			float meanSignal = (p1.signal + p2.signal)/2;
			p1.signal = p2.signal = meanSignal;
		//Cells from differant organisms
		} else {
			//Eat the other cell if you are predatory
			if (p1.type == Pred) harvestParticle(p1, p2, p2_index)
			if (p2.type == Pred) harvestParticle(p2, p1, p1_index)
		}
	//If p1 is a cell
	} else if (p1.particleType == Cell) {
		if ((p1.type == Photo && p2.particleType == Energy) ||
			(p1.type == Pred  && p2.particleType == Pellet)
		) harvestParticle(p1, p2, p2_index)
	}
	//If p2 is a cell
	else if (p2.particleType == Cell) {
		if ((p2.type == Photo && p1.particleType == Energy) ||
			(p2.type == Pred  && p1.particleType == Pellet)
		) harvestParticle(p2, p1, p1_index)
	}

	if(p1.particleType == Cell && p1.type == Sense) p1.signal += 0.1f;
	if(p2.particleType == Cell && p2.type == Sense) p2.signal += 0.1f;

	xyz f = u * (REPULSION_FORCE * (1-ratio) - attraction);

	addVector(p1.f, f);
	addVector(p2.f, -f);

	//p1.color = p1.signal;
	//p2.color = p2.signal;
)

void setDefaultCellValues(Particle *cell) {
	cell->alpha		= 1.0f;
	cell->radius		= 1.0f;
	cell->lifeTime	= CELL_LIFETIME;
	cell->density	= FLUID_DENSITY;
	cell->particleType = Cell;
}

void applyPhenotype(vector<float> output, Particle *cell) {
    float max = output[0]; cell->type = (CellType) 0;
    for(int j=1; j<N_CELL_TYPES; j++) {
        if(output[j] > max) {
            max = output[j];
            cell->type = (CellType) j;
        }
    }
    switch(cell->type) {
        case Photo:     cell->color = GREEN;  break;
        case Pred:      cell->color = RED;    break;
        case Move:      cell->color = YELLOW; break;
        case Sense:     cell->color = BLUE;   break;
        case Ballast:   cell->color = CYAN;   break;
        case Sex:       cell->color = ORANGE; break;
    }
    cell->growthProb = output[N_CELL_TYPES];
}

// Initialize new organism, not inheriting anything
void initializeNewOrganism(Particle *cell) {
	// Define number of in- and outputs
	int inputs = N_INPUTS; 				// X, Y, Z, Dist
	int nonCelltypeOutputs = 1;			// Growth prob
	int outputs = N_CELL_TYPES + nonCelltypeOutputs;

	cell->genome = Genome(inputs, outputs);
	cell->genome.mutate();
	//cell->genome.printMathematica();

	cell->organism	= currGenomeIndex++;
	printf("First batch of species! index: %i\n", cell->organism);
	cell->r			= make_xyz_uniform() * W;
	cell->origin		= cell;
	cell->reproduce	= false;
	setDefaultCellValues(cell);

	vector<float> input(inputs, 0.0f); //Input origin
	vector<float> output = cell->genome.getOutput(input);

   applyPhenotype(output, cell);
}

// Initialize organism, inheriting from parent
void initializeOffspring(Particle *cell) {
	cell->genome.mutate();
	cell->organism = currGenomeIndex++;
	cell->origin		= cell;
	cell->reproduce	= false;
	printf("New species! index: %i\n", cell->organism);

	// Define number of in- and outputs
	int inputs = N_INPUTS; 						// X, Y, Z, Dist

	vector<float> input(inputs, 0.0f); //Input origin
	vector<float> output = cell->genome.getOutput(input);

   applyPhenotype(output, cell);
}

void growCell(Particle *parent, Particle *child) {
	normal_distribution<float> rndNormal(0.0f, 1.0f);

	//	Copy constructor
	*child = Particle(*parent);
	child->lifeTime	= CELL_LIFETIME;

	// Displace particles from each other
	xyz displacement = xyz_norm(
		make_xyz(
			rndNormal(rndGen),
			rndNormal(rndGen),
			rndNormal(rndGen)
		)) * parent->radius;

	parent->r -= displacement;
	child->r  += displacement;

	xyz dr = child->r - child->origin->r;

	vector<float> input;
	input.push_back(dr.x);
	input.push_back(dr.y);
	input.push_back(dr.z);
	input.push_back(xyz_len(dr));

	vector<float> output = child->genome.getOutput(input);

	//printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
	//printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

   applyPhenotype(output, child);
}

int main() {
	Fluidix<> *fx = new Fluidix<>(&g);
	int setA = fx->createParticleSet(N);
	//fx->createGlobalArray(&g.toGrow, N * sizeof(int));
	//for(int i=0; i<N; i++)
	//	g.toGrow[i] = -1;
	currGenomeIndex = 0;

	fx->runEach(init(), setA);

	Particle *p = fx->getParticleArray(setA);
	for(int i=0; i<N_ORIGIN_CELLS; i++)
		initializeNewOrganism(&p[i]);
	fx->applyParticleArray(setA);

	for(int step=0; step < N_STEPS; step++) {
		fx->runPair(particlePair(), setA, setA, RANGE);
		fx->runEach(moveParticle(), setA);
		fx->runEach(buoyancy(), setA);
		fx->runEach(boundary(), setA);
		fx->runEach(integrate(), setA);
		fx->runEach(age(), setA);
		fx->runEach(reproduce(), setA);

		for(int i=0; i<N; i++) {
			if(p[i].particleType == Cell) {
				if(p[i].toGrow != -1) {
					int parent = i;
					int child = p[i].toGrow;
					//printf("toGrow! parent: %i, child: %i\n", parent, child);
					growCell(&p[parent], &p[child]);
					//fx->addLink(
					//	g.organisms[p[i].organism].linkSet,
					//	setA, parent, setA, child
					//);
					p[i].toGrow = -1;
					fx->applyParticleArray(setA);
				}
				// Create offspring:
				else if(p[i].type == Sex && p[i].reproduce) {
					initializeOffspring(&p[i]);
					fx->applyParticleArray(setA);
				}
			}
			// Remove dead organisms:
			//else if(p[i].origin == &p[i])
			//	g.organisms.erase(p[i].organism);
		}

		if (step % 10 == 0) {
			printf("step %d\n", step);
			fx->outputFrame("output");
		}
	}
	//createCells(fx, make_xyz(0, 0, 0), L, setA);

	delete fx;
}
