#include "hip/hip_runtime.h"
#include "fluidix.h"
#include "../lib/genome.h"

#define DT 0.01f // integration time-step
#define CELL_LIFETIME 50.0f
#define PELLET_LIFETIME 50.0f

#define BOX_MAX 20
#define BOX_MIN 1

#define W 200
#define N 40000
#define N_ORIGIN_CELLS 100
#define N_STEPS 100000

#define RANGE 3.0f

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 250

#define FLUID_DENSITY 1.0f
#define G -9.81f

using namespace std;

enum CellType {Photo, Pred, Sense, Move,
	N_CELL_TYPES
};
enum ParticleType {Cell, Energy, Pellet,
	N_PARTICLE_TYPES
};

struct Organism {
	Genome genome;
	int linkSet;

	Organism(Genome g): genome(g), linkSet(-1) {}
};

struct Global {
	vector<Organism> organisms;
} g;

struct Particle {
	ParticleType particleType;
	float growthProb;
	xyz r, v, f;
	float color;
	float radius;
	float alpha;
	float density;
	float lifeTime;
	int organism;
	int toGrow;
	Particle *origin;
	CellType type;
};

FUNC_EACH(init,
	p.r = (make_xyz_uniform() * W) + make_xyz(0, W, 0);
	p.color = 0.7f;
	p.alpha = 0.3f;
	p.radius = 0.5f;
	p.density = FLUID_DENSITY * 2;
	p.particleType = Energy;
	p.toGrow = -1;
)

FUNC_EACH(integrate,
	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
	p.v *= 0.97f;
)


#define turnIntoPellet(p) {			\
	p.particleType = Pellet;			\
	p.lifeTime = PELLET_LIFETIME;	\
	p.density = FLUID_DENSITY * 2;	\
	p.alpha = 0.5f;						\
	p.organism = NULL;					\
}											\

#define turnIntoEnergy(p) {			\
	p.particleType = Energy;			\
	p.r = make_xyz(						\
		rnd_uniform()*W,				\
		W,									\
		rnd_uniform()*W					\
	);										\
	p.color = 0.7f;						\
	p.alpha = 0.3f;						\
	p.radius = 0.5f;					\
	p.density = FLUID_DENSITY * 2;	\
}

FUNC_EACH(age,
	if (p.particleType != Energy) {
		p.lifeTime -= DT;
		if (p.lifeTime <= 0) {
			if (p.particleType == Cell){
				turnIntoPellet(p);
			} else if (p.particleType == Pellet){
				turnIntoEnergy(p);
			}
		}
	}
)

FUNC_EACH(buoyancy,
	//if(p.type == Ballast)
	//	p.density += sin(p.signal);
	float volume = p.radius * p.radius * PI;
	p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }

	if(p.particleType == Energy){
		if (p.r.y < 0) {
			p.r.x = rnd_uniform() * W;
			p.r.z = rnd_uniform() * W;
			p.r.y = W;
		}
	} else {
		if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
		if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	}
)

#define harvestParticle(a, b, b_index) { 		\
	if(rnd_uniform() < a.growthProb)				\
		a.toGrow = b_index; 							\
	else {													\
		a.origin->lifeTime	+= CELL_LIFETIME/2;	\
		a.lifeTime 			+= 	CELL_LIFETIME/2;	\
		turnIntoEnergy(b);								\
	}														\
}

FUNC_PAIR(particlePair,
	float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
	//float ratio = dr/range;
	float attraction = 0.0f;
	if(p1.particleType == Cell && 
		p2.particleType == Cell && 
		p1.organism == p2.organism) 
	{
		attraction = ATTRACTION_FORCE * ratio;
	}
	xyz f = u * (REPULSION_FORCE * (1-ratio) - attraction);

	addVector(p1.f, f);
	addVector(p2.f, -f);

	if(	p1.organism != p2.organism && (	// Don't eat yourself.
			p1.particleType == Cell || 		// You have to be a cell to
			p2.particleType == Cell			// be able to eat
	)){
		// Photosynthesis
		if(p1.type == Photo && p2.particleType == Energy) {
			harvestParticle(p1, p2, p2_index)
		} else if(p2.type == Photo && p1.particleType == Energy) {
			harvestParticle(p2, p1, p1_index)
		// Consume other cells or pellets
		} else if(p1.type == Pred && !p2.particleType == Energy) {
			harvestParticle(p1, p2, p2_index)
		} else if(p2.type == Pred && !p1.particleType == Energy) {
			harvestParticle(p2, p1, p1_index)
		}
	}
/*
	if(p1.type == Sense) p1.signal += 0.1f;
	if(p2.type == Sense) p2.signal += 0.1f;

	if(p1.organism == p1.organism){
		float meanSignal = (p1.signal + p2.signal)/2;
		meanSignal = sin(meanSignal);
		p1.signal = p2.signal = meanSignal;
	}
*/
	//p1.color = p1.signal;
	//p2.color = p2.signal;
)

void setDefaultCellValues(Particle *cell) {
	cell->alpha		= 1.0f;
	cell->radius		= 1.0f;
	cell->lifeTime	= CELL_LIFETIME;
	cell->density	= FLUID_DENSITY;
	cell->particleType = Cell;
}

// Initialize new organism, not inheriting anything
void initializeNewOrganism(Fluidix<> *fx, Particle *cell, Global &g) {
	// Define number of in- and outputs
	int inputs = 4; 						// X, Y, Z, Dist

	int nonCelltypeOutputs = 1;			// Growth prob
	int outputs = N_CELL_TYPES + nonCelltypeOutputs;

	Organism o(Genome(inputs, outputs));
	o.linkSet = fx->createLinkSet();
	o.genome.mutate();
	o.genome.printMathematica();

	g.organisms.push_back(o);
	cell->organism	= g.organisms.size()-1;
	cell->r			= make_xyz_uniform() * W;
	cell->origin		= cell;
	setDefaultCellValues(cell);

	vector<float> input(inputs, 0.0f); //Input origin
	vector<float> output = o.genome.getOutput(input);

	float max = output[0]; cell->type = (CellType) 0;
	for(int j=1; j<N_CELL_TYPES; j++) {
		if(output[j] > max) {
			max = output[j];
			cell->type = (CellType) j;
		}
	}
	switch(cell->type) {
		case Photo:		cell->color = 0.5f; break; // Green
		case Pred:		cell->color = 1.0f; break; // Red
		case Move:		cell->color = 0.7f; break; // Yellow
		//case Sense:		cell->color = 0.3f; break; // Cyan
		//case Move:		cell->color = 1.0f; break; // Red
		//case Ballast:	cell->color = 0.0f; break; // Blue
	}
	cell->growthProb = output[N_CELL_TYPES];
}

// Initialize organism, inheriting from parent
void initializeOffspring(Fluidix<> *fx, Particle *parent, Particle *child, Global &g) {
	Organism o = Organism(g.organisms[parent->organism]);
	o.linkSet = fx->createLinkSet();
	o.genome.mutate();
	o.genome.printMathematica();

	g.organisms.push_back(o);
	child->organism = g.organisms.size()-1;
}

void growCell(Fluidix<> *fx, Particle *parent, Particle *child) {
	normal_distribution<float> rndNormal(0.0f, 1.0f);
	
	//	Copy constructor
	*child = Particle(*parent);
	child->lifeTime	= CELL_LIFETIME;
	
	// Displace particles from each other
	xyz displacement = xyz_norm(
		make_xyz(
			rndNormal(rndGen),
			rndNormal(rndGen),
			rndNormal(rndGen)
		)) * parent->radius;
		
	parent->r -= displacement;
	child->r  += displacement;

	Organism o = g.organisms[parent->organism];

	xyz dr = child->r - child->origin->r;

	vector<float> input;
	input.push_back(dr.x);
	input.push_back(dr.y);
	input.push_back(dr.z);
	input.push_back(xyz_len(dr));

	vector<float> output = o.genome.getOutput(input);

	//printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
	//printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

	float max = output[0];
	for(int j=1; j<N_CELL_TYPES; j++) {
		if(output[j] > max) {
			max = output[j];
			child->type = (CellType) j;
		}
	}
	switch(child->type) {
		case Photo:		child->color = 0.5f; break; // Green
		case Pred:		child->color = 1.0f; break; // Red
		case Move:		child->color = 0.7f; break; // Yellow
	}
	child->growthProb = output[N_CELL_TYPES];
}

int main() {
	Fluidix<> *fx = new Fluidix<>(&g);
	int setA = fx->createParticleSet(N);
	//fx->createGlobalArray(&g.toGrow, N * sizeof(int));
	//for(int i=0; i<N; i++)
	//	g.toGrow[i] = -1;

	fx->runEach(init(), setA);
	
	Particle *p = fx->getParticleArray(setA);
	for(int i=0; i<N_ORIGIN_CELLS; i++)
		initializeNewOrganism(fx, &p[i], g);
	fx->applyParticleArray(setA);
	
	for(int step=0; step < N_STEPS; step++) {
		fx->runPair(particlePair(), setA, setA, RANGE);
		fx->runEach(buoyancy(), setA);
		fx->runEach(boundary(), setA);
		fx->runEach(integrate(), setA);
		fx->runEach(age(), setA);
		
		// Grow cells that shold do that
		for(int i=0; i<N; i++) {
			if(p[i].toGrow != -1) {
				int parent = i;
				int child = p[i].toGrow;
				printf("toGrow! parent: %i, child: %i\n", parent, child);
				growCell(fx, &p[parent], &p[child]);
				//fx->addLink(
				//	g.organisms[p[i].organism].linkSet,
				//	setA, parent, setA, child
				//);
				p[i].toGrow = -1;
				fx->applyParticleArray(setA);
			}
		}

		if (step % 10 == 0) {
			printf("step %d\n", step);
			fx->outputFrame("output");
		}
	}
	//createCells(fx, make_xyz(0, 0, 0), L, setA);

	delete fx;
}
