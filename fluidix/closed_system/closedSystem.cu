#include "hip/hip_runtime.h"
#include "fluidix.h"
#include "../lib/genome.h"

#define DT 0.1f // integration time-step

#define BOX_MAX 20
#define BOX_MIN 1

#define W 100

using namespace std;

enum CellType {Blank, Sense, Move, Ballast};

struct Global {
	float grid_pack;
	int3 grid_num;
	xyz grid_origin;
	vector<Genome> organisms;
} g;

struct Particle {
	bool isCell;
	xyz r, v, f;
	float color;
	float radius;
	float alpha;
	int organism;
	Particle *origin;
	CellType type;
};

FUNC_EACH(init,
	p.r = (make_xyz_uniform() * W) + make_xyz(0 , W, 0);
	p.color = 0.7f;
	p.alpha = 0.5f;
	p.radius = 0.5f;
	p.isCell = false;
)

FUNC_EACH(integrate,
	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
	p.v *= 0.97f;
)

FUNC_EACH(gravity,
	p.f.y -= 9.81f;
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }

	if(p.isCell){
		if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
		if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	} else {
		if (p.r.y < 0) p.r.y = W;
	}
)

// Initialize new organism, not inheriting anything
void initializeNewOrganism(Particle *cell, Global &g){
	int inputs = 4;
	int outputs = 4;

	Genome genome(inputs, outputs);
	genome.printMathematica();

	g.organisms.push_back(genome);
	cell->organism	= g.organisms.size()-1;
	cell->alpha		= 1.0f;
	cell->radius		= 1.0f;
	cell->r			= make_xyz_uniform() * W;
	cell->isCell		= true;

	vector<float> input(inputs, 0.0f);
	vector<float> output = genome.getOutput(input);

	float max = output[0];
	for(int j=1; j<output.size(); j++) {
		if(output[j] > max) {
			max = output[j];
			cell->type = (CellType) j;
		}
	}
	switch(cell->type) {
		case Blank:		cell->color = 0.6f; break;
		case Sense:		cell->color = 0.3f; break;
		case Move:		cell->color = 1.0f; break;
		case Ballast:	cell->color = 0.0f; break;
	}
}

// Initialize organism, inheriting from parent
void initializeOffspring(Particle *parent, Particle *child, Global &g){
	Genome genome = g.organisms[parent->organism];
	genome.mutate();
	genome.printMathematica();

	g.organisms.push_back(genome);
	child->organism = g.organisms.size()-1;
}

void growCell(Particle *parent, Particle *child) {
	//	Copy constructor?
	*child = Particle(*parent);

	// Displace particles from each other
	xyz displacement = xyz_norm(make_xyz_uniform()) * parent->radius;		
	parent->r -= displacement;
	child->r  += displacement;

	Genome genome = g.organisms[parent->organism];

	xyz dr = parent->r - parent->origin->r;

	vector<float> input;
	input.push_back(dr.x);
	input.push_back(dr.y);
	input.push_back(dr.z);
	input.push_back(xyz_len(dr));

	vector<float> output = genome.getOutput(input);

	//printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
	//printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

	float max = output[0];
	for(int j=1; j<output.size(); j++) {
		if(output[j] > max) {
			max = output[j];
			child->type = (CellType) j;
		}
	}
	switch(child->type) {
		case Blank:		child->color = 0.6f; break;
		case Sense:		child->color = 0.3f; break;
		case Move:		child->color = 1.0f; break;
		case Ballast:	child->color = 0.0f; break;
	}
}

int main() {
	Fluidix<> *fx = new Fluidix<>(&g);
	int setA = fx->createParticleSet(10000);
	fx->runEach(init(), setA);
	
	Particle *p = fx->getParticleArray(setA);
	for(int i=0; i<10; i++) {
		initializeNewOrganism(&p[i], g);
		fx->applyParticleArray(setA);
	}
	
	for(int step=0; step < 10000; step++) {
		fx->runEach(integrate(), setA);
		fx->runEach(gravity(), setA);
		fx->runEach(boundary(), setA);
		fx->outputFrame("output");
	}
	//createCells(fx, make_xyz(0, 0, 0), L, setA);

	delete fx;
}
