#include "hip/hip_runtime.h"
#include "fluidix.h"
#include "../lib/genome.h"

#define DT 0.01f // integration time-step

#define BOX_MAX 20
#define BOX_MIN 1

#define W 100
#define N 10000
#define RANGE 3.0f

#define REPULSION_FORCE 150
#define ATTRACTION_FORCE 150

#define FLUID_DENSITY 1.0f
#define G -9.81f

using namespace std;

enum CellType {Photo, Pred, Blank};

struct Organism {
	Genome genome;
	int linkSet;

	Organism(Genome g): genome(g), linkSet(-1) {}
};

struct Global {
	vector<Organism> organisms;
} g;

struct Particle {
	bool isCell;
	xyz r, v, f;
	float color;
	float radius;
	float alpha;
	float density;
	int organism;
	int toGrow;
	Particle *origin;
	CellType type;
};

FUNC_EACH(init,
	p.r = (make_xyz_uniform() * W) + make_xyz(0 , W, 0);
	p.color = 0.7f;
	p.alpha = 0.3f;
	p.radius = 0.5f;
	p.density = FLUID_DENSITY * 2;
	p.isCell = false;
	p.type = Blank;
	p.toGrow = -1;
)

FUNC_EACH(integrate,
	p.v += p.f * DT;
	p.r += p.v * DT;
	p.f = make_xyz(0, 0, 0);
	p.v *= 0.97f;
)

FUNC_EACH(buoyancy,
	//if(p.type == Ballast)
	//	p.density += sin(p.signal);
	float volume = p.radius * p.radius * PI;
	p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)

// bouncing hard wall boundary condition
FUNC_EACH(boundary,
	if (p.r.x < 0) { p.v.x = 0.9f * (0 - p.r.x) / DT; p.r.x = 0; }
	if (p.r.x > W) { p.v.x = 0.9f * (W - p.r.x) / DT; p.r.x = W; }
	if (p.r.z < 0) { p.v.z = 0.9f * (0 - p.r.z) / DT; p.r.z = 0; }
	if (p.r.z > W) { p.v.z = 0.9f * (W - p.r.z) / DT; p.r.z = W; }

	if(p.isCell){
		if (p.r.y < 0) { p.v.y = 0.9f * (0 - p.r.y) / DT; p.r.y = 0; }
		if (p.r.y > W) { p.v.y = 0.9f * (W - p.r.y) / DT; p.r.y = W; }
	} else {
		if (p.r.y < 0) {
			p.r.x = rnd_uniform() * W;
			p.r.z = rnd_uniform() * W;
			p.r.y = W;
		}
	}
)

FUNC_PAIR(particlePair,
	float ratio = (dr - p1.radius - p2.radius) / (range - p1.radius - p2.radius);
	//float ratio = dr/range;
	float attraction = 0.0f;
	if((p1.isCell && p2.isCell) && (p1.organism == p2.organism))
		attraction = ATTRACTION_FORCE * ratio;
	xyz f = u * (REPULSION_FORCE * (1-ratio) - attraction);

	addVector(p1.f, f);
	addVector(p2.f, -f);

	if(p1.organism != p2.organism){
		if(p1.type == Photo && !p2.isCell) p1.toGrow = p2_index;
		else if(p2.type == Photo && !p1.isCell)	p2.toGrow = p1_index;
	
		else if(p1.type == Pred && p2.isCell) p1.toGrow = p2_index;
		else if(p2.type == Pred && p1.isCell) p2.toGrow = p1_index;
	}
/*
	if(p1.type == Sense) p1.signal += 0.1f;
	if(p2.type == Sense) p2.signal += 0.1f;

	if(p1.organism == p1.organism){
		float meanSignal = (p1.signal + p2.signal)/2;
		meanSignal = sin(meanSignal);
		p1.signal = p2.signal = meanSignal;
	}
*/
	//p1.color = p1.signal;
	//p2.color = p2.signal;
)

void setDefaultCellValues(Particle *cell) {
	cell->alpha		= 1.0f;
	cell->radius		= 1.0f;
	cell->density	= FLUID_DENSITY;
	cell->isCell		= true;
}

// Initialize new organism, not inheriting anything
void initializeNewOrganism(Fluidix<> *fx, Particle *cell, Global &g) {
	int inputs = 4;
	int outputs = 3;

	Organism o(Genome(inputs, outputs));
	o.linkSet = fx->createLinkSet();
	o.genome.mutate();
	o.genome.printMathematica();

	g.organisms.push_back(o);
	cell->organism	= g.organisms.size()-1;
	cell->r			= make_xyz_uniform() * W;
	cell->origin		= cell;
	setDefaultCellValues(cell);

	vector<float> input(inputs, 0.0f);
	vector<float> output = o.genome.getOutput(input);

	float max = output[0]; cell->type = (CellType) 0;
	for(int j=1; j<output.size(); j++) {
		if(output[j] > max) {
			max = output[j];
			cell->type = (CellType) j;
		}
	}
	switch(cell->type) {
		case Photo:		cell->color = 0.5f; break; // Green
		case Pred:		cell->color = 1.0f; break; // Red
		case Blank:		cell->color = 0.7f; break; // Yellow
		//case Sense:		cell->color = 0.3f; break; // Cyan
		//case Move:		cell->color = 1.0f; break; // Red
		//case Ballast:	cell->color = 0.0f; break; // Blue
	}
}

// Initialize organism, inheriting from parent
void initializeOffspring(Fluidix<> *fx, Particle *parent, Particle *child, Global &g) {
	Organism o = Organism(g.organisms[parent->organism]);
	o.linkSet = fx->createLinkSet();
	o.genome.mutate();
	o.genome.printMathematica();

	g.organisms.push_back(o);
	child->organism = g.organisms.size()-1;
}

void growCell(Fluidix<> *fx, Particle *parent, Particle *child) {
	normal_distribution<float> rndNormal(0.0f, 1.0f);
	
	//	Copy constructor
	*child = Particle(*parent);
	
	// Displace particles from each other
	xyz displacement = xyz_norm(
		make_xyz(
			rndNormal(rndGen),
			rndNormal(rndGen),
			rndNormal(rndGen)
		)) * parent->radius;
		
	parent->r -= displacement;
	child->r  += displacement;

	Organism o = g.organisms[parent->organism];

	xyz dr = child->r - child->origin->r;

	vector<float> input;
	input.push_back(dr.x);
	input.push_back(dr.y);
	input.push_back(dr.z);
	input.push_back(xyz_len(dr));

	vector<float> output = o.genome.getOutput(input);

	//printf("input: ");  for(float i : input)  printf("%.2f ",i); printf("\t");
	//printf("output: "); for(float o : output) printf("%.2f ",o); printf("\n");

	float max = output[0];
	for(int j=1; j<output.size(); j++) {
		if(output[j] > max) {
			max = output[j];
			child->type = (CellType) j;
		}
	}
	switch(child->type) {
		case Photo:		child->color = 0.5f; break; // Green
		case Pred:		child->color = 1.0f; break; // Red
		case Blank:		child->color = 0.7f; break; // Yellow
	}
}

int main() {
	Fluidix<> *fx = new Fluidix<>(&g);
	int setA = fx->createParticleSet(N);
	//fx->createGlobalArray(&g.toGrow, N * sizeof(int));
	//for(int i=0; i<N; i++)
	//	g.toGrow[i] = -1;

	fx->runEach(init(), setA);
	
	Particle *p = fx->getParticleArray(setA);
	for(int i=0; i<10; i++)
		initializeNewOrganism(fx, &p[i], g);
	fx->applyParticleArray(setA);
	
	for(int step=0; step < 100000; step++) {
		fx->runPair(particlePair(), setA, setA, RANGE);
		fx->runEach(buoyancy(), setA);
		fx->runEach(boundary(), setA);
		fx->runEach(integrate(), setA);
		
		for(int i=0; i<N; i++) {
			if(p[i].toGrow != -1) {
				int parent = i;
				int child = p[i].toGrow;
				printf("toGrow! parent: %i, child: %i\n", parent, child);
				growCell(fx, &p[parent], &p[child]);
				fx->addLink(
					g.organisms[p[i].organism].linkSet,
					setA, parent, setA, child
				);
				p[i].toGrow = -1;
				fx->applyParticleArray(setA);
			}
		}
		if (step % 10 == 0) {
			printf("step %d\n", step);
			fx->outputFrame("output");
		}
	}
	//createCells(fx, make_xyz(0, 0, 0), L, setA);

	delete fx;
}
