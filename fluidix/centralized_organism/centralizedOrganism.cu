#include "hip/hip_runtime.h"
//#include "fluidix.h"
#define PARTICLE_BITS 28
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/genome.h"
#include "../lib/nerveSystem.h"
#include <queue>
#include <ppl.h>
#include <concurrent_queue.h>
#include <unordered_map>

#define DT 0.01f // integration time-step
#define PELLET_LIFETIME 5.0f

#define W make_int3(300, 100, 300)
#define N 300000
#define N_ORIGIN_ORGANISM 1000
#define BUFFER_SIZE 1000
#define N_STEPS 1000000
#define INITIAL_ORGANISM_DIMENSIONS make_int3(1, 1, 1)

//Inputs x,y,z,d:
#define N_INPUTS 4

#define RANGE 10.0f
#define MOVE_FACTOR 100

#define REPULSION_FORCE 300
#define SPRING_K 150.0f // spring constant
#define WALL 100.0f // repulsive wall force

#define CELL_INITIAL_ENERGY 2.0f
#define CELL_MIN_ENERGY 1.0f
#define PELLET_MIN_ENERGY 0.01f
#define DIVISION_ENERGY (CELL_MIN_ENERGY * 2.5f)
#define ENERGY_PARTICLE_ENERGY 0.5f
#define MAX_CELL_DIVISIONS 100
#define CELL_EXISTENCE_THRESHOLD 0.0f

#define CELL_METABOLISM 0.02f
#define CELL_DECAY_RATE 0.01f

#define FLUID_DENSITY 1.0f
#define G -9.81f

#define GREEN   0.5f
#define RED     1.0f
#define YELLOW  0.7f
#define BLUE    0.0f
#define CYAN    0.3f
#define ORANGE  0.8f

#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;


enum CellType {
    Photo, Digest, Sting, Vascular, Fat, Sense, Ballast, Egg,
    N_CELL_TYPES
};
enum ParticleType {
    Cell, Energy, Pellet, Buffer,
    N_PARTICLE_TYPES
};
enum Neigbour { Front, Right, Up, Back, Left, Down };

int currGenomeIndex;

struct Organism {
    Genome genome;
    NerveSystem nervSystem;
    vector<int> cells;
};

typedef std::unordered_map<int, Organism> organismMap;

struct Global {
    int nEggs;
    float *nerveWeights;
} g;

struct Particle {
    ParticleType particleType;
    xyz r, v, f;
    float color;
    float radius;
    float alpha;
    float density;
    float energy;
    float energyIn;
    float energyOut;
    float maxEnergy;
    int nDivisions;
    float signal;
    int organism;
    bool toBuffer;
    bool toReproduce;
    int links[6];
    CellType type;
};

#define turnIntoPellet(p) {         \
    p.particleType = Pellet;        \
    p.energy = PELLET_LIFETIME;     \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.5f;                 \
    p.organism = -1;                \
}                                   \

#define turnIntoEnergy(p) {         \
    p.particleType = Energy;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W.x,          \
        rnd_uniform()*W.y + W.y,    \
        rnd_uniform()*W.z           \
    );                              \
    p.color = 0.7f;                 \
    p.energy = ENERGY_PARTICLE_ENERGY; \
    p.signal = 0.0f;                \
    p.alpha = 0.3f;                 \
    p.radius = 0.5f;                \
    p.density = 10.0f; 	  				\
}

#define turnIntoBuffer(p) {         \
    p.particleType = Buffer;        \
    p.r = make_xyz(                 \
        rnd_uniform()*W.x,          \
        rnd_uniform()*W.y - W.y,    \
        rnd_uniform()*W.z           \
    );                              \
    p.density = FLUID_DENSITY * 2;  \
    p.alpha = 0.1f;                 \
    p.color = 0.5f;                 \
    p.radius = 1.0f;                \
    p.organism = -1;                \
}

FUNC_EACH(init,
    turnIntoEnergy(p);
)

FUNC_EACH(integrate,
    p.v += p.f * DT; //Mass?
    p.r += p.v * DT;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
)

FUNC_EACH(handleEnergy,
    switch (p.particleType) {
    case Cell:
        p.energy -= CELL_METABOLISM * DT;
        if (p.energy < CELL_MIN_ENERGY )
            turnIntoPellet(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Pellet:
        p.energy -= CELL_DECAY_RATE * DT;
        if (p.energy <= PELLET_MIN_ENERGY)
            p.toBuffer = true;
        break;
    }
    //p.color = mapf(p.energy, 0.0f, 3.0f, 0.0f, 1.0f);
)

FUNC_EACH(buoyancy,
    //if (p.particleType == Cell && p.type == Ballast)
    //    p.density = clamp(p.density + p.signal, 0.5f, 2.0f);
    float volume = p.radius * p.radius * PI;
    p.f.y += (p.density - FLUID_DENSITY) * G * volume;
)
/*
FUNC_EACH(moveParticle,
    if (p.particleType == Cell && p.type == Motor) {
        xyz f = make_xyz(0, 0, 0);
        if (p.links[Left]  < 0)  f.x += 1;
        if (p.links[Up]    < 0)  f.y += 1;
        if (p.links[Back]  < 0)  f.z += 1;
        if (p.links[Right] < 0)  f.x -= 1;
        if (p.links[Down]  < 0)  f.y -= 1;
        if (p.links[Front] < 0)  f.z -= 1;
        f = xyz_norm(f) * MOVE_FACTOR * clamp(p.signal, -1.0f, 1.0f);

        addVector(p.f, f);
    }
)
*/

FUNC_EACH(reproduction,
    if (p.particleType == Cell && p.type == Egg)
        if (p.energy >= p.maxEnergy){
            p.toReproduce = true;
            printf("toReproduce!!!\n");
        }
)


// bouncing hard wall boundary condition
FUNC_EACH(boundary,
    if (p.particleType == Cell && p.type == Egg)
        addInteger(g.nEggs, 1);

    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        //printf("Weird particle!!! type (%i:%i)\n", p.particleType, p.type);
        turnIntoBuffer(p);
        p.toBuffer = true;
    } if (p.particleType != Buffer) {
        if (p.r.x < 0)   p.f.x += WALL * (0 - p.r.x);
        if (p.r.x > W.x) p.f.x += WALL * (W.x - p.r.x);
        if (p.r.z < 0)   p.f.z += WALL * (0 - p.r.z);
        if (p.r.z > W.x) p.f.z += WALL * (W.x - p.r.z);

        if (p.particleType == Energy){
            if (p.r.y < 0) {
                p.toBuffer = true;
            }
        }
        else {
            if (p.r.y < 0)   { p.v.y = 0.9f * (0 - p.r.y)   / DT; p.r.y = 0; }
            if (p.r.y > W.y) { p.v.y = 0.9f * (W.y - p.r.y) / DT; p.r.y = W.y; }
        }
    }
    else if (p.r.y < -2 * W.y) p.r.y += W.y;
)

#define consumeParticle(a, b) {             \
    addFloat(a.energy, b.energy);    \
    b.energy = 0;                           \
    b.toBuffer = true;                      \
}

FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        xyz f = u * maxf(
            (REPULSION_FORCE * (1 - dr / (p1.radius + p2.radius))),
            0
        );
        if (p1.particleType == Cell &&
            p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                bool neighbours = false;
                for (int n = 0; n < 6; n++) {
                    if (p1.links[n] == p2_index || p2.links[n] == p1_index) {
                        neighbours = true;
                        break;
                    }
                }
                if (neighbours) {
                    //Spring force between neighbours
                    f = -u * ((dr - (p1.radius + p2.radius)/2) * SPRING_K);

                    //Signalling between cells of same organism
                    //float meanSignal = (p1.signal + p2.signal) / 2;
                    //p1.signal = p2.signal = meanSignal;

                    //Energy transmission
                    float p1Surplus = maxf(p1.energy - CELL_MIN_ENERGY, 0);
                    float p2Surplus = maxf(p2.energy - CELL_MIN_ENERGY, 0);
                    transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                    transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
                }
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoPellet(p2);
                    transmitFloat(p2.energy, p1.energy, 0.05f);
                }
                if (p2.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoPellet(p2);
                    transmitFloat(p1.energy, p2.energy, 0.05f);
                }
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Pellet)
                ) consumeParticle(p1, p2)
        }
        //If p2 is a cell
        else if (p2.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Pellet)
                ) consumeParticle(p2, p1)
        }

        if (p1.particleType == Cell && p1.type == Sense) addFloat(p1.signal, 1.0f);
        if (p2.particleType == Cell && p2.type == Sense) addFloat(p1.signal, 1.0f);

        addVector(p1.f, f);
        addVector(p2.f, -f);

        //p1.color = p1.signal;
        //p2.color = p2.signal;
    }
)

FUNC_SURFACE(collideGround,
if (p.particleType != Energy){
    if (dr > 1) dr = 1;
    p.f += WALL * u * dr;
    /*xyz f = u * 50 * dr;
    p.f += f;
    
    if (p3) f /= 3;
    else if (p2) f /= 2;
    if (p1) addVector(p1->f, -f);
    if (p2) addVector(p2->f, -f);
    if (p3) addVector(p3->f, -f);
    */
}
)

void setDefaultCellValues(Particle *cell) {
    cell->alpha = 1.0f;
    cell->radius = 1.0f;
    cell->energy = CELL_INITIAL_ENERGY;
    cell->density = FLUID_DENSITY * 1.10f;
    cell->particleType = Cell;
}

bool applyPhenotype(vector<float> output, Particle *cell) {
    // If cell should not exist, return
    if (output[N_CELL_TYPES + 1] < CELL_EXISTENCE_THRESHOLD)
        return false;

    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    switch (cell->type) {
    case Photo:
        cell->color = GREEN;
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 3.0f;
        break;
    case Digest:
        cell->color = RED;
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 3.0f;
        break;
    case Fat:
        cell->color = YELLOW;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 10.0f;
        break;
    case Sense:
        cell->color = BLUE;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Ballast:
        cell->color = CYAN;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Egg:
        cell->color = ORANGE;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 1000.0f;
        break;
    case Vascular:
        cell->color = 0.2;
        cell->energyIn = 1.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 1.0f;
        break;
    case Sting:
        cell->color = nanf("");
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
    }
    cell->nDivisions = output[N_CELL_TYPES] * MAX_CELL_DIVISIONS;
    return true;
}

void disconnectCell(Particle *p, int cell, int code) {
    for (int i = 0; i < 6; i++) {
        if (p[cell].links[i] >= 0)
            p[p[cell].links[i]].links[(i + 3) % 6] = code;
    }
    turnIntoBuffer(p[cell]);
}
void emptyCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -1);
}
void deadCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -2);
}

int getIdxFromCoord(int x, int y, int z, int3 br)
{
    x += br.x; y += br.y; z += br.z;
    //int lX = 2 * br.z + 1;
    int lY = 2 * br.y + 1;
    int lZ = 2 * br.z + 1;
    return x*lY*lZ + y*lZ + z;
}
#define iFromCoord(x,y,z) cellBuff.at(getIdxFromCoord(x,y,z,br))

// Initialize new organism
int spawnOrganism(
    Fluidix<> *fx, int particleSet,
    xyz origin, concurrent_queue<int> *particleBuffer,
    Particle *p, Genome *parentGenome, organismMap *organisms)
{
    Genome genome(*parentGenome);
    genome.mutate();
    
    int nParticlesNeeded = genome.getMaxCellsReq();
    if (nParticlesNeeded > particleBuffer->unsafe_size()) {
        printf("Not enought particles in buffer\n");
        return -1;
    }
    vector<int> cellBuff;
    while (nParticlesNeeded) {
        int particle;
        if (particleBuffer->try_pop(particle)) {
            cellBuff.push_back(particle);
            nParticlesNeeded--;
        }
    }

    int3 br = genome.getBoundingRadius();

    int organismID = currGenomeIndex++;
    vector<int> removedCells;
    vector<int> addedCells;
    
    int nSensors = 0;

    for (int x = -br.x; x <= br.x; x++)
    for (int y = -br.y; y <= br.y; y++)
    for (int z = -br.z; z <= br.z; z++) {
        Particle *cell = &p[iFromCoord(x, y, z)];
        cell->organism = organismID;
        cell->r = origin + make_xyz(x, y, z);
        cell->toReproduce = false;
        cell->energy = CELL_INITIAL_ENERGY;
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x,y,z)));

        vector<float> output = genome.getOutput(input);

        if (applyPhenotype(output, cell)) {
            cell->links[Left] = x + 1 < br.x ? iFromCoord(x + 1, y, z) : -1;
            cell->links[Up] = y + 1 < br.y ? iFromCoord(x, y + 1, z) : -1;
            cell->links[Back] = z + 1 < br.z ? iFromCoord(x, y, z + 1) : -1;
            cell->links[Right] = x - 1 >= 0 ? iFromCoord(x - 1, y, z) : -1;
            cell->links[Down] = y - 1 >= 0 ? iFromCoord(x, y - 1, z) : -1;
            cell->links[Front] = z - 1 >= 0 ? iFromCoord(x, y, z - 1) : -1;

            if (cell->type == Sense)
                nSensors++;
            addedCells.push_back(iFromCoord(x, y, z));
        }
        else
            removedCells.push_back(iFromCoord(x, y, z));
    }
    for (int i : removedCells)
        emptyCellPos(p, i);

    NerveSystem nervSys(nSensors, 3);
    Organism organism = { genome, nervSys, addedCells };
    organisms->emplace(organismID, organism);

    return organismID;
}

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

int generateTerrain(Fluidix<> *fx){
    exponential_distribution<float> rndUniform(1);

    int terrDimX = 10;
    int terrDimZ = 10;

    int nParticles = (terrDimX*terrDimZ * 2);

    int meshParticles = fx->createParticleSet(nParticles);
    int meshLinks     = fx->createLinkSet();
    Particle *mesh    = fx->getParticleArray(meshParticles);

    float dx = W.x / (terrDimX-1);
    float dz = W.z / (terrDimZ-1);

    for (int x = 0; x < terrDimX; x++)
    for (int z = 0; z < terrDimX; z++){
        int i = x*terrDimZ + z;
        mesh[i].r = make_xyz(
            x*dx,
            rndUniform(rndGen) * 10 + 10,
            z*dz
        );
        mesh[i + nParticles/2].r = make_xyz(
            x*dx,
            0,
            z*dz
        );

        //Link terrain particles together:
        int s = (x - 1)*terrDimZ + z;
        int w = x*terrDimZ + (z - 1);
        int sw = (x - 1)*terrDimZ + (z - 1);

        if ((x - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, s);
        if ((z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, w);
        if ((x - 1) >= 0 && (z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, sw);

        if ((x - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, s + (nParticles / 2));
        if ((z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, w + (nParticles / 2));
        if ((x - 1) >= 0 && (z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, sw + (nParticles / 2));

        if (x % (terrDimX - 1) == 0 || z % (terrDimZ - 1) == 0){
            fx->addLink(meshLinks, meshParticles, i, meshParticles, i + (nParticles / 2));
            //if (x > 0 && z > 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, s + (nParticles / 2));
        }
        fx->applyParticleArray(meshParticles);
        fx->outputFrame("output");
    }
    return meshLinks;
}

int main() {
    Fluidix<> *fx = new Fluidix<>(&g);
    int pSet = fx->createParticleSet(N);

    int terrain = generateTerrain(fx);

    currGenomeIndex = 0;
    g.nEggs = 0;
    organismMap organisms;

    fx->runEach(init(), pSet);
    concurrent_queue<int> particleBuffer;
    Particle *p = fx->getParticleArray(pSet);

    int initialBufferSize =
        (INITIAL_ORGANISM_DIMENSIONS.x * 2 + 1) *
        (INITIAL_ORGANISM_DIMENSIONS.y * 2 + 1) *
        (INITIAL_ORGANISM_DIMENSIONS.z * 2 + 1) *
        N_ORIGIN_ORGANISM +
        BUFFER_SIZE;

    for (int i = 0; i < initialBufferSize; i++) {
        turnIntoBuffer(p[i]);
        p[i].r.y -= W.y;
        particleBuffer.push(i);
    }

    for (int i = 0; i < N_ORIGIN_ORGANISM; i++) {
        int3 gridDim = INITIAL_ORGANISM_DIMENSIONS; //genomes[iOrigin].gridDim;

        // Define number of in- and outputs
        int inputs = N_INPUTS;              // X, Y, Z, Dist
        int nonCelltypeOutputs = 2;         // Growth prob
        int outputs = N_CELL_TYPES + nonCelltypeOutputs;
        Genome g = Genome(inputs, outputs, gridDim);
        g.mutate(); g.mutate(); g.mutate(); g.mutate(); g.mutate();
        xyz origin = make_xyz_uniform() * int3_to_xyz(W);
        origin.y /= 2;
        origin.y += W.y / 2;

        spawnOrganism(fx, pSet, origin, &particleBuffer, p, &g, &organisms);
    }
    fx->applyParticleArray(pSet);

    for (int step = 0; step < N_STEPS; step++) {
        g.nEggs = 0;
        fx->runEach(boundary(), pSet);
        fx->runSurface(collideGround(), terrain, pSet,RANGE);
        fx->runPair(particlePair(), pSet, pSet, RANGE);


        p = fx->getParticleArray(pSet);
        vector<int> organismsToRemove;
        for (auto& iOrg : organisms) {
            Organism *o = &iOrg.second;
            vector<float> inputs;
            int nLiving = 0;
            int nDead = 0;
            for (int i : o->cells){
                if (p[i].particleType == Cell){
                    if (p[i].type == Sense)
                        inputs.push_back(p[i].signal);
                    nLiving++;
                }
                else
                nDead++;
            }
            if (nDead > nLiving) {
                for (int i : o->cells)
                if (p[i].particleType == Cell)
                    turnIntoPellet(p[i]);
                organismsToRemove.push_back(iOrg.first);
                continue;
            }
            vector<float> output = o->nervSystem.getOutput(inputs);

            xyz f = make_xyz(output[0], output[1], output[2]);
            for (int i : o->cells) {
                p[i].f += f * MOVE_FACTOR;
                p[i].signal *= 0.5f;
            }
        }
        for (int i : organismsToRemove)
            organisms.erase(i);
        fx->applyParticleArray(pSet);

        //fx->runEach(moveParticle(), pSet);
        fx->runEach(buoyancy(), pSet);
        fx->runEach(handleEnergy(), pSet);
        fx->runEach(integrate(), pSet);
        //parallel_for (int(0), N, [&](int i)
        for (int i = 0; i<N; i++)
        {
            if (p[i].toBuffer) {
                if (particleBuffer.unsafe_size() > BUFFER_SIZE) {
                    turnIntoEnergy(p[i]);                  
                } else {
                    turnIntoBuffer(p[i]);
                    particleBuffer.push(i);
                }
                p[i].toBuffer = false;
                fx->applyParticleArray(pSet);
            }
            if (p[i].particleType == Cell) {
                // Create offspring:
                if (p[i].type == Egg) { //&& p[i].toReproduce) {
                    //printf("Has %.2f, needs %.2f - ", p[i].energy, genomes[i].getMaxCellsReq() * CELL_INITIAL_ENERGY);
                    Genome g = organisms.at(p[i].organism).genome;
                    if (g.getMaxCellsReq() * CELL_INITIAL_ENERGY <= p[i].energy) {
                        g.mutate();
                        int orgID = spawnOrganism(fx, pSet, p[i].r, &particleBuffer, p, &g, &organisms);
                        organisms.at(orgID).nervSystem.mutate();
                        p[i].toReproduce = false;
                        p[i].toBuffer = true;
                        fx->applyParticleArray(pSet);
                    }
                }
            }
        }//);

        //fx->applyParticleArray(pSet);

        if (step % 10 == 0) {
            printf("nEggs: %i\t", g.nEggs);
            printf("currgenomeIndex: %i\t", currGenomeIndex);
            printf("step %d\n", step);
            fx->outputFrame("output");
        }

        if (!g.nEggs) {
            break;
        }
    }
    delete fx;

    system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
}
