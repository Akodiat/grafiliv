#include "hip/hip_runtime.h"
//#include "fluidix.h"
#define PARTICLE_BITS 28
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/structures.h"
#include "../lib/genome.h"
#include "../lib/nerveSystem.h"
#include "../lib/linearAlgebra.h"
#include "../lib/io.h"
#include <queue>
#include <ppl.h>

// Transfer amount f from a to b
#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}

#define sphereVolume(r) (4.0f / 3.0f) * r * r * r * PI;

// Check if particle position is defined correctly
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;

int currGenomeIndex;
int step;

// Turn a particle into Detritus (dead cell) type
// Previous energy is preserved
#define turnIntoDetritus(p) {         \
    p.particleType = Detritus;        \
    p.density = g.fluidDensity * 2; \
    p.organism = -1;                \
}                                   \

// Turn a particle into energy type
#define resetEnergy(p) {               \
    p.particleType = Energy;           \
    p.r.x = rnd_uniform() * g.w.x;     \
    p.r.y += g.w.y;                    \
    p.r.z = rnd_uniform() * g.w.z;     \
    p.energy = g.energyParticleEnergy; \
    p.signal = 0.0f;                   \
    p.radius = g.energyParticleRadius; \
    p.density = 10.0f;                 \
}

// Turn a pasticle into buffer type
// Place below arena
#define turnIntoBuffer(p) {            \
    p.particleType = Buffer;           \
    p.r = make_xyz(                    \
        rnd_uniform() * g.w.x,         \
        rnd_uniform() * g.w.y - g.w.y, \
        rnd_uniform() * g.w.z          \
    );                                 \
    p.density = g.fluidDensity;        \
    p.color = 0.5f;                    \
    p.radius = 1.0f;                   \
    p.organism = -1;                   \
}

// Update position r of particles given
// velocity v and force f
FUNC_EACH(integrate,
    p.v += p.f * g.dt;
    p.r += p.v * g.dt;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
    )

// Decrease energy through metabolism in
// cells and decay in detritus.
// cell --> detritus --> buffer
FUNC_EACH(handleEnergy,
    p.color = p.energy * 0.2f;
    switch (p.particleType) {
    case Cell:
        //printf("Cell energy: %.2f\n", p.energy);
        p.energy -= p.metabolism * g.dt;
        if (p.energy < g.minCellEnergy)
            turnIntoDetritus(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Detritus:
        p.energy -= g.cellDecayRate * g.dt;
        if (p.energy <= g.minDetritusEnergy)
            p.toBuffer = true;
        break;
    case Energy:
        if (p.energy <= 0)
            p.toBuffer = true;
    }
)

// Particles float depending on their density
FUNC_EACH(buoyancy,
    float volume = sphereVolume(p.radius);
    float weight = p.density * volume;
    float displacedFluidWeight = g.fluidDensity * volume;
    float apparentWeight = weight - displacedFluidWeight;
    p.f.y += apparentWeight * g.gravity;
)

FUNC_EACH(countParticles,
    switch (p.particleType) {
    case Cell:
        addInteger(g.nCells, 1); break;
    case Detritus:
        addInteger(g.nDetritus, 1); break;
    case Buffer:
        addInteger(g.nBuffer, 1); break;
    case Energy:
        addInteger(g.nEnergy, 1); break;
    }
)

// Periodic boundary conditions
FUNC_EACH(boundary,
    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        printf("Wierd particle at (%.2f, %.2f, %.2f) type=%i\n", p.r.x, p.r.y, p.r.z, p.particleType);
        if (p.particleType == Energy) {
            p.r.y = rnd_uniform() * g.w.y;
            resetEnergy(p);
        }
        else {
            turnIntoBuffer(p);
            p.toBuffer = true;
        }
    } 
    if (p.particleType != Buffer) {
        if (p.r.x < 0)     p.r.x = g.w.x;
        if (p.r.x > g.w.x) p.r.x = 0;
        if (p.r.z < 0)     p.r.z = g.w.z;
        if (p.r.z > g.w.x) p.r.z = 0;

        if (p.particleType == Energy) {
            if (p.r.y < 0) {
                resetEnergy(p);
            }
        }
        else {
            if (p.r.y < 0) {
				p.v.y = 0.9f * (0 - p.r.y) / g.dt;
				p.r.y = 0;
			}
            if (p.r.y > g.w.y) {
				p.v.y = 0.9f * (g.w.y - p.r.y) / g.dt;
				p.r.y = g.w.y;
			}
        }
    }
    else if (p.r.y < -2 * g.w.y) p.r.y += g.w.y;
)

// Let particle a eat of particle b until full
#define getEnergyNeed(a,b) maxf(minf((a.maxEnergy - a.energy), b.energy),0)

// For each particle within a predifined distance
FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        xyz f = u * maxf(
            (g.repulsiveForce * (1 - dr / (p1.radius + p2.radius))),
            0
        );
        if (p1.particleType == Cell && p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                bool neighbours = false;
                for (int n = 0; n < 6; n++) {
                    if (p1.links[n] == p2_index || p2.links[n] == p1_index) {
                        neighbours = true;
                        break;
                    }
                }
                if (neighbours) {
                    //Spring force between neighbours
                    f = -u * ((dr - (p1.radius + p2.radius)/2) * g.springForce);

                    float p1Surplus = maxf(p1.energy - g.minCellEnergy, 0);
                    float p2Surplus = maxf(p2.energy - g.minCellEnergy, 0);
                    transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                    transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
                }
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoDetritus(p2);
                    //transmitFloat(p2.energy, p1.energy, 0.5f);
                    float need = getEnergyNeed(p1, p2);
                    transmitFloat(p2.energy, p1.energy, need);
                }
                if (p2.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoDetritus(p2);
                    //transmitFloat(p1.energy, p2.energy, 0.5f);
                    float need = getEnergyNeed(p2, p1);
                    transmitFloat(p1.energy, p2.energy, need);
                }
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Detritus)
                )
            {
                float need = getEnergyNeed(p1, p2);
                transmitFloat(p2.energy, p1.energy, need);
            }
        }
        //If p2 is a cell
        else if (p2.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Detritus)
                )
            {
                float need = getEnergyNeed(p2, p1);
                transmitFloat(p1.energy, p2.energy, need);
            }
        }

        if (p1.particleType == Cell && p1.type == Sense) addFloat(p1.signal, 1.0f/dr);
        if (p2.particleType == Cell && p2.type == Sense) addFloat(p1.signal, 1.0f/dr);

        addVector(p1.f, f);
        addVector(p2.f, -f);
    }
)

// Collision with terrain
FUNC_SURFACE(collideGround,
	if (p.particleType != Energy){
		if (dr > 1) dr = 1;
		p.f += g.groundRepulsiveForce * u * dr;
	}
)

// Initialize particle as cell
void setDefaultCellValues(Particle *cell) {
    //cell->radius = 1.0f;
    cell->energy = g.initialCellEnergy;
    //cell->density = g.fluidDensity * 1.10f;
    cell->particleType = Cell;
}

// Given a phenotype network output, apply it to the cell
bool applyPhenotype(vector<float> output, Particle *cell) {
    // If cell should not exist, return
    if (output[N_CELL_TYPES] < g.cellExistenceThreshold)
        return false;
    float radius = output[N_CELL_TYPES + 1];
    cell->radius = clamp((radius/2) + 0.5f, 0.5f, 1.0f);
    //if (cell->radius < 0) cell->radius = 0.5f;
    //printf("Radius mapped from %.2f to %.2f\n", radius, cell->radius);
    //float volume = sphereVolume(cell->radius);
    //float mass = 1.0f;

    cell->density = g.fluidDensity * 1.1f; //mass/volume;

    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    // If no outputs had a positive value, return
    if (max <= 0) {
        return false;
    }
    switch (cell->type) {
    case Photo:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.5f;
        cell->maxEnergy = 10.0f;
        break;
    case Digest:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.5f;
        cell->maxEnergy = 10.0f;
        break;
    case Fat:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 50.0f;
        break;
    case Sense:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
        break;
    case Egg:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 1000.0f;
        break;
    case Vascular:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 3.0f;
        break;
    case Sting:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.5f;
        cell->maxEnergy = 10.0f;
        break;
    case Buoyancy:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
        cell->density = g.fluidDensity * 0.01f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
    }

    return true;
}

// Remove cell links from and to cell
void disconnectCell(Particle *p, int cell, int code) {
    for (int i = 0; i < 6; i++) {
        if (p[cell].links[i] >= 0)
            p[p[cell].links[i]].links[(i + 3) % 6] = code;
    }
    p[cell].toBuffer = true;
}
void emptyCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -1);
}
void deadCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -2);
}

// Helper function to get the 1-dimensional index
// given x,y,z and box size br
int getIdxFromCoord(int x, int y, int z, int3 br)
{
    x += br.x; y += br.y; z += br.z;
    int lY = 2 * br.y + 1;
    int lZ = 2 * br.z + 1;
    return x*lY*lZ + y*lZ + z;
}
#define iFromCoord(x,y,z) cellBuff.at(getIdxFromCoord(x,y,z,br))

// Create cells of an organism given a genome and a nervous system
pair<int, vector<int>> createCellsFromGenotype(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, Genome *genome, NerveSystem *nerveSys, OrganismMap *organisms)
{
    int nParticlesNeeded = genome->getMaxCellsReq();
    if (nParticlesNeeded > particleBuffer->size()) {
        cerr << "Not enought particles in buffer\n" << endl;
    }
    vector<int> cellBuff;
/*  
    while (nParticlesNeeded) {
        int particle;
        if (particleBuffer->try_pop(particle)) {
            cellBuff.push_back(particle);
            nParticlesNeeded--;
        }
        else
            printf("Failed to retrive from buffer, trying again\n");
    }
*/
    while (nParticlesNeeded--) {
        int particle = particleBuffer->front();
        particleBuffer->pop();
        cellBuff.push_back(particle); 
    }

    int3 br = genome->getBoundingRadius();

    int organismID = currGenomeIndex++;
    vector<int> removedCells;
    vector<int> addedCells;

    int nSensors = 0;

    for (int x = -br.x; x <= br.x; x++)
    for (int y = -br.y; y <= br.y; y++)
    for (int z = -br.z; z <= br.z; z++) {
        Particle *cell = &p[iFromCoord(x, y, z)];
        cell->organism = organismID;
        cell->r = origin + make_xyz(x, y, z);
        cell->energy = g.initialCellEnergy;
        cell->metabolism = g.cellMetabolism +
            g.nerveCost * nerveSys->getSize();
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x, y, z)));

        vector<float> output = genome->getOutput(input);

        if (applyPhenotype(output, cell)) {
            cell->links[Left] = x + 1 < br.x ? iFromCoord(x + 1, y, z) : -1;
            cell->links[Up] = y + 1 < br.y ? iFromCoord(x, y + 1, z) : -1;
            cell->links[Back] = z + 1 < br.z ? iFromCoord(x, y, z + 1) : -1;
            cell->links[Right] = x - 1 >= 0 ? iFromCoord(x - 1, y, z) : -1;
            cell->links[Down] = y - 1 >= 0 ? iFromCoord(x, y - 1, z) : -1;
            cell->links[Front] = z - 1 >= 0 ? iFromCoord(x, y, z - 1) : -1;

            if (cell->type == Sense)
                nSensors++;

            float volume = cell->radius * cell->radius * cell->radius * PI * 4 / 3;
            cell->metabolism += volume * 0.05f;
            addedCells.push_back(iFromCoord(x, y, z));
        }
        else
            removedCells.push_back(iFromCoord(x, y, z));
    }
    for (int i : removedCells)
        emptyCellPos(p, i);

    nerveSys->updateInputs(nSensors);

    return pair<int, vector<int>>(organismID, addedCells);
}

//Initialize new organism (without parent)
int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, Genome genome, NerveSystem nerveSys, OrganismMap *organisms)
{
    pair<int, vector<int>> o = createCellsFromGenotype(
        origin, particleBuffer, p, &genome, &nerveSys, organisms
    );
    int organismID    = o.first;
    vector<int> cells = o.second;

    Organism organism = { genome, nerveSys, cells, -1, 100 };

    //Add organism to organism map
    organisms->emplace(organismID, organism);

    //Output organism to disk
    outputOrganism(&organism, organismID, step);

    return organismID;
}

// Initialize new organism from parent
int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, int parent, OrganismMap *organisms)
{
	Genome genome;
    NerveSystem nerveSys;

	if(parent == -1) {
		int3 gridDim = make_int3(1,1,1);

		// Define number of in- and outputs
		int inputs = g.nGenomeInputs;       // X, Y, Z, Dist
		int nonCelltypeOutputs = 2;         // Cell existence, cell radius
		int outputs = N_CELL_TYPES + nonCelltypeOutputs;
		genome = Genome(inputs, outputs, gridDim);

        int nerveOutputs = 3;
        nerveSys = NerveSystem(nerveOutputs);
	}
	else {
		genome = Genome(organisms->at(parent).genome);
        nerveSys = NerveSystem(organisms->at(parent).nerveSystem);
	}

    genome.mutate();
    nerveSys.mutate();

    pair<int, vector<int>> o = createCellsFromGenotype(
        origin, particleBuffer, p, &genome, &nerveSys, organisms
        );
    int organismID    = o.first;
    vector<int> cells = o.second;

    Organism organism = { genome, nerveSys, cells, parent, 100 };

    //Add organism to organism map
    organisms->emplace(organismID, organism);

    //Output organism to disk
    outputOrganism(&organism, organismID, step);

    return organismID;
}

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

// Initialize a random organism
int initializeOrganism(ParticleBuffer *particleBuffer, Particle *p, OrganismMap *organisms)
{
    xyz origin = make_xyz_uniform() * int3_to_xyz(g.w);
    origin.y /= 2;
    origin.y += g.w.y / 2;

    return spawnOrganism(origin, particleBuffer, p, -1, organisms);
}

Matrix3 getTransform(xyz front, xyz right, xyz up, xyz back, xyz left, xyz down) {
    return Matrix3(
        xyz_norm(right - left),
        xyz_norm(up - down),
        xyz_norm(front - back)
    );
}

int main() {
    // Create Fluidix library object
    Fluidix<> *fx = new Fluidix<>(&g);

    // Load configuration file
    g = loadConfig("conf.txt");

    // Create a particle set with number of particles
    // equal to g.nParticles
    int pSet = fx->createParticleSet(g.nParticles);

    currGenomeIndex = 0;
    OrganismMap organisms;

    // Initialize buffer
    ParticleBuffer particleBuffer;

    Particle *p = fx->getParticleArray(pSet);

    int i = 0;
    int neededEnergy = g.energyParticleCount;
      
    // Initialize energy particles
    while(neededEnergy--) {
        p[i].r.y = rnd_uniform() * g.w.y;
        resetEnergy(p[i]);
        i++;
    }

    // Turn the rest of the particles into buffer
    while (i < g.nParticles) {
        turnIntoBuffer(p[i]);
        particleBuffer.push(i);
        i++;
    }

    //for (int i = 0; i < 1000; i++)
    loadOrg("initOrg.json", &particleBuffer, p, &organisms);

    fx->applyParticleArray(pSet);

    FILE *out = fopen("countCells.csv", "w");
    fprintf(out, "nDetritus,nBuffer,nEnergy,nCells\n");

    step = 0;
    while(step++ < g.nSteps) {
        fx->runEach(boundary(), pSet);
        fx->runPair(particlePair(), pSet, pSet, g.interactionRange);

        p = fx->getParticleArray(pSet);
        vector<int> organismsToRemove;
        for (auto& iOrg : organisms) {
            Organism *o = &iOrg.second;
            o->health -= g.dt;

            vector<float> inputs;
            vector<int> eggs;
            int nLiving = 0;
            int nDead = 0;
            for (int i : o->cells){
                if (p[i].particleType == Cell){
                    if (p[i].type == Sense)
                        inputs.push_back(p[i].signal);
                    if (p[i].type == Egg)
                        eggs.push_back(i);
                    nLiving++;
                }
                else
                nDead++;
            }
            if (o->health <= 0 || nDead > nLiving) {
                for (int i : o->cells)
                if (p[i].particleType == Cell)
                    turnIntoDetritus(p[i]);
                organismsToRemove.push_back(iOrg.first);
                continue;
            }
            vector<float> output = o->nerveSystem.getOutput(inputs);

            xyz f = make_xyz(output[0], output[1], output[2]);
            for (int i : o->cells) {
                if (p[i].particleType == Cell){
                    
                    xyz front = p[i].links[Front] >= 0 ? p[p[i].links[Front]].r : make_xyz(0, 0, 1);
                    xyz right = p[i].links[Right] >= 0 ? p[p[i].links[Right]].r : make_xyz(1, 0, 0);
                    xyz up = p[i].links[Up] >= 0 ? p[p[i].links[Up]].r : make_xyz(0, 1, 0);
                    xyz back = p[i].links[Back] >= 0 ? p[p[i].links[Back]].r : make_xyz(0, 0, -1);
                    xyz left = p[i].links[Left] >= 0 ? p[p[i].links[Left]].r : make_xyz(-1, 0, 0);
                    xyz down = p[i].links[Down] >= 0 ? p[p[i].links[Down]].r : make_xyz(0, -1, 0);

                    Matrix3 m = getTransform(
                        front - p[i].r,
                        right - p[i].r,
                        up - p[i].r,
                        back - p[i].r,
                        left - p[i].r,
                        down - p[i].r
                    );
                    p[i].f += m.dot(f) * g.moveFactor;
                    
                    //p[i].f += f;
                    p[i].energy -= xyz_len(f) * g.moveCost;
                    p[i].signal *= 0.5f;
                }
            }
            // Hatch eggs if they have enought energy:
            for (int i : eggs) {
                int maxReqEnergy =
                    g.initialCellEnergy *
                    o->genome.getMaxCellsReq() +
                    o->genome.getSize() * g.genomeCost;

                if (p[i].energy >= maxReqEnergy + g.initialCellEnergy) {
                    spawnOrganism(
                        p[i].r, &particleBuffer,
                        p, p[i].organism, &organisms
                    );
                    p[i].energy -= maxReqEnergy;
                    fx->applyParticleArray(pSet);
                }
            }
        }
        for (int i : organismsToRemove)
            organisms.erase(i);
        fx->applyParticleArray(pSet);

        fx->runEach(buoyancy(), pSet);
        fx->runEach(handleEnergy(), pSet);
        fx->runEach(integrate(), pSet);
        
        //if (step % 100 == 0) { ParticleBuffer empty; swap(particleBuffer, empty); }
        for (int i = 0; i<g.nParticles; i++)
        {
            if (p[i].toBuffer && p[i].particleType != Energy) {
                turnIntoBuffer(p[i]);
                particleBuffer.push(i);
                p[i].toBuffer = false;
                fx->applyParticleArray(pSet);
            }
            //else if (step % 100 == 0 && p[i].particleType == Buffer){
            //    particleBuffer.push(i);
            //}

        }

        g.nDetritus = g.nBuffer = g.nEnergy = g.nCells = 0;
        fx->runEach(countParticles(), pSet);

        //If buffer is getting to small, increase it
        //by adding more particles to the simulation
        if (particleBuffer.size() < g.bufferSize) {
            int currentParticleCount = g.nParticles;
            g.nParticles += g.bufferSize;
            printf("Increasing particle array size from %i to %i\n", currentParticleCount, g.nParticles);
            fx->resizeParticleSet(pSet, g.nParticles);
            p = fx->getParticleArray(pSet);
            for (int i = currentParticleCount; i < g.nParticles; i++) {
                turnIntoBuffer(p[i]);
                particleBuffer.push(i);
            }
            fx->applyParticleArray(pSet);
        }
        else if (particleBuffer.size() > g.nParticles / 2) {
            //printf("Decreasing buffer size from %i", g.nParticles);
            int nBuffersAtEnd = 0;
            //printf("End particle type: %i (buffer is %i)\n", p[g.nParticles - nBuffersAtEnd - 1].particleType, Buffer);
            while (p[g.nParticles - nBuffersAtEnd - 1].particleType == Buffer &&
                particleBuffer.size() - nBuffersAtEnd > g.nParticles / 2
            ){
                nBuffersAtEnd++;
            }
            if (nBuffersAtEnd > 0){
                printf("Decreasing particle array size from %i to %i\n", g.nParticles, g.nParticles - nBuffersAtEnd);
                g.nParticles -= nBuffersAtEnd;
                fx->resizeParticleSet(pSet, g.nParticles);
                p = fx->getParticleArray(pSet);
                fx->applyParticleArray(pSet);

                ParticleBuffer empty;
                swap(particleBuffer, empty);

                for (int i = 0; i<g.nParticles; i++)
                    if (p[i].particleType == Buffer)
                        particleBuffer.push(i);
            }
        }

        if (step % 100 == 0) {
            printf("nOrgs: %i\t", organisms.size());
            printf("currgenomeIndex: %i\t", currGenomeIndex);
            printf("buffer: %i (in queue), %i (actual)\t", particleBuffer.size(), g.nBuffer);
            printf("step %d\n", step);
            outputParticles(p, g.nParticles, step);
            fprintf(out, "%i,%i,%i,%i\n", g.nDetritus, g.nBuffer, g.nEnergy, g.nCells);

        }

        if (step % 10000 == 0) fx->outputFrame("temp");

        if (organisms.size() == 0) {
            printf("All organisms died. End of simulation\n");
            break;
        }
    }
    fclose(out);
    delete fx;
    //system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
}
