#include "hip/hip_runtime.h"
//#include "fluidix.h"
#define PARTICLE_BITS 28
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/structures.h"
#include "../lib/genome.h"
#include "../lib/nerveSystem.h"
#include "../lib/linearAlgebra.h"
#include "../lib/io.h"
#include <queue>
#include <ppl.h>

// Transfer amount f from a to b
#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}

#define sphereVolume(r) (4.0f / 3.0f) * r * r * r * PI; 

// Check if particle position is defined correctly
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;

int currGenomeIndex;

// Turn a particle into pellet (dead cell) type
// Previous energy is preserved
#define turnIntoPellet(p) {         \
    p.particleType = Pellet;        \
    p.density = g.fluidDensity * 2; \
    p.organism = -1;                \
}                                   \

// Turn a particle into energy type
#define turnIntoEnergy(p) {            \
    p.particleType = Energy;           \
    p.r = make_xyz(                    \
        rnd_uniform() * g.w.x,         \
        rnd_uniform() * g.w.y + g.w.y, \
        rnd_uniform() * g.w.z          \
    );                                 \
    p.color = 0.7f;                    \
    p.energy = g.energyParticleEnergy; \
    p.signal = 0.0f;                   \
    p.radius = g.energyParticleRadius; \
    p.density = 10.0f;                 \
}

// Turn a pasticle into buffer type
// Place below arena
#define turnIntoBuffer(p) {            \
    p.particleType = Buffer;           \
    p.r = make_xyz(                    \
        rnd_uniform() * g.w.x,         \
        rnd_uniform() * g.w.y - g.w.y, \
        rnd_uniform() * g.w.z          \
    );                                 \
    p.density = g.fluidDensity * 2;    \
    p.color = 0.5f;                    \
    p.radius = 1.0f;                   \
    p.organism = -1;                   \
}

// Initialization function
// Turn all particles into energy particles
FUNC_EACH(init,
    turnIntoEnergy(p);
)

// Update position r of particles given
// velocity v and force f
FUNC_EACH(integrate,
    p.v += p.f * g.dt;
    p.r += p.v * g.dt;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
)

// Decrease energy through metabolism in
// cells and decay in pellets.
// cell --> pellet --> buffer
FUNC_EACH(handleEnergy,
    switch (p.particleType) {
    case Cell:
        p.energy -= p.metabolism * g.dt;
        if (p.energy < g.minCellEnergy )
            turnIntoPellet(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Pellet:
        p.energy -= g.cellDecayRate * g.dt;
        if (p.energy <= g.minPelletEnergy)
            p.toBuffer = true;
        break;
    }
)

// Particles float depending on their density
FUNC_EACH(buoyancy,
    float volume = sphereVolume(p.radius);
    p.f.y += (g.fluidDensity - p.density) * volume; //buoyancy
    p.f.y += g.gravity * volume * p.density;        //gravity
)

FUNC_EACH(countParticles,
    switch (p.particleType) {
    case Cell:
        addInteger(g.nCells, 1); break;
    case Pellet:
        addInteger(g.nPellets, 1); break;
    case Buffer:
        addInteger(g.nBuffer, 1); break;
    case Energy:
        addInteger(g.nEnergy, 1); break;
    }
)

// Periodic boundary conditions
FUNC_EACH(boundary,
    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        turnIntoBuffer(p);
        p.toBuffer = true;
    } if (p.particleType != Buffer) {
        if (p.r.x < 0)     p.r.x = g.w.x;
        if (p.r.x > g.w.x) p.r.x = 0;
        if (p.r.z < 0)     p.r.z = g.w.z;
        if (p.r.z > g.w.x) p.r.z = 0;

        if (p.particleType == Energy) {
            if (p.r.y < 0) {
                p.toBuffer = true;
            }
        }
        else {
            if (p.r.y < 0) {
				p.v.y = 0.9f * (0 - p.r.y) / g.dt;
				p.r.y = 0;
			}
            if (p.r.y > g.w.y) {
				p.v.y = 0.9f * (g.w.y - p.r.y) / g.dt;
				p.r.y = g.w.y;
			}
        }
    }
    else if (p.r.y < -2 * g.w.y) p.r.y += g.w.y;
)

// Let particle a eat particle b
#define consumeParticle(a, b) {      \
    addFloat(a.energy, b.energy);    \
    b.energy = 0;                    \
    b.toBuffer = true;               \
}

// For each particle within a predifined distance
FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        xyz f = u * maxf(
            (g.repulsiveForce * (1 - dr / (p1.radius + p2.radius))),
            0
        );
        if (p1.particleType == Cell &&
            p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                bool neighbours = false;
                for (int n = 0; n < 6; n++) {
                    if (p1.links[n] == p2_index || p2.links[n] == p1_index) {
                        neighbours = true;
                        break;
                    }
                }
                if (neighbours) {
                    //Spring force between neighbours
                    f = -u * ((dr - (p1.radius + p2.radius)/2) * g.springForce);

                    //Energy transmission
                    float p1Surplus = maxf(p1.energy - g.minCellEnergy, 0);
                    float p2Surplus = maxf(p2.energy - g.minCellEnergy, 0);
                    transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                    transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
                }
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoPellet(p2);
                    transmitFloat(p2.energy, p1.energy, 0.05f);
                }
                if (p2.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoPellet(p2);
                    transmitFloat(p1.energy, p2.energy, 0.05f);
                }
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Pellet)
                ) consumeParticle(p1, p2)
        }
        //If p2 is a cell
        else if (p2.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Pellet)
                ) consumeParticle(p2, p1)
        }

        if (p1.particleType == Cell && p1.type == Sense) addFloat(p1.signal, 1.0f/dr);
        if (p2.particleType == Cell && p2.type == Sense) addFloat(p1.signal, 1.0f/dr);

        addVector(p1.f, f);
        addVector(p2.f, -f);
    }
)

// Collision with terrain
FUNC_SURFACE(collideGround,
	if (p.particleType != Energy){
		if (dr > 1) dr = 1;
		p.f += g.groundRepulsiveForce * u * dr;
	}
)

// Initialize particle as cell
void setDefaultCellValues(Particle *cell) {
    //cell->radius = 1.0f;
    cell->energy = g.initialCellEnergy;
    //cell->density = g.fluidDensity * 1.10f;
    cell->particleType = Cell;
}

// Given a phenotype network output, apply it to the cell
bool applyPhenotype(vector<float> output, Particle *cell) {
    // If cell should not exist, return
    if (output[N_CELL_TYPES] < g.cellExistenceThreshold)
        return false;
    float radius = output[N_CELL_TYPES + 1];
    cell->radius = clamp(radius, 0.5f, 2.0f);
    float volume = sphereVolume(cell->radius);
    float mass = 1.0f;
    cell->density = mass/volume;

    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    // If no outputs had a positive value, return
    if (max <= 0) {
        return false;
    }
    switch (cell->type) {
    case Photo:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 5.0f;
        break;
    case Digest:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 5.0f;
        break;
    case Fat:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 10.0f;
        break;
    case Sense:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
        break;
    case Egg:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 1000.0f;
        break;
    case Vascular:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 3.0f;
        break;
    case Sting:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 5.0f;
    }

    return true;
}

// Remove cell links from and to cell
void disconnectCell(Particle *p, int cell, int code) {
    for (int i = 0; i < 6; i++) {
        if (p[cell].links[i] >= 0)
            p[p[cell].links[i]].links[(i + 3) % 6] = code;
    }
    turnIntoBuffer(p[cell]);
}
void emptyCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -1);
}
void deadCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -2);
}

// Helper function to get the 1-dimensional index
// given x,y,z and box size br
int getIdxFromCoord(int x, int y, int z, int3 br)
{
    x += br.x; y += br.y; z += br.z;
    int lY = 2 * br.y + 1;
    int lZ = 2 * br.z + 1;
    return x*lY*lZ + y*lZ + z;
}
#define iFromCoord(x,y,z) cellBuff.at(getIdxFromCoord(x,y,z,br))

// Create cells of an organism given a genome and a nervous system
pair<int, vector<int>> createCellsFromGenotype(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, Genome *genome, NerveSystem *nerveSys, OrganismMap *organisms)
{
    int nParticlesNeeded = genome->getMaxCellsReq();
    if (nParticlesNeeded > particleBuffer->unsafe_size()) {
        cerr << "Not enought particles in buffer\n" << endl;
    }
    vector<int> cellBuff;
    while (nParticlesNeeded) {
        int particle;
        if (particleBuffer->try_pop(particle)) {
            cellBuff.push_back(particle);
            nParticlesNeeded--;
        }
    }

    int3 br = genome->getBoundingRadius();

    int organismID = currGenomeIndex++;
    vector<int> removedCells;
    vector<int> addedCells;

    int nSensors = 0;

    for (int x = -br.x; x <= br.x; x++)
    for (int y = -br.y; y <= br.y; y++)
    for (int z = -br.z; z <= br.z; z++) {
        Particle *cell = &p[iFromCoord(x, y, z)];
        cell->organism = organismID;
        cell->r = origin + make_xyz(x, y, z);
        cell->energy = g.initialCellEnergy;
        cell->metabolism = g.cellMetabolism +
            g.nerveCost * nerveSys->getSize();
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x, y, z)));

        vector<float> output = genome->getOutput(input);

        if (applyPhenotype(output, cell)) {
            cell->links[Left] = x + 1 < br.x ? iFromCoord(x + 1, y, z) : -1;
            cell->links[Up] = y + 1 < br.y ? iFromCoord(x, y + 1, z) : -1;
            cell->links[Back] = z + 1 < br.z ? iFromCoord(x, y, z + 1) : -1;
            cell->links[Right] = x - 1 >= 0 ? iFromCoord(x - 1, y, z) : -1;
            cell->links[Down] = y - 1 >= 0 ? iFromCoord(x, y - 1, z) : -1;
            cell->links[Front] = z - 1 >= 0 ? iFromCoord(x, y, z - 1) : -1;

            if (cell->type == Sense)
                nSensors++;

            float volume = cell->radius * cell->radius * cell->radius * PI * 4 / 3;
            cell->metabolism += volume * 0.05;
            addedCells.push_back(iFromCoord(x, y, z));
        }
        else
            removedCells.push_back(iFromCoord(x, y, z));
    }
    for (int i : removedCells)
        emptyCellPos(p, i);

    nerveSys->updateInputs(nSensors);

    return pair<int, vector<int>>(organismID, addedCells);
}

//Initialize new organism (without parent)
int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, Genome genome, NerveSystem nerveSys, OrganismMap *organisms)
{
    pair<int, vector<int>> o = createCellsFromGenotype(
        origin, particleBuffer, p, &genome, &nerveSys, organisms
    );
    int organismID    = o.first;
    vector<int> cells = o.second;

    Organism organism = { genome, nerveSys, cells, -1, 1000 };

    //Add organism to organism map
    organisms->emplace(organismID, organism);

    //Output organism to disk
    outputOrganism(&organism, organismID);

    return organismID;
}

// Initialize new organism from parent
int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, int parent, OrganismMap *organisms)
{
	Genome genome;
    NerveSystem nerveSys;

	if(parent == -1) {
		int3 gridDim = g.initialOrganismDimensions;

		// Define number of in- and outputs
		int inputs = g.nGenomeInputs;       // X, Y, Z, Dist
		int nonCelltypeOutputs = 2;         // Cell existence, cell radius
		int outputs = N_CELL_TYPES + nonCelltypeOutputs;
		genome = Genome(inputs, outputs, gridDim);

        int nerveOutputs = 3;
        nerveSys = NerveSystem(nerveOutputs);
	}
	else {
		genome = Genome(organisms->at(parent).genome);
        nerveSys = NerveSystem(organisms->at(parent).nerveSystem);
	}

    genome.mutate();
    nerveSys.mutate();

    pair<int, vector<int>> o = createCellsFromGenotype(
        origin, particleBuffer, p, &genome, &nerveSys, organisms
        );
    int organismID    = o.first;
    vector<int> cells = o.second;

    Organism organism = { genome, nerveSys, cells, parent, 1000 };

    //Add organism to organism map
    organisms->emplace(organismID, organism);

    //Output organism to disk
    outputOrganism(&organism, organismID);

    return organismID;
}

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

int generateTerrain(Fluidix<> *fx){
    exponential_distribution<float> rndUniform(1);

    int terrDimX = 10;
    int terrDimZ = 10;

    int nParticles = (terrDimX*terrDimZ * 2);

    int meshParticles = fx->createParticleSet(nParticles);
    int meshLinks     = fx->createLinkSet();
    Particle *mesh    = fx->getParticleArray(meshParticles);

    float dx = g.w.x / (terrDimX-1);
    float dz = g.w.z / (terrDimZ-1);

    float margin = 1.2f;
    float shiftX = ((margin - 1)*g.w.x) / 2;
    float shiftZ = ((margin - 1)*g.w.z) / 2;

    for (int x = 0; x < terrDimX; x++)
    for (int z = 0; z < terrDimX; z++){
        int i = x*terrDimZ + z;
        mesh[i].r = make_xyz(
            (x*dx)*margin - shiftX,
            rndUniform(rndGen) * 10 + 10,
            (z*dz)*margin - shiftZ
        );
        mesh[i + nParticles/2].r = make_xyz(
            x*dx,
            0,
            z*dz
        );

        //Link terrain particles together:
        int s = (x - 1)*terrDimZ + z;
        int w = x*terrDimZ + (z - 1);
        int sw = (x - 1)*terrDimZ + (z - 1);

        if ((x - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, s);
        if ((z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, w);
        if ((x - 1) >= 0 && (z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, sw);

        if ((x - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, s + (nParticles / 2));
        if ((z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, w + (nParticles / 2));
        if ((x - 1) >= 0 && (z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, sw + (nParticles / 2));

        if (x % (terrDimX - 1) == 0 || z % (terrDimZ - 1) == 0){
            fx->addLink(meshLinks, meshParticles, i, meshParticles, i + (nParticles / 2));
            //if (x > 0 && z > 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, s + (nParticles / 2));
        }
        fx->applyParticleArray(meshParticles);
        fx->outputFrame("output");
    }
    return meshLinks;
}

// Initialize a random organism
int initializeOrganism(ParticleBuffer *particleBuffer, Particle *p, OrganismMap *organisms)
{
    xyz origin = make_xyz_uniform() * int3_to_xyz(g.w);
    origin.y /= 2;
    origin.y += g.w.y / 2;

    return spawnOrganism(origin, particleBuffer, p, -1, organisms);
}

Matrix3 getTransform(xyz front, xyz right, xyz up, xyz back, xyz left, xyz down) {
    return Matrix3(
        xyz_norm(right - left),
        xyz_norm(up - down),
        xyz_norm(front - back)
    );
}

int main() {
    // Create Fluidix library object
    Fluidix<> *fx = new Fluidix<>(&g);

    // Load configuration file
    g = loadConfig("conf.txt");

    // Create a particle set with number of particles
    // equal to g.nParticles
    int pSet = fx->createParticleSet(g.nParticles);

    //int terrain = generateTerrain(fx);

    currGenomeIndex = 0;
    OrganismMap organisms;

    // Initialize all particles
    fx->runEach(init(), pSet);

    ParticleBuffer particleBuffer;
    Particle *p = fx->getParticleArray(pSet);

    int initialBufferSize =
        (g.initialOrganismDimensions.x * 2 + 1) *
        (g.initialOrganismDimensions.y * 2 + 1) *
        (g.initialOrganismDimensions.z * 2 + 1) *
        g.nInitialOrganisms +
        g.bufferSize;

    // Turn a large enought number of the particles into buffer
    for (int i = 0; i < initialBufferSize; i++) {
        turnIntoBuffer(p[i]);
        p[i].r.y -= g.w.y;
        particleBuffer.push(i);
    }

    loadOrg("initOrg.json", &particleBuffer, p, &organisms);

    //for (int i = 0; i < g.nInitialOrganisms; i++) {
    //    initializeOrganism(&particleBuffer, p, &organisms);
    //}

    fx->applyParticleArray(pSet);

    FILE *out = fopen("countCells.csv", "w");
    fprintf(out, "nPellets,nBuffer,nEnergy,nCells\n");
    for (int step = 0; step < g.nSteps; step++) {
        fx->runEach(boundary(), pSet);
        //fx->runSurface(collideGround(), terrain, pSet);
        fx->runPair(particlePair(), pSet, pSet, g.interactionRange);

        p = fx->getParticleArray(pSet);
        vector<int> organismsToRemove;
        for (auto& iOrg : organisms) {
            Organism *o = &iOrg.second;

            o->health -= g.dt;

            vector<float> inputs;
            vector<int> eggs;
            int nLiving = 0;
            int nDead = 0;
            for (int i : o->cells){
                if (p[i].particleType == Cell){
                    if (p[i].type == Sense)
                        inputs.push_back(p[i].signal);
                    if (p[i].type == Egg)
                        eggs.push_back(i);
                    nLiving++;
                }
                else
                nDead++;
            }
            if (o->health <= 0 || nDead > nLiving) {
                for (int i : o->cells)
                if (p[i].particleType == Cell)
                    turnIntoPellet(p[i]);
                organismsToRemove.push_back(iOrg.first);
                continue;
            }
            vector<float> output = o->nerveSystem.getOutput(inputs);

            xyz f = make_xyz(output[0], output[1], output[2]);
            for (int i : o->cells) {
                if (p[i].particleType == Cell){
                    xyz front = p[i].links[Front] >= 0 ? p[p[i].links[Front]].r : make_xyz(0, 0, 1);
                    xyz right = p[i].links[Right] >= 0 ? p[p[i].links[Right]].r : make_xyz(1, 0, 0);
                    xyz up = p[i].links[Up] >= 0 ? p[p[i].links[Up]].r : make_xyz(0, 1, 0);
                    xyz back = p[i].links[Back] >= 0 ? p[p[i].links[Back]].r : make_xyz(0, 0, -1);
                    xyz left = p[i].links[Left] >= 0 ? p[p[i].links[Left]].r : make_xyz(-1, 0, 0);
                    xyz down = p[i].links[Down] >= 0 ? p[p[i].links[Down]].r : make_xyz(0, -1, 0);


                    Matrix3 m = getTransform(
                        front - p[i].r,
                        right - p[i].r,
                        up - p[i].r,
                        back - p[i].r,
                        left - p[i].r,
                        down - p[i].r
                    );
                    p[i].f += m.dot(f) * g.moveFactor;
                    //printf("Energy before: %.2f\t", p[i].energy);
                    p[i].energy -= xyz_len(f) * g.moveCost;
                    //printf("Energy after: %.2f\n", p[i].energy);
                    p[i].signal *= 0.5f;
                }
            }
            // Hatch eggs if they have enought energy:
            for (int i : eggs) {
                int maxReqEnergy =
                    g.initialCellEnergy *
                    o->genome.getMaxCellsReq() +
                    o->genome.getSize() * g.genomeCost;

                if (p[i].energy >= maxReqEnergy + g.initialCellEnergy) {
                    spawnOrganism(
                        p[i].r, &particleBuffer,
                        p, p[i].organism, &organisms
                    );
                    p[i].energy -= maxReqEnergy;
                    fx->applyParticleArray(pSet);
                }
            }
        }
        for (int i : organismsToRemove)
            organisms.erase(i);
        fx->applyParticleArray(pSet);

        //fx->runEach(moveParticle(), pSet);
        fx->runEach(buoyancy(), pSet);
        fx->runEach(handleEnergy(), pSet);
        fx->runEach(integrate(), pSet);
        parallel_for (int(0), g.nParticles, [&](int i)
        //for (int i = 0; i<g.nParticles; i++)
        {
            if (p[i].toBuffer) {
                if (particleBuffer.unsafe_size() > g.bufferSize) {
                    turnIntoEnergy(p[i]);
                } else {
                    turnIntoBuffer(p[i]);
                    particleBuffer.push(i);
                }
                p[i].toBuffer = false;
                fx->applyParticleArray(pSet);
            }
        });

        //fx->applyParticleArray(pSet);

        g.nPellets = g.nBuffer = g.nEnergy = g.nCells = 0;
        fx->runEach(countParticles(), pSet);
        fprintf(out, "%i,%i,%i,%i\n", g.nPellets, g.nBuffer, g.nEnergy, g.nCells);

        if (step % 10 == 0) {// && step > 1000000
			//|| step % 10000 == 0) {
            printf("nOrgs: %i\t", organisms.size());
            printf("currgenomeIndex: %i\t", currGenomeIndex);
            printf("buffer: %i (%i)\t", particleBuffer.unsafe_size(), g.nBuffer);
            printf("step %d\n", step);
            outputParticles(p, g.nParticles, step);
        }

        if (organisms.size() == 0) {
            printf("All organisms died. End of simulation\n");
            break;
        }
    }
    fclose(out);
    delete fx;

    //system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
}
