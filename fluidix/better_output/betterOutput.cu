#include "hip/hip_runtime.h"
//#include "fluidix.h"
#define PARTICLE_BITS 28
#include "C:\Program Files (x86)\Fluidix\include\fluidix.h"
#include "../lib/structures.h"
#include "../lib/genome.h"
#include "../lib/nerveSystem.h"
#include "../lib/io.h"
#include <queue>
#include <ppl.h>

#define transmitFloat(a, b, f) {addFloat(a, -f); addFloat(b, f);}
#define isWeirdParticle(p) (p.r.x != p.r.x || p.r.y != p.r.y || p.r.z != p.r.z)
#define isWeirdParticlePointer(p) (p->r.x != p->r.x || p->r.y != p->r.y || p->r.z != p->r.z)

using namespace std;
using namespace concurrency;


int currGenomeIndex;

#define turnIntoPellet(p) {         \
    p.particleType = Pellet;        \
    p.energy = g.pelletLifetime;    \
    p.density = g.fluidDensity * 2; \
    p.alpha = 0.5f;                 \
    p.organism = -1;                \
}                                   \

#define turnIntoEnergy(p) {            \
    p.particleType = Energy;           \
    p.r = make_xyz(                    \
        rnd_uniform() * g.w.x,         \
        rnd_uniform() * g.w.y + g.w.y, \
        rnd_uniform() * g.w.z          \
    );                                 \
    p.color = 0.7f;                    \
    p.energy = g.energyParticleEnergy; \
    p.signal = 0.0f;                   \
    p.alpha = 0.3f;                    \
    p.radius = g.energyParticleRadius; \
    p.density = 10.0f;                 \
}

#define turnIntoBuffer(p) {            \
    p.particleType = Buffer;           \
    p.r = make_xyz(                    \
        rnd_uniform() * g.w.x,         \
        rnd_uniform() * g.w.y - g.w.y, \
        rnd_uniform() * g.w.z          \
    );                                 \
    p.density = g.fluidDensity * 2;    \
    p.alpha = 0.1f;                    \
    p.color = 0.5f;                    \
    p.radius = 1.0f;                   \
    p.organism = -1;                   \
}

FUNC_EACH(init,
    turnIntoEnergy(p);
)

FUNC_EACH(integrate,
    p.v += p.f * g.dt;
    p.r += p.v * g.dt;
    p.f = make_xyz(0, 0, 0);
    p.v *= 0.97f;
)

FUNC_EACH(handleEnergy,
    switch (p.particleType) {
    case Cell:
        p.energy -= g.cellMetabolism * g.dt;
        if (p.energy < g.minCellEnergy )
            turnIntoPellet(p)
        else if (p.energy > p.maxEnergy)
            p.energy -= (p.energy - p.maxEnergy) * 0.1f;
        break;
    case Pellet:
        p.energy -= g.cellDecayRate * g.dt;
        if (p.energy <= g.minPelletEnergy)
            p.toBuffer = true;
        break;
    }
)

FUNC_EACH(buoyancy,
    float volume = p.radius * p.radius * PI;
    p.f.y += (p.density - g.fluidDensity) * g.gravity * volume;
)


// bouncing hard wall boundary condition
FUNC_EACH(boundary,
    if (p.particleType == Cell && p.type == Egg)
        addInteger(g.nEggs, 1);

    // Check for wierd 1.#R values... NaN?
    if (isWeirdParticle(p)) {
        //printf("Weird particle!!! type (%i:%i)\n", p.particleType, p.type);
        turnIntoBuffer(p);
        p.toBuffer = true;
    } if (p.particleType != Buffer) {       
        if (p.r.x < 0)     p.r.x = g.w.x;
        if (p.r.x > g.w.x) p.r.x = 0;
        if (p.r.z < 0)     p.r.z = g.w.z;
        if (p.r.z > g.w.x) p.r.z = 0;
        
        if (p.particleType == Energy) {
            if (p.r.y < 0) {
                p.toBuffer = true;
            }
        }
        else {
            if (p.r.y < 0) { 
				p.v.y = 0.9f * (0 - p.r.y) / g.dt;
				p.r.y = 0;
			}
            if (p.r.y > g.w.y) {
				p.v.y = 0.9f * (g.w.y - p.r.y) / g.dt;
				p.r.y = g.w.y;
			}
        }
    }
    else if (p.r.y < -2 * g.w.y) p.r.y += g.w.y;
)

#define consumeParticle(a, b) {      \
    addFloat(a.energy, b.energy);    \
    b.energy = 0;                    \
    b.toBuffer = true;               \
}

FUNC_PAIR(particlePair,
    if (p1.particleType != Buffer && p2.particleType != Buffer) {
        xyz f = u * maxf(
            (g.repulsiveForce * (1 - dr / (p1.radius + p2.radius))),
            0
        );
        if (p1.particleType == Cell &&
            p2.particleType == Cell)
        {
            //Cells from the same organism
            if (p1.organism == p2.organism) {
                bool neighbours = false;
                for (int n = 0; n < 6; n++) {
                    if (p1.links[n] == p2_index || p2.links[n] == p1_index) {
                        neighbours = true;
                        break;
                    }
                }
                if (neighbours) {
                    //Spring force between neighbours
                    f = -u * ((dr - (p1.radius + p2.radius)/2) * g.springForce);

                    //Energy transmission
                    float p1Surplus = maxf(p1.energy - g.minCellEnergy, 0);
                    float p2Surplus = maxf(p2.energy - g.minCellEnergy, 0);
                    transmitFloat(p1.energy, p2.energy, p1Surplus * p1.energyOut * p2.energyIn);
                    transmitFloat(p2.energy, p1.energy, p2Surplus * p2.energyOut * p2.energyIn);
                }
            }
            //Cells from different organisms
            else {
                //Kill the other cell if you are sting
                if (p1.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoPellet(p2);
                    transmitFloat(p2.energy, p1.energy, 0.05f);
                }
                if (p2.type == Sting && dr <= (p1.radius + p2.radius)) {
                    //turnIntoPellet(p2);
                    transmitFloat(p1.energy, p2.energy, 0.05f);
                }
            }
        }
        //If p1 is a cell
        else if (p1.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p1.type == Photo && p2.particleType == Energy) ||
                (p1.type == Digest  && p2.particleType == Pellet)
                ) consumeParticle(p1, p2)
        }
        //If p2 is a cell
        else if (p2.particleType == Cell && dr <= (p1.radius + p2.radius)) {
            if ((p2.type == Photo && p1.particleType == Energy) ||
                (p2.type == Digest  && p1.particleType == Pellet)
                ) consumeParticle(p2, p1)
        }

        if (p1.particleType == Cell && p1.type == Sense) addFloat(p1.signal, 1.0f/dr);
        if (p2.particleType == Cell && p2.type == Sense) addFloat(p1.signal, 1.0f/dr);

        addVector(p1.f, f);
        addVector(p2.f, -f);
    }
)

FUNC_SURFACE(collideGround,
	if (p.particleType != Energy){
		if (dr > 1) dr = 1;
		p.f += g.groundRepulsiveForce * u * dr;
	}
)

void setDefaultCellValues(Particle *cell) {
    cell->alpha = 1.0f;
    cell->radius = 1.0f;
    cell->energy = g.initialCellEnergy;
    cell->density = g.fluidDensity * 1.10f;
    cell->particleType = Cell;
}

bool applyPhenotype(vector<float> output, Particle *cell) {
    // If cell should not exist, return
    if (output[N_CELL_TYPES] < g.cellExistenceThreshold)
        return false;

    float max = output[0]; cell->type = (CellType)0;
    for (int j = 1; j<N_CELL_TYPES; j++) {
        if (output[j] > max) {
            max = output[j];
            cell->type = (CellType)j;
        }
    }
    switch (cell->type) {
    case Photo:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 3.0f;
        break;
    case Digest:
        cell->energyIn = 0.01f;
        cell->energyOut = 0.6f;
        cell->maxEnergy = 3.0f;
        break;
    case Fat:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.01f;
        cell->maxEnergy = 10.0f;
        break;
    case Sense:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    case Egg:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 1000.0f;
        break;
    case Vascular:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.2f;
        cell->maxEnergy = 1.0f;
        break;
    case Sting:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
        break;
    default:
        cell->energyIn = 1.0f;
        cell->energyOut = 0.0f;
        cell->maxEnergy = 3.0f;
    }

    return true;
}

void disconnectCell(Particle *p, int cell, int code) {
    for (int i = 0; i < 6; i++) {
        if (p[cell].links[i] >= 0)
            p[p[cell].links[i]].links[(i + 3) % 6] = code;
    }
    turnIntoBuffer(p[cell]);
}
void emptyCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -1);
}
void deadCellPos(Particle *p, int cell) {
    disconnectCell(p, cell, -2);
}

int getIdxFromCoord(int x, int y, int z, int3 br)
{
    x += br.x; y += br.y; z += br.z;
    //int lX = 2 * br.z + 1;
    int lY = 2 * br.y + 1;
    int lZ = 2 * br.z + 1;
    return x*lY*lZ + y*lZ + z;
}
#define iFromCoord(x,y,z) cellBuff.at(getIdxFromCoord(x,y,z,br))

int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, Genome genome, NerveSystem nerveSys, OrganismMap *organisms)
{
    int nParticlesNeeded = genome.getMaxCellsReq();
    if (nParticlesNeeded > particleBuffer->unsafe_size()) {
        printf("Not enought particles in buffer\n");
        return -1;
    }
    vector<int> cellBuff;
    while (nParticlesNeeded) {
        int particle;
        if (particleBuffer->try_pop(particle)) {
            cellBuff.push_back(particle);
            nParticlesNeeded--;
        }
    }

    int3 br = genome.getBoundingRadius();

    int organismID = currGenomeIndex++;
    vector<int> removedCells;
    vector<int> addedCells;

    int nSensors = 0;

    for (int x = -br.x; x <= br.x; x++)
    for (int y = -br.y; y <= br.y; y++)
    for (int z = -br.z; z <= br.z; z++) {
        Particle *cell = &p[iFromCoord(x, y, z)];
        cell->organism = organismID;
        cell->r = origin + make_xyz(x, y, z);
        cell->energy = g.initialCellEnergy;
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x, y, z)));

        vector<float> output = genome.getOutput(input);

        if (applyPhenotype(output, cell)) {
            cell->links[Left] = x + 1 < br.x ? iFromCoord(x + 1, y, z) : -1;
            cell->links[Up] = y + 1 < br.y ? iFromCoord(x, y + 1, z) : -1;
            cell->links[Back] = z + 1 < br.z ? iFromCoord(x, y, z + 1) : -1;
            cell->links[Right] = x - 1 >= 0 ? iFromCoord(x - 1, y, z) : -1;
            cell->links[Down] = y - 1 >= 0 ? iFromCoord(x, y - 1, z) : -1;
            cell->links[Front] = z - 1 >= 0 ? iFromCoord(x, y, z - 1) : -1;

            if (cell->type == Sense)
                nSensors++;
            addedCells.push_back(iFromCoord(x, y, z));
        }
        else
            removedCells.push_back(iFromCoord(x, y, z));
    }
    for (int i : removedCells)
        emptyCellPos(p, i);

    Organism organism = { genome, nerveSys, addedCells, -1};

    //Add organism to organism map
    organisms->emplace(organismID, organism);

    //Output organism to disk
    outputOrganism(&organism, organismID);

    return organismID;
}

// Initialize new organism
int spawnOrganism(
    xyz origin, ParticleBuffer *particleBuffer,
    Particle *p, int parent, OrganismMap *organisms)
{
	Genome genome;
	
	if(parent == -1){
		int3 gridDim = g.initialOrganismDimensions; //genomes[iOrigin].gridDim;

		// Define number of in- and outputs
		int inputs = g.nGenomeInputs;              // X, Y, Z, Dist
		int nonCelltypeOutputs = 1;         // Cell existence
		int outputs = N_CELL_TYPES + nonCelltypeOutputs;
		genome = Genome(inputs, outputs, gridDim);
		//g.mutate(); g.mutate(); g.mutate(); g.mutate(); g.mutate();
	}
	else {
		genome = Genome(organisms->at(parent).genome);
	}

    genome.mutate();
    
    int nParticlesNeeded = genome.getMaxCellsReq();
    if (nParticlesNeeded > particleBuffer->unsafe_size()) {
        printf("Not enought particles in buffer\n");
        return -1;
    }
    vector<int> cellBuff;
    while (nParticlesNeeded) {
        int particle;
        if (particleBuffer->try_pop(particle)) {
            cellBuff.push_back(particle);
            nParticlesNeeded--;
        }
    }

    int3 br = genome.getBoundingRadius();

    int organismID = currGenomeIndex++;
    vector<int> removedCells;
    vector<int> addedCells;
    
    int nSensors = 0;

    for (int x = -br.x; x <= br.x; x++)
    for (int y = -br.y; y <= br.y; y++)
    for (int z = -br.z; z <= br.z; z++) {
        Particle *cell = &p[iFromCoord(x, y, z)];
        cell->organism = organismID;
        cell->r = origin + make_xyz(x, y, z);
        cell->energy = g.initialCellEnergy;
        setDefaultCellValues(cell);

        vector<float> input;
        input.push_back(x);
        input.push_back(y);
        input.push_back(z);
        input.push_back(xyz_len(make_xyz(x,y,z)));

        vector<float> output = genome.getOutput(input);

        if (applyPhenotype(output, cell)) {
            cell->links[Left] = x + 1 < br.x ? iFromCoord(x + 1, y, z) : -1;
            cell->links[Up] = y + 1 < br.y ? iFromCoord(x, y + 1, z) : -1;
            cell->links[Back] = z + 1 < br.z ? iFromCoord(x, y, z + 1) : -1;
            cell->links[Right] = x - 1 >= 0 ? iFromCoord(x - 1, y, z) : -1;
            cell->links[Down] = y - 1 >= 0 ? iFromCoord(x, y - 1, z) : -1;
            cell->links[Front] = z - 1 >= 0 ? iFromCoord(x, y, z - 1) : -1;

            if (cell->type == Sense)
                nSensors++;
            addedCells.push_back(iFromCoord(x, y, z));
        }
        else
            removedCells.push_back(iFromCoord(x, y, z));
    }
    for (int i : removedCells)
        emptyCellPos(p, i);

    NerveSystem nervSys;
    if (parent == -1){
        nervSys = NerveSystem(nSensors, 3);
    } else {
        nervSys = NerveSystem(organisms->at(parent).nerveSystem);
        nervSys.updateInputs(nSensors);
	}
	
	nervSys.mutate();
	
    Organism organism = { genome, nervSys, addedCells, parent };
	
	//Add organism to organism map
    organisms->emplace(organismID, organism);
	
	//Output organism to disk
	outputOrganism(&organism, organismID);

    return organismID;
}

#define printP(chr, p, i) printf("%c\tp[%i].r=(%.2f, %.2f, %.2f)\n", chr, i, p.r.x, p.r.y, p.r.z)

int generateTerrain(Fluidix<> *fx){
    exponential_distribution<float> rndUniform(1);

    int terrDimX = 10;
    int terrDimZ = 10;

    int nParticles = (terrDimX*terrDimZ * 2);

    int meshParticles = fx->createParticleSet(nParticles);
    int meshLinks     = fx->createLinkSet();
    Particle *mesh    = fx->getParticleArray(meshParticles);

    float dx = g.w.x / (terrDimX-1);
    float dz = g.w.z / (terrDimZ-1);

    float margin = 1.2;
    float shiftX = ((margin - 1)*g.w.x) / 2;
    float shiftZ = ((margin - 1)*g.w.z) / 2;

    for (int x = 0; x < terrDimX; x++)
    for (int z = 0; z < terrDimX; z++){
        int i = x*terrDimZ + z;
        mesh[i].r = make_xyz(
            (x*dx)*margin - shiftX,
            rndUniform(rndGen) * 10 + 10,
            (z*dz)*margin - shiftZ
        );
        mesh[i + nParticles/2].r = make_xyz(
            x*dx,
            0,
            z*dz
        );

        //Link terrain particles together:
        int s = (x - 1)*terrDimZ + z;
        int w = x*terrDimZ + (z - 1);
        int sw = (x - 1)*terrDimZ + (z - 1);

        if ((x - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, s);
        if ((z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, w);
        if ((x - 1) >= 0 && (z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, sw);

        if ((x - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, s + (nParticles / 2));
        if ((z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, w + (nParticles / 2));
        if ((x - 1) >= 0 && (z - 1) >= 0) fx->addLink(meshLinks, meshParticles, i + (nParticles / 2), meshParticles, sw + (nParticles / 2));

        if (x % (terrDimX - 1) == 0 || z % (terrDimZ - 1) == 0){
            fx->addLink(meshLinks, meshParticles, i, meshParticles, i + (nParticles / 2));
            //if (x > 0 && z > 0) fx->addLink(meshLinks, meshParticles, i, meshParticles, s + (nParticles / 2));
        }
        fx->applyParticleArray(meshParticles);
        fx->outputFrame("output");
    }
    return meshLinks;
}

int initializeOrganism(ParticleBuffer *particleBuffer, Particle *p, OrganismMap *organisms)
{
    xyz origin = make_xyz_uniform() * int3_to_xyz(g.w);
    origin.y /= 2;
    origin.y += g.w.y / 2;

    return spawnOrganism(origin, particleBuffer, p, -1, organisms);
}

int main() {

    Fluidix<> *fx = new Fluidix<>(&g);

    g = loadConfig("conf.txt");

    int pSet = fx->createParticleSet(g.nParticles);

    //int terrain = generateTerrain(fx);

    currGenomeIndex = 0;
    g.nEggs = 0;
    OrganismMap organisms;

    fx->runEach(init(), pSet);
    ParticleBuffer particleBuffer;
    Particle *p = fx->getParticleArray(pSet);

    int initialBufferSize =
        (g.initialOrganismDimensions.x * 2 + 1) *
        (g.initialOrganismDimensions.y * 2 + 1) *
        (g.initialOrganismDimensions.z * 2 + 1) *
        g.nInitialOrganisms +
        g.bufferSize;

    for (int i = 0; i < initialBufferSize; i++) {
        turnIntoBuffer(p[i]);
        p[i].r.y -= g.w.y;
        particleBuffer.push(i);
    }

    loadOrg("initOrg.json", &particleBuffer, p, &organisms);
    /*
    for (int i = 0; i < g.nInitialOrganisms; i++) {
        initializeOrganism(&particleBuffer, p, &organisms);
    }
    */
    fx->applyParticleArray(pSet);

    int nReboots = 0;
	
    for (int step = 0; step < g.nSteps; step++) {
        g.nEggs = 0;
        fx->runEach(boundary(), pSet);
        //fx->runSurface(collideGround(), terrain, pSet);
        fx->runPair(particlePair(), pSet, pSet, g.interactionRange);


        p = fx->getParticleArray(pSet);
        vector<int> organismsToRemove;
        for (auto& iOrg : organisms) {
            Organism *o = &iOrg.second;
            vector<float> inputs;
            vector<int> eggs;
            int nLiving = 0;
            int nDead = 0;
            for (int i : o->cells){
                if (p[i].particleType == Cell){
                    if (p[i].type == Sense)
                        inputs.push_back(p[i].signal);
                    if (p[i].type == Egg)
                        eggs.push_back(i);
                    nLiving++;
                }
                else
                nDead++;
            }
            if (nDead > nLiving) {
                for (int i : o->cells)
                if (p[i].particleType == Cell)
                    turnIntoPellet(p[i]);
                organismsToRemove.push_back(iOrg.first);
                continue;
            }
            vector<float> output = o->nerveSystem.getOutput(inputs);

            xyz f = make_xyz(output[0], output[1], output[2]);
            for (int i : o->cells) {
                if (p[i].particleType == Cell){
                    p[i].f += f * g.moveFactor;
                    p[i].signal *= 0.5f;
                }
            }
            // Hatch eggs if they have enought energy:
            for (int i : eggs) {
                int maxReqEnergy = 
                    g.initialCellEnergy * 
                    organisms.at(p[i].organism).genome.getMaxCellsReq();

                if (p[i].energy >= maxReqEnergy + g.initialCellEnergy) {
                    spawnOrganism(
                        p[i].r, &particleBuffer,
                        p, p[i].organism, &organisms
                    );
                    p[i].energy -= maxReqEnergy;
                    fx->applyParticleArray(pSet);
                }
            }
        }
        for (int i : organismsToRemove)
            organisms.erase(i);
        fx->applyParticleArray(pSet);

        //fx->runEach(moveParticle(), pSet);
        fx->runEach(buoyancy(), pSet);
        fx->runEach(handleEnergy(), pSet);
        fx->runEach(integrate(), pSet);
        parallel_for (int(0), g.nParticles, [&](int i)
        //for (int i = 0; i<g.nParticles; i++)
        {
            if (p[i].toBuffer) {
                if (particleBuffer.unsafe_size() > g.bufferSize) {
                    turnIntoEnergy(p[i]);                  
                } else {
                    turnIntoBuffer(p[i]);
                    particleBuffer.push(i);
                }
                p[i].toBuffer = false;
                fx->applyParticleArray(pSet);
            }
        });

        //fx->applyParticleArray(pSet);

        if (step % 10 == 0) {// && step > 1000000
			//|| step % 10000 == 0) {
            printf("nEggs: %i\t", g.nEggs);
            printf("currgenomeIndex: %i\t", currGenomeIndex);
            printf("nReboots: %i\t", nReboots);
            printf("step %d\n", step);
            //if (currGenomeIndex - g.nInitialOrganisms - nReboots > 0)
                outputParticles(p, g.nParticles, step);
        }

        if (!g.nEggs) break;
    }
    delete fx;

    //system("shutdown -s -c \"Simulation done, shutting down in two minutes\" -t 120");
}
